#include "hip/hip_runtime.h"
#ifndef MCTS_Tests_CU
#define MCTS_Tests_CU

#include "../../Games/SRC/Game.cpp"
#include "../../Games/SRC/TTT/TTT.cpp"
#include "../../Games/SRC/UTTT/UTTT.cpp"
#include "../SRC/MCHS.cu"


#include <iostream>
#include <chrono>
//#include "chrono_io"



/*
//Test TTT with MCTS
TTT_Player Player0 = TTT_Player(0,'X');
TTT_Player Player1 = TTT_Player(1,'O');
Game *_Game = new TTT({&Player0,&Player1});

TreeSimulation *Sim = new MCTS(_Game);
Sim->Search(15000,&Player0);

//delete &Player0;
//delete &Player1;
//delete _Game;
delete Sim;
*/
bool UTTT_Player_Init(){
  //std::cout << "Hello World!";
  //UTTT_Player Player0 = UTTT_Player(0,'X');
  //delete Player0;
  return 0;
}
/*

  TTT_Player Player0 = TTT_Player(0,'X');
  TTT_Player Player1 = TTT_Player(1,'O');

  //Player* Player0 = static_cast<Player*>(&TTTPlayer0);
  //Player* Player1 = static_cast<Player*>(&TTTPlayer1);

  TTT *_Game = new TTT({&Player0,&Player1});


  MCHS<TTT,TTT_Player> *Sim = new MCHS<TTT,TTT_Player>(_Game,{&Player0,&Player1});
  Sim->Search(100000);
*/
/*
UTTT_Player Player0 = UTTT_Player(0,'X');
UTTT_Player Player1 = UTTT_Player(1,'O');

//Player* Player0 = static_cast<Player*>(&TTTPlayer0);
//Player* Player1 = static_cast<Player*>(&TTTPlayer1);

UTTT *_Game = new UTTT({&Player0,&Player1});

MCHS<UTTT,UTTT_Player> *Sim = new MCHS<UTTT,UTTT_Player>(_Game,{&Player0,&Player1});
Sim->Search(1000000);

//delete &Player0;
//delete &Player1;
//delete _Game;
delete Sim;
*/


/*
TODO: Fix Rotating Winner priority.
As of right now, Both players are attempting to give Player0 the win.
+I belive its fixed, need further testing, MCTS_Node values are (negative).
*/
int main(int argc, char *argv[]) {
  long int SearchDepth = 10;
  bool DisplayResults = false;
  for (int i = 1; i < argc; i++) {

      if (strcmp(argv[i],"-sd")==0) {
          SearchDepth = atol(argv[i+1]);
          printf("SearchDepth: %ld",SearchDepth);
      } else if (strcmp(argv[i],"-d")==0) {
          DisplayResults = true;
      }

  }

  std::clock_t    start;
  start = std::clock();

  UTTT_Player Player0 = UTTT_Player(0,'X');
  UTTT_Player Player1 = UTTT_Player(1,'O');

  //Player* Player0 = static_cast<Player*>(&TTTPlayer0);
  //Player* Player1 = static_cast<Player*>(&TTTPlayer1);

  UTTT *_Game = new UTTT({&Player0,&Player1});

  MCHS<UTTT,UTTT_Player> *Sim = new MCHS<UTTT,UTTT_Player>(_Game,{&Player0,&Player1});
  Sim->Search(SearchDepth);

  if(DisplayResults){
    Sim->DisplayStats(1);
  }
  //delete &Player0;
  //delete &Player1;
  //delete _Game;

  std::cout << "Time: " << (std::clock() - start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;

  //delete &Player0;
  //delete &Player1;
  //delete _Game;
  delete Sim;

 return 0;
}

#endif //MCTS_Tests_CU
