#include "hip/hip_runtime.h"
#ifndef MCTS_CU
#define MCTS_CU


#include <iostream>
#include <string>
#include <list>
#include <cmath>
#include <bits/stdc++.h>

#include "TreeSearch.cu"



#define Pause int ASDF; std::cin >> ASDF;

class MCTS_Node
{
private:

public:
    int    NodeVisits;
    double ValueSum;
    Game* GivenGame = NULL;

    MCTS_Node*           Parent       = NULL;
    MCTS_Node*           RollOutChild = NULL;
    std::list<MCTS_Node*> Children;
    MCTS_Node(){
      GivenGame  = NULL;
      Children   = {};
      NodeVisits = 0;
      ValueSum   = 0;
    }
    MCTS_Node(Game* Instance){
      GivenGame  = Instance;
      Children   = {};
      NodeVisits = 0;
      ValueSum   = 0;
    }

    ~MCTS_Node(){
      for (MCTS_Node* Node : Children){
        delete Node;
      }
      if (RollOutChild != NULL)
      {
          delete RollOutChild;
      }
      delete GivenGame;
    }
    double     Find_UCB1();
    MCTS_Node* Find_MAX_UCB1_Child();
    MCTS_Node* RollOut();
    int        AddChildren(std::list<Game*> PossibleMoves);
    void       BackPropagation(double GivenPlayer);
    double     GetAverageValue();
    void       DisplayTree();
    void       DisplayTree(int Depth);
    void       DisplayStats();
};

MCTS_Node* get(std::list<MCTS_Node*> _list, int _i){
    std::list<MCTS_Node*>::iterator it = _list.begin();
    for(int i=0; i<_i; i++){
        ++it;
    }
    return *it;
}


MCTS_Node* MCTS_Node::Find_MAX_UCB1_Child(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  MCTS_Node* HighestNode  = NULL;

  for (MCTS_Node* Node : Children){
      NodesValue = Node->Find_UCB1();

      if (HighestValue < NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}

//Preform MonteCarlo's UCB1 evaluation algorithm on a given node.
double MCTS_Node::Find_UCB1(){
  double ExploreBy = 1.4142;
  if(NodeVisits == 0)
	{
		return INT_MAX;
	}
  float _NodeVisits;
  if (Parent != NULL){
    _NodeVisits = Parent->NodeVisits;
  }
  else{
    _NodeVisits = 0;
  }
  //Preform UCB1 Formula
  double Value = (ValueSum/NodeVisits) + ExploreBy*sqrt(log(_NodeVisits/NodeVisits));
/*
printf("Value:%f\n", Value);
printf("\tNodeVisits:%i\n", NodeVisits);
printf("\tValueSum:%f\n", ValueSum);
*/
  return Value;
}



//For each element within a list of PossibleInstances(Different Game States)
//Add as different Childeren
int MCTS_Node::AddChildren(std::list<Game*> PossibleInstances){
  int ChildrenAdded = 0;
  MCTS_Node* NewNode;
  for (Game* Instance : PossibleInstances){
      if(Instance != NULL)
      {
        NewNode = new MCTS_Node(Instance);
        NewNode->Parent = this;
        Children.push_back(NewNode);
        ChildrenAdded++;
      }
  }
  return ChildrenAdded;
}


MCTS_Node* MCTS_Node::RollOut(){

  Game* RollOutGame = GivenGame->CopyGame();
  RollOutGame->RollOut();

  //printf("RO_WinningPlayer:%p\n",RollOutGame->WinningPlayer);
  //TODO Check if game is finished
  RollOutChild = new MCTS_Node(RollOutGame);
  RollOutChild->Parent = this;
  return RollOutChild;
}

void MCTS_Node::BackPropagation(double EvaluatedValue)
{
  NodeVisits++;
  ValueSum += EvaluatedValue;
  if (Parent != NULL)
  {
    Parent->BackPropagation(EvaluatedValue);
  }
}

double MCTS_Node::GetAverageValue()
{
  return ValueSum/NodeVisits;
}
void MCTS_Node::DisplayStats(){
  std::cout << "----------------------------------------\n";
  printf("ValueSum:%f\n", ValueSum);
  printf("\tNodeVisits:%i\n", NodeVisits);
  printf("\tValueSum:%f\n", ValueSum);
  std::cout << GivenGame->Generate_StringRepresentation();
}

void MCTS_Node::DisplayTree(int Depth){

  std::cout << "Displaying Depth:" << Depth << "\n";
  std::cout << "Children length:" << Children.size() << "\n";
  if (Children.size() > 0){
    for (MCTS_Node* Child : Children) { // c++11 range-based for loop
         Child->DisplayStats();
      }
    if((Depth-1)>0){
      for (MCTS_Node* Child : Children) { // c++11 range-based for loop
           Child->DisplayTree(Depth-1);
        }
      }
  }
}

void MCTS_Node::DisplayTree(){
  for (MCTS_Node* Child : Children) { // c++11 range-based for loop
      Child->DisplayStats();
    }
  std::cout << "----------------------------------------\n";
  std::cout << GivenGame->Generate_StringRepresentation();
  for (MCTS_Node* Child : Children) { // c++11 range-based for loop
       Child->DisplayTree();
    }

}






class MCTS : public TreeSimulation
{
public:
  double Value;
  double Visits;
  Game* GivenGame;
  Player* GivenPlayer;

  //MCTS_Node* TransversedNode;
  MCTS_Node* HeadNode;
  std::list<MCTS_Node*>MCTS_List;
  Game* SimulatedGame;

    MCTS(Game*_Game){
      Value  = 0;
      Visits = 0;

      //HeadNode  = NULL;
      HeadNode  = new MCTS_Node(_Game);
      GivenGame = _Game;
    }

    ~MCTS(){
      delete HeadNode;
    }
    MCTS_Node* Algorithm(MCTS_Node* TransversedNode);
    void EvaluateTransversal(MCTS_Node* TransversedNode,Player* GivenPlayer);
    //double BackPropagation(MCTS_Node* TransversedNode,double GivenPlayer);
    void Search(int Depth,Player* GivenPlayer);
    void ParallelSearch(int Depth);
    //MCTS_Node* Find_Highest_UCB1(std::list<MCTS_Node*>MCTS_List);
    void GetPossibleMoves();

    void CreateChildren();
    void TreeTraversal();
    void CreateNode();
    void RollOut();
};







void MCTS::GetPossibleMoves()
{
    std::list<GameMove*> Moves = SimulatedGame->PossibleMoves();
    std::list<Game*> Games = SimulatedGame->PossibleGames();
}



MCTS_Node* MCTS::Algorithm(MCTS_Node* TransversedNode)
{
  /*
    Helper Function for MCTS::Search & EvaluateTransversal.
    Performs an itteration of the MCTS Algorithm on 'TransversedNode'
  */


//TransversedNode->Children.size()
//int Leaf =TransversedNode->Children.size();

  //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
  /*
  std::cout << "TransversedNode:" <<TransversedNode << "\n";
  std::cout << "NodeVisits:" <<TransversedNode->NodeVisits << "\n";
  std::cout << "Children:"   <<TransversedNode->Children.size() << "\n";
  */


  //Pause;


  if(TransversedNode->Children.size() == 0){
    //If Node is LeafNode
    //std::cout << "LeafNode Detected  :"   << TransversedNode << "\n";

    if(TransversedNode->NodeVisits == 0){
      //std::cout << "About to rool out on:"   << TransversedNode << "\n";

      return TransversedNode->RollOut();
    }


    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
    std::list<Game*> Games = TransversedNode->GivenGame->PossibleGames();
    //std::cout << "Adding Children Size:" << Games.size() << "\n";
    if (Games.size() == 0)
    {
      return TransversedNode;
    }
    TransversedNode->AddChildren(Games);


    MCTS_Node* NextNode = *TransversedNode->Children.begin();
    //std::cout << "Selecting Next Node:" <<NextNode << "\n";
    return Algorithm(NextNode);
    //return NULL;

  }
  else{

    //Not Leaf Node, Transverse TreeSearch: Find Max Child UCB1 value.
    MCTS_Node* MAXNode = TransversedNode->Find_MAX_UCB1_Child();
    //TODO: Implement Node Visits within BackProp...
    //TransversedNode->NodeVisits++;
    return Algorithm(MAXNode);
    //return NULL;
  }
}


//


void MCTS::EvaluateTransversal(MCTS_Node* TransversedNode,Player* GivenPlayer)
{
/*
  Helper Function for MCTS::Search.
  Performs an itteration of the MCTS Algorithm on the parameter 'TransversedNode'
  and tests if the current Player is the winner of the transversal.
  Depending on the winner of the Transversal/RollOut aggregates the appropriate
  Value through MCTS's Back Propagation.
*/
    TransversedNode = Algorithm(TransversedNode);
    double EvaluatedValue = -1;


    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();

    if(TransversedNode->GivenGame->TestForWinner() == GivenPlayer)
    {
      EvaluatedValue = 1;
    }
    else if( TransversedNode->GivenGame->WinningPlayer == NULL )
    {
      //printf("Draw Game\n");
      EvaluatedValue = 0;
    }
    TransversedNode->BackPropagation(EvaluatedValue);
}



void MCTS::Search(int Depth,Player* GivenPlayer)
{
  /*
    Interface to initate MCTS Tree Searches.
    Given Depth,
  */
    std::cout << "Searching Depth:" << Depth << "\n";
    for (int i = 0; i < Depth; i++) {
      //printf("\tDepth: %d\n",i);
      EvaluateTransversal(HeadNode,GivenPlayer);
    }

    HeadNode->DisplayTree(1);


}


void MCTS::ParallelSearch(int Depth)
{
    std::cout << "Searching Depth:" << Depth << "\n";

}


#endif //MCTS_CU
