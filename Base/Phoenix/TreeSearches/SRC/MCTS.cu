#include "hip/hip_runtime.h"
#ifndef MCTS_CU
#define MCTS_CU


#include <iostream>
#include <string>
#include <list>
#include <cmath>
#include <bits/stdc++.h>

#include "TreeSearch.cu"

#include "../../Games/SRC/Game.cu"
#include "../../Games/SRC/TTT.cu"
#include "../../Games/SRC/UTTT.cu"



class MCTS_Node
{
private:

public:
    int    NodeVisits;
    double ValueSum;
    Game* GivenGame;

    MCTS_Node*           Parent;
    std::list<MCTS_Node*> Children;
    MCTS_Node(){
      GivenGame  = NULL;
      Children   = {};
      Parent     = NULL;
      NodeVisits = 0;
      ValueSum   = 0;
    }
    MCTS_Node(Game* Instance){
      GivenGame  = Instance;
      Children   = {};
      Parent     = NULL;
      NodeVisits = 0;
      ValueSum   = 0;
    }

    ~MCTS_Node(){
      for (MCTS_Node* Node : Children){
        delete Node;
      }
      delete GivenGame;
    }
    double     Find_UCB1();
    MCTS_Node* Find_MAX_UCB1_Child();
    void       RollOut();
    int        AddChildren(std::list<Game*> PossibleMoves);

};

MCTS_Node* get(std::list<MCTS_Node*> _list, int _i){
    std::list<MCTS_Node*>::iterator it = _list.begin();
    for(int i=0; i<_i; i++){
        ++it;
    }
    return *it;
}


MCTS_Node* MCTS_Node::Find_MAX_UCB1_Child(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  MCTS_Node* HighestNode  = NULL;

  for (MCTS_Node* Node : Children){
      NodesValue = Node->Find_UCB1();
      if (HighestValue < NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}

//Preform MonteCarlo's UCB1 evaluation algorithm on a given node.
double MCTS_Node::Find_UCB1(){
  double ExploreBy = 1.4142;
  if(NodeVisits == 0)
	{
		return INT_MAX;
	}
  float _NodeVisits;
  if (Parent != NULL){
    _NodeVisits = Parent->NodeVisits;
  }
  else{
    _NodeVisits = 0;
  }
  //Preform UCB1 Formula
  return (ValueSum/NodeVisits) + ExploreBy*sqrt(log(_NodeVisits/NodeVisits));
}



//For each element within a list of PossibleInstances(Different Game States)
//Add as different Childeren
int MCTS_Node::AddChildren(std::list<Game*> PossibleInstances){
  int ChildrenAdded = 0;
  MCTS_Node NewNode;
  for (Game* Instance : PossibleInstances){
      NewNode = MCTS_Node(Instance);
      NewNode.Parent = this;
      Children.push_back(&NewNode);
      ChildrenAdded++;
  }
  return ChildrenAdded;
}

void MCTS_Node::RollOut(){
  GivenGame->RollOut();
}




class MCTS : public TreeSimulation
{
public:
  double Value;
  double Visits;
  Game* GivenGame;
  Player* GivenPlayer;

  //MCTS_Node* TransversedNode;
  MCTS_Node* HeadNode;
  std::list<MCTS_Node*>MCTS_List;
  Game* SimulatedGame;

    MCTS(Game*_Game){
      Value  = 0;
      Visits = 0;

      //HeadNode  = NULL;
      HeadNode  = new MCTS_Node(_Game);
      GivenGame = _Game;
    }

    ~MCTS(){
      delete HeadNode;
    }
    MCTS_Node* Algorithm(MCTS_Node* TransversedNode);
    double EvaluateTransversal(MCTS_Node* TransversedNode,Player* GivenPlayer);
    void Search(int Depth,Player* GivenPlayer);
    void ParallelSearch(int Depth);
    //MCTS_Node* Find_Highest_UCB1(std::list<MCTS_Node*>MCTS_List);
    void GetPossibleMoves();

    void CreateChildren();
    void TreeTraversal();
    void CreateNode();
    void RollOut();
};






// Provide implementation for the first method
void MCTS::GetPossibleMoves()
{
    std::list<GameMove*> Moves = SimulatedGame->PossibleMoves();
    std::list<Game*> Games = SimulatedGame->PossibleGames();
}


// Provide implementation for the first method
MCTS_Node* MCTS::Algorithm(MCTS_Node* TransversedNode)
{
//TransversedNode->Children.size()
//int Leaf =TransversedNode->Children.size();
//int Leaf = HeadNode->Children.size();
  if(0 == 0){
    //If Node is LeafNode
/*
if(TransversedNode->NodeVisits == 0){
  TransversedNode->RollOut();
}
std::list<Game*> Games = GivenGame->PossibleGames();
TransversedNode->AddChildren(Games);
MCTS_Node* NextNode = *TransversedNode->Children.begin();
return Algorithm(NextNode);*/
return NULL;

  }
  else{
/*
//Not Leaf Node, Transverse TreeSearch: Find Max Child UCB1 value.
MCTS_Node* MAXNode = TransversedNode->Find_MAX_UCB1_Child();
return Algorithm(MAXNode);*/
return NULL;
  }
}

// Provide implementation for the first method
double MCTS::EvaluateTransversal(MCTS_Node* TransversedNode,Player* GivenPlayer)
{
    Algorithm(HeadNode);
    return Value;
}


// Provide implementation for the first method
void MCTS::Search(int Depth,Player* GivenPlayer)
{
    for (int i = 0; i < Depth; i++) {
      EvaluateTransversal(HeadNode,GivenPlayer);
    }

    std::cout << "Searching Depth:" << Depth << "\n";

}

// Provide implementation for the first method
void MCTS::ParallelSearch(int Depth)
{
    std::cout << "Searching Depth:" << Depth << "\n";

}


#endif //MCTS_CU
