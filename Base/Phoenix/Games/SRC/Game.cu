/*
Anthony M Schroeder


Purpose:
Create a Game interface in order to implement and integrate different games
easily.

Using a method called: Pure Virtual Functions.

*/


#include <hip/hip_runtime.h>
#include <string>
#include <iostream>


class Game
{
private:

  public:
      Game(){}
      ~Game(){}

        //The following Methods use the 'Pure Virtual Function' method,
        //  where "= 0" part makes this method pure virtual,
        //  and also makes this class abstract.

        //** Not standardized ....
      virtual bool Move(int Row,int Col) = 0;
      //virtual void AvaliableMoves(int Depth) = 0;

      virtual std::string GenerateStringRepresentation() = 0;

      //virtual void DisplayInTerminal(int Depth) = 0;
      virtual void PlayAsHuman() = 0;

      //virtual void StepSimulation() = 0;
      //virtual void CopySimulation() = 0;
      //virtual void SaveSimulation() = 0;
      //virtual void ReadSimulation() = 0;
      //virtual void RollOut()        = 0;
};
