/*
Anthony M Schroeder


Purpose:
Create a TreeSearch & TreeSimulation interface in order to implement and
integrate different Search Methods easily.

Using a method called: Pure Virtual Functions.
(Link:https://www.tutorialspoint.com/pure-virtual-functions-and-abstract-classes-in-cplusplus)

*/


#include <hip/hip_runtime.h>
#include <string>
#include <iostream>

class TreeSimulation
{

  public:
      TreeSimulation(){}
      ~TreeSimulation(){}

        //The following Methods use the 'Pure Virtual Function' method,
        //  where "= 0" part makes this method pure virtual,
        //  and also makes this class abstract.
      virtual void Search(int Depth) = 0;
      //virtual void PruneTree() = 0;

      //virtual void StepSimulation() = 0;
      //virtual void CopySimulation() = 0;
      //virtual void SaveSimulation() = 0;
      //virtual void ReadSimulation() = 0;

};
