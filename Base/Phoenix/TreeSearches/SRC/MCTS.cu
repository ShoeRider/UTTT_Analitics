#include "hip/hip_runtime.h"
#ifndef MCTS_CU
#define MCTS_CU


#include <iostream>
#include <string>
#include <list>
#include <cmath>
#include <bits/stdc++.h>

#include "TreeSearch.cu"

#include "../../Games/SRC/Game.cu"
#include "../../Games/SRC/TTT.cu"
#include "../../Games/SRC/UTTT.cu"
class MCTS_Node
{
private:

public:
    int    NodeVisits;
    int    Depth;
    double ValueSum;
    double SimulationRep;
    bool LeafNode;
    Game* GivenGame;

    MCTS_Node*           Parent;
    std::list<MCTS_Node*> Children;
    MCTS_Node(){}
    MCTS_Node(Game* Instance){
      GivenGame = Instance;
    }

    ~MCTS_Node(){}
    double     Find_UCB1();
    MCTS_Node* Find_MAX_UCB1_Child();
    void       RollOut();
    int        AddChildren(std::list<Game*> PossibleMoves);

};

MCTS_Node* get(std::list<MCTS_Node*> _list, int _i){
    std::list<MCTS_Node*>::iterator it = _list.begin();
    for(int i=0; i<_i; i++){
        ++it;
    }
    return *it;
}


MCTS_Node* MCTS_Node::Find_MAX_UCB1_Child(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  MCTS_Node* HighestNode  = NULL;

  for (MCTS_Node* Node : Children){
      NodesValue = Node->Find_UCB1();
      if (HighestValue < NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}

//Preform MonteCarlo's UCB1 evaluation algorithm on a given node.
double MCTS_Node::Find_UCB1(){
  double ExploreBy = 1.4142;
  if(NodeVisits == 0)
	{
		return INT_MAX;
	}
  float _NodeVisits;
  if (Parent != NULL){
    _NodeVisits = (float)Parent->NodeVisits;
  }
  else{
    _NodeVisits = 0;
  }
  //Preform UCB1 Formula
  return (ValueSum/NodeVisits) + ExploreBy*sqrt(log(_NodeVisits/NodeVisits));
}



//For each element within a list of PossibleInstances(Different Game States)
//Add as different Childeren
int MCTS_Node::AddChildren(std::list<Game*> PossibleInstances){
  int ChildrenAdded = 0;
  MCTS_Node NewNode;
  for (Game* Instance : PossibleInstances){
      NewNode = MCTS_Node(Instance);
      NewNode.Parent = this;
      Children.push_back(&NewNode);
      ChildrenAdded++;
  }
  return ChildrenAdded;
}

void MCTS_Node::RollOut(){
  GivenGame->RollOut();
}




class MCTS : public TreeSimulation
{
public:
  double Value;
  double Visits;
  int Nodes;
  Game* GivenGame;

  //MCTS_Node* TransversedNode;
  MCTS_Node* HeadNode;
  std::list<MCTS_Node*>MCTS_List;
  Game* SimulatedGame;

    MCTS(Game*_Game){
      GivenGame = _Game;
    }
    ~MCTS(){}
    MCTS_Node* Algorithm(MCTS_Node* TransversedNode);
    double EvaluateTransversal(MCTS_Node* TransversedNode,Player* GivenPlayer);
    void Search(int Depth);
    void ParallelSearch(int Depth);
    //MCTS_Node* Find_Highest_UCB1(std::list<MCTS_Node*>MCTS_List);
    void GetPossibleMoves();

    void CreateChildren();
    void TreeTraversal();
    void CreateNode();
    void RollOut();
};






// Provide implementation for the first method
void MCTS::GetPossibleMoves()
{
    std::list<GameMove*> Moves = SimulatedGame->PossibleMoves();
    std::list<Game*> Games = SimulatedGame->PossibleGames();
}


// Provide implementation for the first method
MCTS_Node* MCTS::Algorithm(MCTS_Node* TransversedNode)
{

  if(TransversedNode->Children.size()==0){
    //If Node is LeafNode
    if(TransversedNode->NodeVisits == 0){
      TransversedNode->RollOut();
    }
    std::list<Game*> Games = GivenGame->PossibleGames();
    TransversedNode->AddChildren(Games);
    MCTS_Node* NextNode = *TransversedNode->Children.begin();
    return Algorithm(NextNode);

  }
  else{
    //Not Leaf Node, Transverse TreeSearch: Find Max Child UCB1 value.
    MCTS_Node* MAXNode = TransversedNode->Find_MAX_UCB1_Child();
    return Algorithm(MAXNode);
  }
}

// Provide implementation for the first method
double MCTS::EvaluateTransversal(MCTS_Node* TransversedNode,Player* GivenPlayer)
{
    Algorithm(TransversedNode);
    return Value;
}


// Provide implementation for the first method
void MCTS::Search(int Depth)
{

    std::cout << "Searching Depth:" << Depth << "\n";

}

// Provide implementation for the first method
void MCTS::ParallelSearch(int Depth)
{
    std::cout << "Searching Depth:" << Depth << "\n";

}


#endif //MCTS_CU
