#include "hip/hip_runtime.h"
#ifndef MMath_CU
#define MMath_CU

#include "Matrix.h"

void MMath_V()
{
  printf("Matrix Math \t\tV:2.00\n");
}
void MMath_D()
{
  printf("Matrix Math \t\tV:2.00\n");
}


#define ReLU(X)      ((x > 0) ? x : 0)
#define d_ReLU(X)    ((x > 0) ? 1 : 0)
#define LeakyReLU(X,_hA)   ((x > 0) ? x : (x * A))
#define d_LeakyReLU(X,_hA) ((x > 0) ? 1 : (A))



void int_MinMaxClip(int Clip,int* Value)
{
  if (*Value > Clip)
  {
    *Value = Clip;
  }
  else if(*Value < -Clip)
  {
    *Value = -Clip;
  }
}



void float_MinMaxClip(float Clip,float* Value)
{
  if (*Value > Clip)
  {
    *Value = Clip;
  }
  else if(*Value < -Clip)
  {
    *Value = -Clip;
  }
}

void FMatrix_t_MinMaxClip(float Clip,FMatrix_t* FMatrix)
{
  for (int i = 0; i < FMatrix->X; i++)
  {
    for (int j = 0; j < FMatrix->Y; j++)
    {
      float_MinMaxClip(Clip,(FMatrix->Array + i*FMatrix->Y + j));
    }
  }
}


IMatrix_t* Create_IMatrix_t(int x,int y)
{
  IMatrix_t* Matrix =(IMatrix_t*) malloc(sizeof(IMatrix_t));

  Matrix->Array = (int *)malloc(x * y * sizeof(int));
  Matrix->X = x;
  Matrix->Y = y;
  qSet_2D_Matrix_Elements(Matrix,0)

  return Matrix;
}

void Copy(IMatrix_t* IMatrix0,IMatrix_t* IMatrix1)
{
  for (int i = 0; i < IMatrix0->X; i++)
  {
    for (int j = 0; j < IMatrix0->Y; j++)
    {
      *(IMatrix1->Array + i*IMatrix1->Y + j) = *(IMatrix0->Array + i*IMatrix0->Y + j);
    }
  }
}


IMatrix_t* Copy(IMatrix_t* IMatrix0)
{
  IMatrix_t* IMatrix1 = Create_IMatrix_t(IMatrix0->Y,IMatrix0->Y);
  for (int i = 0; i < IMatrix0->X; i++)
  {
    for (int j = 0; j < IMatrix0->Y; j++)
    {
      *(IMatrix1->Array + i*IMatrix1->Y + j) = *(IMatrix0->Array + i*IMatrix0->Y + j);
    }
  }
  return IMatrix1;
}

int GetSum_IMatrix(IMatrix_t* IMatrix)
{
  register int Sum = 0;
  for (int i = 0; i < IMatrix->X; i++)
  {
    for (int j = 0; j < IMatrix->Y; j++)
    {
      Sum += *(IMatrix->Array + i*IMatrix->Y + j);
    }
  }
  return Sum;
}

bool Equivalent(IMatrix_t*Matrix0,IMatrix_t*Matrix1)
{
  if((Matrix0->X)   != (Matrix1->X) ||
     (Matrix0->Y)   != (Matrix1->Y)
   )
   {
     return false;
   }
   for(int i = 0; i < Matrix0->X ;i++)
   {
     for(int j = 0; j < Matrix0->Y ;j++)
     {
        if((*(Matrix0->Array+i*Matrix0->Y+j)) != (*(Matrix1->Array+i*Matrix1->Y+j)))
        {
          return false;
        }
     }
   }
   return true;
}

IMatrix_t* CreateIntegerIdentityMatrix(int x,int y)
{
  IMatrix_t* Matrix = (IMatrix_t*) malloc(sizeof(IMatrix_t));

  Matrix->Array = (int *)malloc(x * y * sizeof(int));
  Matrix->X = x;
  Matrix->Y = y;
  for (int i = 0; i < Matrix->X; i++)
  {
    for (int j = 0; j < Matrix->Y; j++)
    {
      if(i == j)
      {
        *(Matrix->Array + i*Matrix->Y + j) = 1;
      }
      else
      {
        *(Matrix->Array + i*Matrix->X + j) = 0;
      }

    }
  }
  return Matrix;
}

//&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&

FMatrix_t* Create_FMatrix_t(int x,int y)
{
  FMatrix_t* FMatrix = (FMatrix_t*)malloc(sizeof(FMatrix_t));

  FMatrix->Array = (float *)malloc(x * y * sizeof(float));
  FMatrix->X = x;
  FMatrix->Y = y;
  //float Zero = 0;
  //qSet_2D_Matrix_Elements(FMatrix,0)
  return FMatrix;
}

FMatrix_t* CreateZero_FMatrix_t(int x,int y)
{
  FMatrix_t* FMatrix = (FMatrix_t*)malloc(sizeof(FMatrix_t));

  FMatrix->Array = (float *)malloc(x * y * sizeof(float));
  FMatrix->X = x;
  FMatrix->Y = y;
  //float Zero = 0;
  qSet_2D_Matrix_Elements(FMatrix,0)
  return FMatrix;
}

FMatrix_t* CreateFloatMatrix(int x,int y)
{
  FMatrix_t* FMatrix = (FMatrix_t*)malloc(sizeof(FMatrix_t));

  FMatrix->Array = (float *)malloc(x * y * sizeof(float));
  FMatrix->X = x;
  FMatrix->Y = y;
  //float Zero = 0;
  qSet_2D_Matrix_Elements(FMatrix,0)
  return FMatrix;
}

FMatrix_t* CreatefloatIdentityMatrix(int x,int y)
{
  FMatrix_t* Matrix =(FMatrix_t*) malloc(sizeof(FMatrix_t));

  Matrix->Array = (float *)malloc(x * y * sizeof(float));
  Matrix->X = x;
  Matrix->Y = y;
  float One = 1;
  for (int i = 0; i < Matrix->X; i++)
  {
    for (int j = 0; j < Matrix->Y; j++)
    {
      if(i == j)
      {
        *(Matrix->Array + i*Matrix->Y + j) = One;
      }
      else
      {
        *(Matrix->Array + i*Matrix->Y + j) = 0;
      }

    }
  }
  return Matrix;
}

FMatrix_t* CreateIdentity_FMatrix_t(int x,int y)
{
  FMatrix_t* Matrix =(FMatrix_t*) malloc(sizeof(FMatrix_t));

  Matrix->Array = (float *)malloc(x * y * sizeof(float));
  Matrix->X = x;
  Matrix->Y = y;
  float One = 1;
  for (int i = 0; i < Matrix->X; i++)
  {
    for (int j = 0; j < Matrix->Y; j++)
    {
      if(i == j)
      {
        *(Matrix->Array + i*Matrix->Y + j) = One;
      }
      else
      {
        *(Matrix->Array + i*Matrix->Y + j) = 0;
      }

    }
  }
  return Matrix;
}

void ZeroFloatMatrix(FMatrix_t* FMatrix)
{
  qSet_2D_Matrix_Elements(FMatrix,0)
}

void Zero_FMatrix_t(FMatrix_t* FMatrix)
{
  qSet_2D_Matrix_Elements(FMatrix,0)
}


void CopyFloatMatrix(FMatrix_t* FMatrix0,FMatrix_t* FMatrix1)
{
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      *(FMatrix1->Array + i*FMatrix1->Y + j) = *(FMatrix0->Array + i*FMatrix0->Y + j);
    }
  }
}

FMatrix_t* MakeCopyFMatrix(FMatrix_t* Matrix0)
{
  FMatrix_t* Copy = CreateFloatMatrix(Matrix0->X,Matrix0->Y);
  for(int XAxis = 0; XAxis < Matrix0->X;XAxis++)
  {

    for(int YAxis = 0; YAxis < Matrix0->Y;YAxis++)
    {
      *(Copy->Array+(Copy->X*YAxis+(XAxis))) = *(Matrix0->Array+(Matrix0->X*YAxis+(XAxis)));
    }
  }
  return Copy;
}

float GetSum_FMatrix(FMatrix_t* FMatrix)
{
  register float Sum = 0;
  for (int i = 0; i < FMatrix->X; i++)
  {
    for (int j = 0; j < FMatrix->Y; j++)
    {
      Sum += *(FMatrix->Array + i*FMatrix->Y + j);
    }
  }
  return Sum;
}

void AddFloatMatrix(FMatrix_t* FMatrix0, FMatrix_t* FMatrix1, FMatrix_t* FMatrix2)
{
  for (int i = 0; i < FMatrix2->X; i++)
  {
    for (int j = 0; j < FMatrix2->Y; j++)
    {
      *(FMatrix2->Array + i*FMatrix2->Y + j) = *(FMatrix0->Array + i*FMatrix0->Y + j)+*(FMatrix1->Array + i*FMatrix1->Y + j);
    }
  }
}

void AddTo_FMatrix_t(FMatrix_t* FMatrix0, FMatrix_t* FMatrix1)
{
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      *(FMatrix0->Array + i*FMatrix0->Y + j) += *(FMatrix1->Array + i*FMatrix1->Y + j);
    }
  }
}

void SubTo_FMatrix_t(FMatrix_t* FMatrix0, FMatrix_t* FMatrix1)
{
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      *(FMatrix0->Array + i*FMatrix0->Y + j) -= *(FMatrix1->Array + i*FMatrix1->Y + j);
    }
  }
}

void Add_FMatrix_t_To_FMatrix_t(FMatrix_t* FMatrix0, FMatrix_t* FMatrix1)
{
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      *(FMatrix0->Array + i*FMatrix0->Y + j) += *(FMatrix1->Array + i*FMatrix1->Y + j);
    }
  }
}


//Tested and works
void FAbs_Sigmoid_FMatrix_t(FMatrix_t* PropigatedActivation)
{
  //PrintFloatMatrix(PropigatedActivation);
  //printf("Sigmoid_FMatrix_t recieved proper PropigatedActivation structure\n");

  //x*log(fabs(x)+1)/fabs(x);
  for(int XAxis = 0; XAxis < PropigatedActivation->X;XAxis++)
  {
    for(int YAxis = 0; YAxis < PropigatedActivation->Y;YAxis++)
    {
      (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
        =          (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
          /(1+fabs((*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))));
    }
  }
  //printf("exiting with PropigatedActivation structure\n");
  //PrintFloatMatrix(PropigatedActivation);
  //printf("exiting with Proper PropigatedActivation structure\n");
}




//Two activation methods for back propigation !
//Front end/ Back end DLL storage !
//Stack storage to implement! storage

//NN Layer Nultiplication
// use Assert -> Create To hide/ Produce Test code
void LayerMultiplication(FMatrix_t* WeightMatrix,FMatrix_t** Given_PropigatedActivation)
{
  FMatrix_t* PropigatedActivation = *Given_PropigatedActivation;
  FMatrix_t* NewPropigatedActivation = (FMatrix_t*)CreateZero_FMatrix_t((PropigatedActivation)->X,(PropigatedActivation)->Y);
  float Element0 = 0;
  float Element1 = 0;
  for(int YAxis = 0; YAxis < (PropigatedActivation)->Y; YAxis++)
  {
    for(int XAxis = 0; XAxis < (PropigatedActivation)->X; XAxis++)
    {

      //Loop Through Weight Matrix and Add To NewPropigatedActivation Matrix
      Element0 = *((PropigatedActivation)->Array+((PropigatedActivation)->X*YAxis)+(XAxis));
      //printf("Activation %f\n",Element0);

      for(int XSlide = 0; XSlide < (PropigatedActivation)->X;XSlide++)
      {
        for(int YSlide = 0; YSlide < (PropigatedActivation)->Y;YSlide++)
        {
          Element1 = *(WeightMatrix->Array+(WeightMatrix->X * (PropigatedActivation)->X*YAxis + (PropigatedActivation)->X*(PropigatedActivation)->Y*XAxis +(PropigatedActivation)->Y*XSlide + YSlide));
          float Result = Element0 * Element1;
          (*(NewPropigatedActivation->Array+NewPropigatedActivation->Y*XSlide+YSlide)) += Result;
          /*
          printf("==================================================================\n");
          printf("\tElement0 %f\n",Element0);
          printf("\tElement1 %f\n",Element1);
          printf("\tResult %f\n",(*(NewPropigatedActivation->Array+NewPropigatedActivation->Y*XSlide+YSlide)));
          printf("Result-> %d\n", NewPropigatedActivation->Y*XSlide+YSlide);
          printf("YAxis %d, XAxis %d, XSlide %d, YSlide %d, Position %d \n",YAxis,XAxis,XSlide,YSlide,(WeightMatrix->X*(*PropigatedActivation)->X*YAxis + (*PropigatedActivation)->X*(*PropigatedActivation)->Y*XAxis +(*PropigatedActivation)->Y*XSlide + YSlide));
          //(*(NewPropigatedActivation->Array+(WeightMatrix->X*(*PropigatedActivation)->X*YAxis + (*PropigatedActivation)->X*(*PropigatedActivation)->Y*XAxis +(*PropigatedActivation)->Y*XSlide + YSlide))) += Element0 * Element1;
          printf("==================================================================\n");
          */

        }
      }



    }
  }

  //printf("Propigated Matrix\n");
  //PrintFloatMatrix(NewPropigatedActivation);

  //PrintFloatMatrix(*Given_PropigatedActivation);
  Free((PropigatedActivation));
  *Given_PropigatedActivation = NewPropigatedActivation;

  //PrintFloatMatrix(*Given_PropigatedActivation);
  //printf("Exiting Layer Multiplication\n");
}




void NN_MaskedLayerMultiplication(FMatrix_t* WeightMatrix,FMatrix_t** Given_PropigatedActivation,FMatrix_t* DropOutMask)
{
  FMatrix_t* PropigatedActivation = *Given_PropigatedActivation;
  FMatrix_t* NewPropigatedActivation = (FMatrix_t*)CreateZero_FMatrix_t((PropigatedActivation)->X,(PropigatedActivation)->Y);
  float Element0 = 0;
  float Element1 = 0;
  for(int YAxis = 0; YAxis < (PropigatedActivation)->Y; YAxis++)
  {
    for(int XAxis = 0; XAxis < (PropigatedActivation)->X; XAxis++)
    {

      //Loop Through Weight Matrix and Add To NewPropigatedActivation Matrix
      Element0 = *((PropigatedActivation)->Array+((PropigatedActivation)->X*YAxis)+(XAxis));
      //printf("Activation %f\n",Element0);
      if(*((DropOutMask)->Array+((DropOutMask)->X*YAxis)+(XAxis)) == 1)
      {
        for(int XSlide = 0; XSlide < (PropigatedActivation)->X;XSlide++)
        {
          for(int YSlide = 0; YSlide < (PropigatedActivation)->Y;YSlide++)
          {
            Element1 = *(WeightMatrix->Array+(WeightMatrix->X * (PropigatedActivation)->X*YAxis + (PropigatedActivation)->X*(PropigatedActivation)->Y*XAxis +(PropigatedActivation)->Y*XSlide + YSlide));
            float Result = Element0 * Element1;
            (*(NewPropigatedActivation->Array+NewPropigatedActivation->Y*XSlide+YSlide)) += Result;
            /*
            printf("==================================================================\n");
            printf("\tElement0 %f\n",Element0);
            printf("\tElement1 %f\n",Element1);
            printf("\tResult %f\n",(*(NewPropigatedActivation->Array+NewPropigatedActivation->Y*XSlide+YSlide)));
            printf("Result-> %d\n", NewPropigatedActivation->Y*XSlide+YSlide);
            printf("YAxis %d, XAxis %d, XSlide %d, YSlide %d, Position %d \n",YAxis,XAxis,XSlide,YSlide,(WeightMatrix->X*(*PropigatedActivation)->X*YAxis + (*PropigatedActivation)->X*(*PropigatedActivation)->Y*XAxis +(*PropigatedActivation)->Y*XSlide + YSlide));
            //(*(NewPropigatedActivation->Array+(WeightMatrix->X*(*PropigatedActivation)->X*YAxis + (*PropigatedActivation)->X*(*PropigatedActivation)->Y*XAxis +(*PropigatedActivation)->Y*XSlide + YSlide))) += Element0 * Element1;
            printf("==================================================================\n");
            */

          }
        }
      }


    }
  }

  //printf("Propigated Matrix\n");
  //PrintFloatMatrix(NewPropigatedActivation);

  //PrintFloatMatrix(*Given_PropigatedActivation);
  Free((PropigatedActivation));
  *Given_PropigatedActivation = NewPropigatedActivation;

  //PrintFloatMatrix(*Given_PropigatedActivation);
  //printf("Exiting Layer Multiplication\n");
}


//Takes the Hypothesised value from a NN for a given input, and takes the
// Desired Output to find the RootMeanSquareError
//RMSE (RootMeanSquareError)
float Cost_RMSE(FMatrix_t* Activation,FMatrix_t* Desired)
{
  //TODO Make Definition for quick defining Methods
  //float Cost = 0;
  float Cost = 0;
  for (int i = 0; i < Activation->X; i++)
  {
    for (int j = 0; j < Activation->Y; j++)
    {
      //TotalChange
      Cost += ((*(Desired->Array + i*Desired->Y + j)) - (*(Activation->Array + i*Activation->Y + j)));
    }
  }
  return Cost/(2* Activation->X * Activation->Y);
}

//TODO Rename this
//Tested and works
void Sigmoid_FMatrix_t(FMatrix_t* PropigatedActivation)
{
  //PrintFloatMatrix(PropigatedActivation);
  //printf("Sigmoid_FMatrix_t recieved proper PropigatedActivation structure\n");

  //x*log(fabs(x)+1)/fabs(x);
  for(int XAxis = 0; XAxis < PropigatedActivation->X;XAxis++)
  {
    for(int YAxis = 0; YAxis < PropigatedActivation->Y;YAxis++)
    {
      (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
        =          (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
          /(1+fabs((*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))));
    }
  }
  //printf("exiting with PropigatedActivation structure\n");
  //PrintFloatMatrix(PropigatedActivation);
  //printf("exiting with Proper PropigatedActivation structure\n");
}

void MaskedSigmoid_FMatrix_t(FMatrix_t* PropigatedActivation,FMatrix_t* DropOutMask)
{
  //PrintFloatMatrix(PropigatedActivation);
  //printf("Sigmoid_FMatrix_t recieved proper PropigatedActivation structure\n");

  //x*log(fabs(x)+1)/fabs(x);
  for(int XAxis = 0; XAxis < PropigatedActivation->X;XAxis++)
  {
    for(int YAxis = 0; YAxis < PropigatedActivation->Y;YAxis++)
    {
      if((*(DropOutMask->Array+(DropOutMask->X*YAxis)+(XAxis))) == 1)
      {
        (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
          =          (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
            /(1+fabs((*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))));
      }
      else
      {
        (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis))) = 0 ;
      }

    }
  }
  //printf("exiting with PropigatedActivation structure\n");
  //PrintFloatMatrix(PropigatedActivation);
  //printf("exiting with Proper PropigatedActivation structure\n");
}

void Free(FMatrix_t* Matrix)
{
  if (Matrix != NULL)
  {
    if(Matrix->Array != NULL)
    {
      free(Matrix->Array);
    }

    free(Matrix);
  }
}
//void Free_DLL_FMatrix_t(DLL_Handle_t* DLL_Handle);
QDefineFree_DLL_GivenStruct(Free,FMatrix_t)
//&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&



//Produces a 'DropOut' Matrix for a described matrix
//Note should always have at least 1 selected element and 1 non selected element
//Rates still apply correctly
//Rate 0-1
FMatrix_t* RandomMask_FMatrixMatrix(int x,int y, float Rate)
{
  FMatrix_t* GeneratedMask = CreateFloatMatrix(x,y);
  //Select 2 Random elements ->
  //increment SumSlide, from Node to Node, and test when Sum Region is reached

  for (int i = 0; i < x; i++)
  {
    for (int j = 0; j < y; j++)
    {
      if ( RandomFloat(0,1) < Rate)
      {
        *(GeneratedMask->Array +y*i +j) = 1;
      }
      else
      {
        *(GeneratedMask->Array +y*i +j) = 0;
      }
    }
  }
  int chance = RandomInteger(0,x*y);
  //printf("%d,%d\n",chance%x,chance/x);
  *(GeneratedMask->Array +y*(chance%x) +(chance/x)) = 1;
  //Get Sum
  return GeneratedMask;
}





FMatrix_t* RandomSelect_FMatrixMatrix(FMatrix_t* FMatrix)
{
  //float Sum = GetSum_FMatrix(FMatrix);
  float SumSlide = 0;
  float RandomNumber = 0;//Generate RandomNumber from [0,Sum]
  FMatrix_t* SelectMatrix = CreateFloatMatrix(FMatrix->X,FMatrix->Y);
  //increment SumSlide, from Node to Node, and test when Sum Region is reached

  for (int i = 0; i < FMatrix->X; i++)
  {
    for (int j = 0; j < FMatrix->Y; j++)
    {
      SumSlide += *(FMatrix->Array + i*FMatrix->Y + j);
      if (SumSlide <= RandomNumber)
      {

        *(SelectMatrix->Array) = *(SelectMatrix->Array + i*FMatrix->Y + j);
      }
    }
  }
  //Get Sum
  return 0;
}


int RandomSelect_FMatrixIndex(FMatrix_t* FMatrix)
{
  float Sum = GetSum_FMatrix(FMatrix);
  int SumSlide = 0;
  int RandomNumber = RandomFloat(0,Sum);//Generate RandomNumber from [0,Sum]
  int SelectedNode = -1;
  //increment SumSlide, from Node to Node, and test when Sum Region is reached
  printf("Matrix(%d,%d)\n", FMatrix->X,FMatrix->Y);

  for (int i = 0; i < FMatrix->X; i++)
  {
    for (int j = 0; j < FMatrix->Y; j++)
    {
      SumSlide += *(FMatrix->Array + i*FMatrix->Y + j);
      printf("%d=>(%d,%d)",RandomNumber, i,j);printf("SS->%d\n",SumSlide);

      if (SumSlide > RandomNumber   &&
        *(FMatrix->Array + i*FMatrix->Y + j) != 0 )
      {
        SelectedNode = (i*FMatrix->Y+j);
        printf("selected %d\n",SelectedNode);
      }
    }
  }
  //Get Sum
  return SelectedNode;
}

int RandomSelect_IMatrixIndex(IMatrix_t* IMatrix)
{
  int Sum = GetSum_IMatrix(IMatrix);
  int SumSlide = 0;
  if (Sum == 0)
  {
    return -1;
  }
  int RandomNumber = RandomInteger(1,Sum);//Generate RandomNumber from [0,Sum]

  int SelectedNode = -1;
  //increment SumSlide, from Node to Node, and test when Sum Region is reached
  //printf("Matrix(%d,%d)\n", IMatrix->X,IMatrix->Y);

  for (int i = 0; i < IMatrix->X; i++)
  {
    for (int j = 0; j < IMatrix->Y; j++)
    {
      SumSlide += *(IMatrix->Array + i*IMatrix->Y + j);
      //printf("%d=>(%d,%d)",RandomNumber, i,j);printf("SS->%d\n",SumSlide);

      if (SumSlide == RandomNumber   &&
        *(IMatrix->Array + i*IMatrix->Y + j) != 0 )
      {
        SelectedNode = (i*IMatrix->Y+j);
        //printf("selected %d\n",SelectedNode);
      }
    }
  }
  //Print(IMatrix);
  //PrintInt(SelectedNode)
  //Get Sum
  return SelectedNode;
}





//Drop out Matrix
IMatrix_t* CreateI_DOM(int x,int y)
{


  IMatrix_t* Matrix = (IMatrix_t*)malloc(sizeof(IMatrix_t));
  Matrix->Array =(int*) malloc(sizeof(int)*x*y);
  Matrix->X = x;
  Matrix->Y = y;
  //initialized N by N matrix
  //now make it an Identity Matrix

  for(int XAxis = 0; XAxis < Matrix->X;XAxis++)
  {

    for(int YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      *(Matrix->Array+(Matrix->X*XAxis)+(YAxis)) = rand()%2;

    }
    //printf("%p",(Matrix->Array+(XAxis*Matrix->Y*sizeof(double))));
  }
  return Matrix;
}


IMatrix_t* CreateI_RW(int y,int x)
{
  srand(time(NULL));   // should only be called once

  IMatrix_t* Matrix = (IMatrix_t*)malloc(sizeof(IMatrix_t));
  Matrix->Array = (int*) malloc(sizeof(int)*x*y);
  Matrix->X = x;
  Matrix->Y = y;
  //initialized N by N matrix
  //now make it an Identity Matrix

  for(int XAxis = 0; XAxis < Matrix->X;XAxis++)
  {

    for(int YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      *(Matrix->Array+(Matrix->X*XAxis)+(YAxis)) = ((int)rand() / (int)RAND_MAX)*(2);
    }
    //printf("%p",(Matrix->Array+(XAxis*Matrix->Y*sizeof(double))));
  }
  return Matrix;
}

FMatrix_t* CreateF_RW(int y,int x)
{
  FMatrix_t* Matrix = (FMatrix_t*) malloc(sizeof(FMatrix_t));
  Matrix->Array = (float*)malloc(sizeof(float)*x*y);
  Matrix->X = x;
  Matrix->Y = y;
  //initialized N by N matrix
  //now make it an Identity Matrix

  for(int XAxis = 0; XAxis < Matrix->X;XAxis++)
  {

    for(int YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      *(Matrix->Array+(Matrix->X*XAxis)+(YAxis)) = ((float)rand() / (float)RAND_MAX)*(2);
    }
    //printf("%p",(Matrix->Array+(XAxis*Matrix->Y*sizeof(double))));
  }
  return Matrix;
}

void MultiplyFMatrices(FMatrix_t* Matrix0,FMatrix_t* Matrix1,FMatrix_t* Result)
{
  double Sum;
  float Element0 = 0;
  float Element1 = 0;
  for(int YAxis = 0; YAxis < Matrix0->Y; YAxis++)
  {
    for(int XAxis = 0; XAxis < Matrix1->X; XAxis++)
    {
    //  printf("%d,%d\n",XAxis,YAxis );
    //  printf("Value:%lf\n",*(Matrix0->Array+(Matrix0->X*XAxis*sizeof(double))+(YAxis*sizeof(double))));
      Sum = 0;
      for(int Slide = 0; Slide < Matrix0->X;Slide++)
      {
        Element0 = *(Matrix0->Array+(Matrix0->X*Slide)+(XAxis));
        Element1 = *(Matrix1->Array+(Matrix1->X*YAxis)+(Slide));
        //printf(" %f ",Element0);
        //printf(" %f ",Element1);
        //printf("=%f\n",Element0);
        Element0 *=  Element1;
        Sum += Element0;

      }

      (*(Result->Array+(Result->X*YAxis)+(XAxis))) = Sum;
    }
  }
}











void Print(IMatrix_t* Matrix)
{
  int XAxis, YAxis;
  printf("Print Matrix:\n");
  printf("-------------\n");
  printf("(%d,%d)\n",Matrix->X,Matrix->Y);
  for(XAxis = 0; XAxis < Matrix->X;XAxis++)
  {
    printf("{");

    for(YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      //printf("( %p,",(Matrix->Array+(Matrix->X*XAxis*sizeof(double))+(YAxis*sizeof(double))));
      //printf(" %p",*(Matrix->Array+(Matrix->X*XAxis*sizeof(double))+(YAxis*sizeof(double))));
      printf(" %d ",*(Matrix->Array+(Matrix->X*YAxis+(XAxis))));
    }
    //printf("%p",(Matrix->Array+(XAxis*Matrix->Y*sizeof(double))));
    printf(" }\n");
  }

}


void CopyMatrices(FMatrix_t* Matrix0,FMatrix_t* Final)
{
  for(int XAxis = 0; XAxis < Matrix0->X;XAxis++)
  {

    for(int YAxis = 0; YAxis < Matrix0->Y;YAxis++)
    {
      *(Final->Array+(Final->X*YAxis+(XAxis))) = *(Matrix0->Array+(Matrix0->X*YAxis+(XAxis)));
    }
  }
}



void Scale_FMatrix_t(float Scale,FMatrix_t* Matrix0)
{
  for(int XAxis = 0; XAxis < Matrix0->X;XAxis++)
  {
    for(int YAxis = 0; YAxis < Matrix0->Y;YAxis++)
    {
      *(Matrix0->Array+(Matrix0->X*YAxis+(XAxis))) *= Scale;
    }
  }
}

void PrintFloatMatrix(FMatrix_t* Matrix)
{
  if(Matrix != NULL)
  {
    int XAxis, YAxis;
    printf("Print Matrix:\n");
    printf("-------------\n");
    printf("(%d,%d)\n",Matrix->X,Matrix->Y);
    for(XAxis = 0; XAxis < Matrix->X;XAxis++)
    {
      printf("{");

      for(YAxis = 0; YAxis < Matrix->Y;YAxis++)
      {
        //printf("( %p,",(Matrix->Array+(Matrix->X*XAxis*sizeof(double))+(YAxis*sizeof(double))));
        //printf(" %p",*(Matrix->Array+(Matrix->X*XAxis*sizeof(double))+(YAxis*sizeof(double))));
        //printf(" %p ",(Matrix->Array+(Matrix->X*XAxis+(YAxis))));
        printf(" %f ",*(Matrix->Array+(Matrix->X*YAxis+(XAxis))));
      }
      //printf("%p",(Matrix->Array+(XAxis*Matrix->Y*sizeof(double))));
      printf(" }\n");
    }
  }
}



void Free(IMatrix_t* Matrix)
{
  free(Matrix->Array);
  free(Matrix);
}






/*
bool Read_IMatrix_Parser(char* FilePath,IMatrix_t* Matrix)
{
  //Check If File Exists
  if(!FileExists(FilePath))
  {
    return false;
  }

  //Create Line To store Data Read From File
  char Line[MAXCHAR];
  //Create LineSlide to Parse through lines, skiping over Blank space,
  //and Configuration Data Structure
  char *LineSlide;
  //Create A File Pointer from the given String
  FILE *FilePointer = fopen(FilePath,"r");
  //fflush(NULL);

  bool Gathering_MV = false;
  while(fgets(Line,MAXCHAR,FilePointer) != NULL)
  {

    //Standard Braket found
    if(SkipToken("<",Line,&LineSlide))
    {
      //Test for ManagedVariable(MV)
      if(SkipToken("MV",LineSlide,&LineSlide))
      {

        //"Name = 'Test'"
        if(SkipToken("Name",LineSlide,&LineSlide))
        {

          SkipGivenCharacters(LineSlide,&Line," \t");
          if(SkipToken("=",LineSlide,&LineSlide))
          {

            SkipGivenCharacters(LineSlide,&LineSlide," \t");
            //look for (")- Start String Varible
            if(SkipToken("\"",LineSlide,&LineSlide))
            {
              //Gather Variable

              SkipGivenCharacters(LineSlide,&LineSlide," \t");
              //look for (")- end String Varible
              if(SkipToken("\"",LineSlide,&LineSlide))
              {
Gathering_MV = true;
              }
            }


          }


        }

      }

    }


  }



  fclose(FilePointer);
}
*/



/*
//"Style = 'FMatrix'"
if(SkipToken("Style",Line,&LineSlide))
{
  //Start Matrix Aquire ProtoColl

}
*/


float MeanAbsError_FMatrix(FMatrix_t* FMatrix0,FMatrix_t* Desired)
{
  float Cost = 0;
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      Cost += abs(*(FMatrix0->Array + i*FMatrix0->Y + j)-*(Desired->Array + i*Desired->Y + j));
    }
  }
  return Cost/(FMatrix0->X * FMatrix0->Y);
}

//Takes the Hypothesised value from a NN for a given input, and takes the
// Desired Output to find the RootMeanSquareError
float RootMeanSquareError(FMatrix_t* FMatrix0,FMatrix_t* Desired)
{
  float Cost = 0;
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      Cost += pow((*(FMatrix0->Array + i*FMatrix0->Y + j)-*(Desired->Array + i*Desired->Y + j)),2);
    }
  }
  return Cost/(2* FMatrix0->X * FMatrix0->Y);
}



//Takes the Hypothesised value from a NN for a given input, and takes the
// Desired Output to find the SquaredError
float SquaredError_FMatrix(FMatrix_t* FMatrix0,FMatrix_t* Desired)
{
  float Cost = 0;
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      Cost += pow((*(FMatrix0->Array + i*FMatrix0->Y + j)-*(Desired->Array + i*Desired->Y + j)),2);
    }
  }
  return Cost/(2* FMatrix0->X * FMatrix0->Y);
}

//Testing

bool MakeCopyFMatrix_t_T()
{
  printf("Testing MakeCopyFMatrix_t_T()\n");
  FMatrix_t* Matrix0 = CreateF_RW(2,2);
  FMatrix_t* Matrix1 = MakeCopyFMatrix(Matrix0);

  PrintFloatMatrix(Matrix0);
  Free(Matrix0);

  PrintFloatMatrix(Matrix1);
  Free(Matrix1);

  return 1;
}


//Testing
bool Test_MultiplyMatrix()
{
  printf("Testing multiplyMatrices\n");
  FMatrix_t* Matrix0 = CreateF_RW(2,2);
  FMatrix_t* Matrix1 = CreateF_RW(2,2);
  FMatrix_t* Matrix2 = CreateF_RW(2,2);
  PrintFloatMatrix(Matrix0);
  PrintFloatMatrix(Matrix1);

  MultiplyFMatrices(Matrix0,Matrix1,Matrix2);

  PrintFloatMatrix(Matrix2);

  //PrintMatrix(SMatrix);
  //system("gnome-terminal -e \"rm -f Test.txt \"");
  //SaveMatrix("Test.txt",IMatrix);
  Free(Matrix0);
  Free(Matrix1);
  Free(Matrix2);
  return 1;
}

void RandomSelect_T()
{
  IMatrix_t* IMatrix0 = CreateIntegerIdentityMatrix(2,2);
  Print(IMatrix0);
  for(int x=0; x<100;x++)
  {
    int y = RandomSelect_IMatrixIndex(IMatrix0);
    printf("Selected Random int");
    PrintInt(y)
    PrintLines(2)
  }

  //Print(IMatrix1);

  Free(IMatrix0);
  //Free(IMatrix1);
}



void MMath_TT()
{
  int SelectedTest = 0;
  printf("0 = MakeCopyFMatrix_t_T()\n");
  printf("1 = \n");
  printf("2 = \n");
  printf("100 =\n");
  GatherTerminalInt("Please Select Test Move:",&SelectedTest);
  if (SelectedTest == 0)
  {
    MakeCopyFMatrix_t_T();
  }
  else if (SelectedTest == 1)
  {

  }
  else if (SelectedTest == 2)
  {

  }
  else if (SelectedTest == 3)
  {

  }
  else if (SelectedTest == 4)
  {

  }
  else if (SelectedTest == 100)
  {

  }
}

void MMath_T(int CallSign)
{
  printf("\n\n");
  printf("Starting MMath Tests:\n");
  printf("----------------------------\n");

  if (CallSign == 1)
  {
    //NNOperations_AT();
  }
  else
  {
    MMath_TT();
  }

  //NNOperations_OpenSave_T();

}

#endif // MMath_CU
