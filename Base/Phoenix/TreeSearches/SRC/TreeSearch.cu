/*
Anthony M Schroeder


Purpose:
Create a TreeSearch & TreeSimulation interface in order to implement and
integrate different Search Methods easily.

Using a method called: Pure Virtual Functions.
(Link:https://www.tutorialspoint.com/pure-virtual-functions-and-abstract-classes-in-cplusplus)

*/


#include <hip/hip_runtime.h>
#include <string>
#include <iostream>

class TreeSimulation
{
private:
  void* MLMethodPointer;
  public:
      TreeSimulation(){}
      ~TreeSimulation(){}

        //The following Methods use the 'Pure Virtual Function' method,
        //  where "= 0" part makes this method pure virtual,
        //  and also makes this class abstract.
      virtual void RollOut()          = 0;
      virtual void RollOut(int Depth) = 0;
      //virtual void PruneTree() = 0;

      virtual void StepSimulation() = 0;
      virtual void CopySimulation() = 0;
      virtual void SaveSimulation() = 0;
      virtual void ReadSimulation() = 0;

};


class SimulationTreeSearch
{
  private:
    void* MLMethodPointer;
    TreeSimulation* Simulation;
  public:
      SimulationTreeSearch(){}
      SimulationTreeSearch(TreeSimulation* Simulation){}
      ~SimulationTreeSearch(){}

        //The following Methods use the 'Pure Virtual Function' method,
        //  where "= 0" part makes this method pure virtual,
        //  and also makes this class abstract.
      virtual void Search(int Depth) = 0;
      //virtual void PruneTree() = 0;


      //Ideas to implement MCTS and ML algorithms
      virtual void Give_MLMethodPointer() = 0;
      //virtual void Aggregate_Search() = 0;
      //virtual void Aggregate_BP() = 0;
};
