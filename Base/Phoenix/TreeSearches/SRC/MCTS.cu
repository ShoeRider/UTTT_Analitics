#include "hip/hip_runtime.h"
#ifndef MCTS_CU
#define MCTS_CU


#include <iostream>
#include <string>
#include <list>
#include <cmath>
#include <bits/stdc++.h>

#include "TreeSearch.cu"

class MCTS_Node
{
private:
    int    NodeVisits;
    int    Depth;
    double ValueSum;
    double SimulationRep;
    bool LeafNode;

    MCTS_Node*           Parent;
    std::list<MCTS_Node> Children;
public:
    MCTS_Node(){}
    MCTS_Node(void* Instance){}

    ~MCTS_Node(){}
    double     Find_UCB1();
    MCTS_Node* Find_Highest_UCB1();
    int        AddChildren(std::list<void*> PossibleMoves);

};

//Preform MonteCarlo's UCB1 evaluation algorithm on a given node.
double MCTS_Node::Find_UCB1(){
  double ExploreBy = 1.4142;
  if(NodeVisits == 0)
	{
		return INT_MAX;
	}
  //Preform UCB1 Formula
  return (ValueSum/NodeVisits + ExploreBy*sqrt(log((float)Parent->NodeVisits)/NodeVisits));
}

MCTS_Node* MCTS_Node::Find_Highest_UCB1(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  MCTS_Node* HighestNode  = NULL;

  for (MCTS_Node Node : Children){
      NodesValue = Node.Find_UCB1();
      if (HighestValue < NodesValue)
      {
        HighestNode = &Node;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}

//For each element within a list of PossibleInstances(Different Game States)
//Add as different Childeren
int MCTS_Node::AddChildren(std::list<void*> PossibleInstances){
  int ChildrenAdded = 0;
  for (void* Instance : PossibleInstances){
      Children.push_back(MCTS_Node(Instance));
      ChildrenAdded++;
  }
  return ChildrenAdded;
}






class MCTS : public SimulationTreeSearch
{
private:
    int Nodes;
    MCTS_Node* TransversedNode;
    MCTS_Node* HeadNode;

public:
    MCTS(){}
    ~MCTS(){}
    void Search(int Depth);

    void Give_MLMethodPointer();
    void CreateChildren();
    void TreeTraversal();
    void CreateNode();
    void RollOut();
};

// Provide implementation for the first method
void MCTS::Search(int Depth)
{
    std::cout << "Searching Depth:" << Depth << "\n";

}

// Provide implementation for the first method
void MCTS::Give_MLMethodPointer()
{
    std::cout << "Hello World!";
}



#endif //MCTS_CU
