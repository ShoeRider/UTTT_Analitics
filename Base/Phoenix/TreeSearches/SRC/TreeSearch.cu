/*
Anthony M Schroeder


Purpose:
create a TreeSearch interface inorder to implement different types of tree
search methods.

Using a meathod called: Pure Virtual Functions.
*/


#include <hip/hip_runtime.h>
#include <string>
#include <iostream>


class TreeSearch
{
private:
  void* MLMethodPointer;
  public:
      TreeSearch(){}
      ~TreeSearch(){}

        //The following Methods use the 'Pure Virtual Function' method,
        //  where "= 0" part makes this method pure virtual,
        //  and also makes this class abstract.
      virtual void Search(int Depth) = 0;
      //virtual void PruneTree() = 0;


      //Ideas to implement MCTS and ML algorithms
      virtual void Give_MLMethodPointer() = 0;
      //virtual void Aggregate_Search() = 0;
      //virtual void Aggregate_BP() = 0;
};
