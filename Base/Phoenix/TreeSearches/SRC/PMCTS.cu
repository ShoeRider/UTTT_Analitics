#include "hip/hip_runtime.h"
/*
====================================================================================================
Description PMCTS(Parallel Monte Carlo Tree Search):
This takes the Monte-Carlo Tree search and adds some multithreading to Create Faster Searches.

Still uses the Game interface, and template structure from MCTS.

Requires:
  "TreeSearch.cu"
  "MCTS.cu"

Possibly requires:


====================================================================================================
Date:           NA
Script Version: 1.0
Name:           Anthony M Schroeder
Email:          as3379@nau.edu
Implemented a 'dispatch evenly' algorithm.
==========================================================
Date:           26 September 2021
Script Version: 1.1
Name:           Anthony M Schroeder
Email:          as3379@nau.edu
Implemented the following 'thread dispatch' algorithms:
- DispatchNaively
- DispatchEvenly
==========================================================
Date:           26 September 2021
Script Version: 1.2
Name:           Anthony M Schroeder
Email:          as3379@nau.edu
Refactored code for Recursive Thread Dispatch.
Implemented the following 'thread dispatch' algorithms:
- UCB1 PMCTS.
==========================================================
*/

#ifndef P_MCTS_CU
#define P_MCTS_CU



#include "PMCTS.h"









/*
MCTS_Node, is a structure within the MCTS which holds structural information,
as well as a game state representation.
Great step by step example found here: https://www.youtube.com/watch?v=UXW2yZndl7U

@Methods:

 * @param
    Game* Instance,

 *
 * @see MCTS::Find_MAX_UCB1_Child()
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
class PMCTS_Node
{
private:

public:
  double PastNodeVisits;
  double PastValueSum;
  //////////////////////////////////////////////////////////////////////////////
  // Values to evaluate UCB1 preformance.
  //////////////////////////////////////////////////////////////////////////////
  double NodeVisits;
  double ValueSum;
  Game_Tp* GivenGame = NULL;
  double UCB1Value;
  double SoftMAX;

  //////////////////////////////////////////////////////////////////////////////
  // List of _Players to maintain turn order.
  //////////////////////////////////////////////////////////////////////////////
  std::list<Player_Tp*> _Players;

  //////////////////////////////////////////////////////////////////////////////
  // pointers to maintain tree structure.
  //////////////////////////////////////////////////////////////////////////////
  PMCTS_Node*           Parent       = NULL;
  PMCTS_Node*           RollOutChild = NULL;
  std::list<PMCTS_Node<Game_Tp,Player_Tp>*> Children;


    //////////////////////////////////////////////////////////////////////////////
    // Initialization method.
    PMCTS_Node(Game_Tp* Instance,std::list<Player_Tp*> _GivenPlayers){
      for (Player_Tp* _Player : _GivenPlayers){
            //printf("adding Player:%p\n",(_Player));
            _Players.push_back(_Player);
      }
      GivenGame  = Instance;
      Children   = {};
      NodeVisits = 0;
      ValueSum   = 0.001;
      UCB1Value       = 0;
      SoftMAX    = 0;
      //printf("Creating MCTS Node w Player:%p\n",*(_Players.begin()));
      //std::cin.get();
    }


    ~PMCTS_Node(){
      for (PMCTS_Node<Game_Tp,Player_Tp>* Node : Children){
        delete Node;
      }
      if (RollOutChild != NULL)
      {
          delete RollOutChild;
      }
      delete GivenGame;
    }

    bool operator <(const PMCTS_Node<Game_Tp,Player_Tp> & Other_PMCTS_Node)
        {
          printf("This:%f\n", Find_UCB1());
          printf("That:%f\n", Other_PMCTS_Node->Find_UCB1());
          printf("-----------------------");
            return Find_UCB1() < Other_PMCTS_Node->Find_UCB1();
        }
/*    bool operator ==(const PMCTS_Node<Game_Tp,Player_Tp> & Other_PMCTS_Node)
        {
          printf("This:%f\n", Find_UCB1());
          printf("That:%f\n", Other_PMCTS_Node->Find_UCB1());
          printf("-----------------------");
            return Find_UCB1() == Other_PMCTS_Node->Find_UCB1();
        }
        */
  //////////////////////////////////////////////////////////////////////////////
  // Method Declarations.
  //////////////////////////////////////////////////////////////////////////////
  double     Find_UCB1();

  PMCTS_Node<Game_Tp,Player_Tp>* Find_MAX_UCB1_Child();
  PMCTS_Node<Game_Tp,Player_Tp>* ReturnBestMove();
  PMCTS_Node<Game_Tp,Player_Tp>* RollOut();
  int        AddChildren(std::list<Game_Tp*> PossibleMoves);
  void       BackPropagation(Player_Tp* GivenPlayer,PMCTS_Node<Game_Tp,Player_Tp>* HeadNode);
  void       BackPropagation(PMCTS_Node<Game_Tp,Player_Tp>* HeadNode);
  void       RefreshWeights();
  double     GetAverageValue();
  void       DisplayTree();
  void       DisplayTree(int Depth);
  std::size_t GetHash();
  void       DisplayStats();
  double Get_UCB1_ChildrenSum();
  double AssignSoftMAX();
  double Get_ChildrenValueSum();
};


//Preform MonteCarlo's UCB1 evaluation algorithm on a given node.
template <typename Game_Tp, typename Player_Tp>
double PMCTS_Node<Game_Tp,Player_Tp>::Find_UCB1(){
  double ExploreBy = 1.4142;
  if(NodeVisits == 0)
	{
		return INT_MAX;
	}
  float _NodeVisits;
  if (Parent != NULL){
    _NodeVisits = Parent->NodeVisits;
  }
  else{
    _NodeVisits = 0;
  }
  //Preform UCB1 Formula
  UCB1Value = (ValueSum/NodeVisits) + ExploreBy*sqrt(log(_NodeVisits/NodeVisits));

  return UCB1Value;
}



template <typename Game_Tp, typename Player_Tp>
PMCTS_Node<Game_Tp,Player_Tp>* PMCTS_Node<Game_Tp,Player_Tp>::Find_MAX_UCB1_Child(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  PMCTS_Node<Game_Tp,Player_Tp>* HighestNode  = NULL;

  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : Children){
      NodesValue = Node->Find_UCB1();

      if (HighestValue < NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}





//Preform MonteCarlo's UCB1 evaluation algorithm on a given node, and return
//the node with the highest UCB1 Value.
template <typename Game_Tp, typename Player_Tp>
PMCTS_Node<Game_Tp,Player_Tp>* PMCTS_Node<Game_Tp,Player_Tp>::ReturnBestMove(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  PMCTS_Node<Game_Tp,Player_Tp>* HighestNode  = NULL;

  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : Children){
      NodesValue = Node->GetAverageValue();

      if (HighestValue < NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}


template <typename Game_Tp, typename Player_Tp>
int PMCTS_Node<Game_Tp,Player_Tp>::AddChildren(std::list<Game_Tp*> PossibleInstances){
  int ChildrenAdded = 0;
  PMCTS_Node<Game_Tp,Player_Tp>* NewNode;

  //////////////////////////////////////////////////////////////////////////////
  // For each element within a list of PossibleInstances(Different Game States)
  // Add as different Childeren/Leaf Nodes
  for (Game_Tp* Instance : PossibleInstances){

      if(Instance != NULL)
      {

        //////////////////////////////////////////////////////////////////////////////
        // For Each Possible Game, Create New MCTS_Node<Game_Tp>, and add it to
        // children list.
        NewNode = new PMCTS_Node<Game_Tp,Player_Tp>(Instance,(Instance->Players));
        NewNode->Parent = this;
        Children.push_back(NewNode);
        ChildrenAdded++;
      }
  }
  return ChildrenAdded;
}


/*
Takes the Node itself, copies itself.
(This also copies the corresponding game state And performs Rollout on the new copy.)
Please note: also sets the copy node's parent as the given Node. (This is
for the BackPropagation step for attributing the Final game state's value back up the tree)
Afterward, it returns the new copy.

@param Nothing
@return pointer to Copied Rollout Node.

*/
template <typename Game_Tp, typename Player_Tp>
PMCTS_Node<Game_Tp,Player_Tp>* PMCTS_Node<Game_Tp,Player_Tp>::RollOut(){
  //GivenGame->TestForWinner();
  Game_Tp* RollOutGame = GivenGame->CopyGame();
  RollOutGame->RollOut();

  //printf("RO_WinningPlayer:%p\n",RollOutGame->WinningPlayer);
  //TODO Check if game is finished
  RollOutChild = new PMCTS_Node<Game_Tp,Player_Tp>(RollOutGame,_Players);
  RollOutChild->Parent = this;
  return RollOutChild;
}



/*
BackPropagation is the final step of the MCTS. It backtracks from a rollout leaf node,
 back up the tree. This attributes Values to each parent node based on the out
 come of the current branch, for each node it tests if the current Player is the winner of the transversal.
 A winning state for that player recieves +1, Losing -1, tie +0

@param (Player* GivenPlayer)The final winner from the rollout evaluation.
@return Nothing(void)

*/
template <typename Game_Tp, typename Player_Tp>
void PMCTS_Node<Game_Tp,Player_Tp>::BackPropagation(Player_Tp* WinningPlayer,PMCTS_Node<Game_Tp,Player_Tp>* HeadNode)
{
  NodeVisits++;


  //If no matching condition is found an apposing player won the RollOut game.
  double EvaluatedValue = -1;
  if(*(_Players.begin()) == WinningPlayer)
  {
    EvaluatedValue = 1;
  }
  else if(WinningPlayer == NULL)
  {
    EvaluatedValue = 0;
  }
  //std::cout << GivenGame->Generate_StringRepresentation();
  //printf("MCTS Node Player:%p\n",*(_Players.begin()));
  //printf("     GivenPlayer:%p\n",GivenPlayer);
  //printf("  EvaluatedValue:%f\n",EvaluatedValue);
  //printf("           Value:%f\n",ValueSum);
  //printf("          Visits:%f\n",NodeVisits);
  ValueSum += EvaluatedValue;
  //printf(" Parent:  %p\n",Parent);
  //printf(" HeadNode:%p\n",HeadNode);
  //If not the head Node, Keep transversing up the Search Tree.
  //<Game_Tp,Player_Tp>


  if (
    Parent != NULL  &&
    this   != HeadNode
  )
  {
    Parent->BackPropagation(WinningPlayer,HeadNode);
  }
}

template <typename Game_Tp, typename Player_Tp>
void PMCTS_Node<Game_Tp,Player_Tp>::RefreshWeights()
{
  NodeVisits = 0;
  ValueSum   = 0.001;
  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : Children){
    NodeVisits+=Node->NodeVisits;
    ValueSum+=Node->ValueSum;

  }
}



/*gets the average Value of a node.
 this is desired over the
O(1) vs O(1)

@param Nothing
@return pointer to Copied Rollout Node.

*/
template <typename Game_Tp, typename Player_Tp>
double PMCTS_Node<Game_Tp,Player_Tp>::GetAverageValue()
{
  return ValueSum/NodeVisits;
}

template <typename Game_Tp, typename Player_Tp>
std::size_t PMCTS_Node<Game_Tp,Player_Tp>::GetHash()
{
  std::hash<Game_Tp>* Hash = new std::hash<Game_Tp>;// = std::hash<TTT>(* _Game);
  std::size_t HashValue = Hash->Hash(GivenGame);
  delete Hash;
  return HashValue;
}


template <typename Game_Tp, typename Player_Tp>
void PMCTS_Node<Game_Tp,Player_Tp>::DisplayStats(){
  if(NodeVisits>0)
  {
    std::cout << "----------------------------------------\n";
    printf("\tNodeVisits:%f\n", NodeVisits);
    printf("\tValueSum:%f\n", ValueSum);
    printf("\tNode Ratio:%f\n", (ValueSum/NodeVisits));
    printf("\tUCB1:%f\n", Find_UCB1());
    printf("\tHash: %zu\n",GetHash());
    //std::cout << GivenGame->Generate_StringRepresentation();
  }

}


// insertion sort template function
// to sort array in ascending order
// n is the size of array
template <typename Game_Tp, typename Player_Tp>
std::list<PMCTS_Node<Game_Tp,Player_Tp>*> InsertionSort(std::list<PMCTS_Node<Game_Tp,Player_Tp>*> OldList)
{

  std::list<PMCTS_Node<Game_Tp,Player_Tp>*> NewList;
//  std::list<PMCTS_Node<Game_Tp,Player_Tp>*> HighestNode;
  typename std::list<PMCTS_Node<Game_Tp,Player_Tp>*>::iterator HighestNode;

  while(OldList.size() > 0){
    typename std::list<PMCTS_Node<Game_Tp,Player_Tp>*>::iterator List_iterator = OldList.begin();
    double newValue,highestValue =-DBL_MAX;

    //Remove Next element from the list.
    while ( List_iterator != OldList.end())
    {
        newValue = (*List_iterator)->Find_UCB1();
        if(newValue > highestValue){
          highestValue = newValue;
          HighestNode = List_iterator;
        }
        else
        {
        }
        ++List_iterator;
    }
    NewList.push_back(*HighestNode);
    OldList.erase(HighestNode);
  }

  return NewList;
}



template <typename Game_Tp, typename Player_Tp>
double PMCTS_Node<Game_Tp,Player_Tp>::Get_UCB1_ChildrenSum(){
  double UCB1_Sum = 0;
  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : Children){
    UCB1_Sum += Node->Find_UCB1();
    //printf("Node->SoftMAX:%f\n",Node->SoftMAX);
  }
  //printf("UCB1_Sum:%f\n",UCB1_Sum);
  return UCB1_Sum;
}


template <typename Game_Tp, typename Player_Tp>
double PMCTS_Node<Game_Tp,Player_Tp>::AssignSoftMAX(){
  double UCB1_Sum = Get_UCB1_ChildrenSum();
  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : Children){
    Node->SoftMAX = Node->UCB1/UCB1_Sum;
  }
  return UCB1_Sum;
}

template <typename Game_Tp, typename Player_Tp>
double PMCTS_Node<Game_Tp,Player_Tp>::Get_ChildrenValueSum(){
  double Sum = 0;
  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : Children){
    Sum += Node->ValueSum;
  }
  return Sum;
}



/*
DisplayTree(int Depth)
  DisplayTree is a recursive function that displays the tree's structure, allowing for further
  analysis of the tree search.

@param (int Depth)
@return Void

*/
template <typename Game_Tp, typename Player_Tp>
void PMCTS_Node<Game_Tp,Player_Tp>::DisplayTree(int Depth){
  printf("Calling Sort \n");
  Children = InsertionSort(Children);


  if (Children.size() > 0){
    for (PMCTS_Node* Child : Children) { // c++11 range-based for loop
         Child->DisplayStats();
      }
    if((Depth-1)>0){
      for (PMCTS_Node* Child : Children) { // c++11 range-based for loop
           Child->DisplayTree(Depth-1);
        }
      }
  }
}

/*
DisplayTree(int Depth)
  DisplayTree is a recursive function that displays the tree's structure, allowing for further
  analysis of the tree search.
  *Shows entire TreeSearch.

@param ()
@return Void

*/
template <typename Game_Tp, typename Player_Tp>
void PMCTS_Node<Game_Tp,Player_Tp>::DisplayTree(){
  // For each branch, display the game's statistics.
  //////////////////////////////////////////////////////////////////////////////
  for (PMCTS_Node* Child : Children) {
      Child->DisplayStats();
    }

  std::cout << "----------------------------------------\n";
  std::cout << GivenGame->Generate_StringRepresentation();
  for (PMCTS_Node* Child : Children) {
       Child->DisplayTree();
    }

}


















/*
JoinThreads.
 * @param
 *    Takes a std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>, and takes the
 *   Finished threads and joins them.
 * Also has a internal wait 100 miliseconds to prevent overutilization of resources.

 */
template <typename Game_Tp, typename Player_Tp>
std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*> _JoinFinishedThreads(std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>ThreadList)
{

/*
printf("/////////////////////////////////////////////////////////////////\n");
printf("Starting _JoinFinishedThreads\n");
printf("ThreadList.size():%lu\n",ThreadList.size());
printf("/////////////////////////////////////////////////////////////////\n");*/
  int ThreadsJoined = 0;

  while(ThreadsJoined <= 0){
      //printf("ThreadList.size():%d\n",ThreadList.size());
      typename std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>::iterator ThreadList_iterator = ThreadList.begin();
      while ( ThreadList_iterator != ThreadList.end())
      {
          if((*ThreadList_iterator)->Finished){
            pthread_join(((*ThreadList_iterator)->Thread), NULL);
            ThreadsJoined++;
            free((*ThreadList_iterator));
            ThreadList.erase(ThreadList_iterator++);
          }
          else
          {
              // move to next item
              ++ThreadList_iterator;
          }
      }
      std::this_thread::sleep_for(std::chrono::milliseconds(5));
    }

  return ThreadList;
}




template <typename Game_Tp, typename Player_Tp>
std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*> _JoinAllThreads(std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>ThreadList)
{
  while(ThreadList.size() != 0){
    ThreadList = _JoinFinishedThreads<Game_Tp,Player_Tp>(ThreadList);
  }
  return ThreadList;
}














//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
// MCTS algorithms
//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////



template <typename Game_Tp, typename Player_Tp>
PMCTS_Node<Game_Tp,Player_Tp>* MCTS_Algorithm(PMCTS_Node<Game_Tp,Player_Tp>*TransversedNode)
{
  /*
    Helper Function for MCTS::Search & EvaluateStep.
    Performs an itteration of the MCTS Algorithm on 'TransversedNode'
  */

  //////////////////////////////////////////////////////////////////////////////
  //If Node is LeafNode, create Children nodes, and select the first node for
  // rollout.
  //////////////////////////////////////////////////////////////////////////////
  if(TransversedNode->Children.size() == 0){

    //std::cout << "LeafNode Detected  :"   << TransversedNode << "\n";


    /////////////////////////////////////////////////////////////////
    // If Leaf Node has no visits, preform rollout.
    /////////////////////////////////////////////////////////////////
    if(TransversedNode->NodeVisits == 0){
      //std::cout << "About to rool out on:"   << TransversedNode << "\n";

      return TransversedNode->RollOut();
    }


    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
    //UTTT* UTTT_Game = static_cast<UTTT*>(TransversedNode->GivenGame);


    /////////////////////////////////////////////////////////////////
    // Find all possible games from branch.
    /////////////////////////////////////////////////////////////////
    std::list<Game_Tp*> Games = TransversedNode->GivenGame->PossibleGames();
    //std::cout << "Adding Children Size:" << Games.size() << "\n";


    /////////////////////////////////////////////////////////////////
    // verify future games have been found.
    /////////////////////////////////////////////////////////////////
    if (Games.size() == 0)
    {
      return TransversedNode;
    }

    /////////////////////////////////////////////////////////////////
    //Takes the new Games and add them to the tree.
    /////////////////////////////////////////////////////////////////
    //printf("TransversedNode->GivenGame->Players.begin():%p\n",*(TransversedNode->GivenGame->_Players.begin()));
    TransversedNode->AddChildren(Games);

    /////////////////////////////////////////////////////////////////
    //select the first posible node.
    /////////////////////////////////////////////////////////////////
    PMCTS_Node<Game_Tp,Player_Tp>* NextNode = *TransversedNode->Children.begin();

    /////////////////////////////////////////////////////////////////
    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    /////////////////////////////////////////////////////////////////
    return MCTS_Algorithm(NextNode);

  }
  //Otherwise, transverse the tree using the UCB1 formula, looking for an 'optimal' branch to evaluate.
  else{

    //Not Leaf Node, Transverse down the Tree: Find the branch with the MAX UCB1 value.
    PMCTS_Node<Game_Tp,Player_Tp>* MAXNode = TransversedNode->Find_MAX_UCB1_Child();

    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    return MCTS_Algorithm(MAXNode);
  }
}


/*
//////////////////////////////////////////////////////////////////////////////
// MCTS_Search
//////////////////////////////////////////////////////////////////////////////
template <typename Game_Tp, typename Player_Tp>
void MCTS_Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode, double ThreadDepth)
{
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* PMCTS_ThreadData = (PMCTS_ThreadData_t<Game_Tp,Player_Tp>*) malloc(sizeof(PMCTS_ThreadData_t<Game_Tp,Player_Tp>));
  //printf("PMCTS_ThreadData_t:%p\n",PMCTS_ThreadData);
  PMCTS_ThreadData->TransversedNode = TransversedNode;
  PMCTS_ThreadData->Depth           = ThreadDepth;
  PMCTS_ThreadData->Finished        = false;
  MCTS_Search_thread<Game_Tp,Player_Tp>(PMCTS_ThreadData);
  free(PMCTS_ThreadData);
}
*/
template <typename Game_Tp, typename Player_Tp>
void MCTS_Search(PMCTS_Node<Game_Tp,Player_Tp>* StartingNode,double Depth)
{

  //////////////////////////////////////////////////////////////////////////////
  // For each Itteration, preform the following steps.
  //////////////////////////////////////////////////////////////////////////////
  for (int i = 0; i < Depth; i++) {

    PMCTS_Node<Game_Tp,Player_Tp>*TransversedNode = StartingNode;
    //printf("Depth:%d\n", i);
    //printf("TransversedNode:%p\n", TransversedNode);
    //////////////////////////////////////////////////////////////////////////////
    // Preform Tree transversal, to build tree.
    //    This returns either a rollout node, or a node from the tree with a completed game(based on MCTS).
    TransversedNode = MCTS_Algorithm<Game_Tp,Player_Tp>(TransversedNode);

    //////////////////////////////////////////////////////////////////////////////
    // Preform BackPropagation, to assign weights.
    TransversedNode->BackPropagation(TransversedNode->GivenGame->TestForWinner(),StartingNode);
  }
}




//////////////////////////////////////////////////////////////////////////////
// MCTS_Search_thread has been implemented to call MCTS_Search as Thread.
//////////////////////////////////////////////////////////////////////////////
template <typename Game_Tp, typename Player_Tp>
void * MCTS_Search_thread(void* GivenPMCTS_ThreadData)
{
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* PMCTS_ThreadData = static_cast<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>(GivenPMCTS_ThreadData);

  MCTS_Search(PMCTS_ThreadData->TransversedNode,PMCTS_ThreadData->Depth);

  //////////////////////////////////////////////////////////////////////////////
  // Thread is finished, Set Flag for Thread Clean up.
  PMCTS_ThreadData->Finished = true;

  return 0;
}
















//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
// PMCTS/MCTS Dispatch algorithms
//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////




template <typename Game_Tp, typename Player_Tp>
PMCTS_ThreadData_t<Game_Tp,Player_Tp>* DispatchMCTS_SearchThread(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Threads, int ThreadDepth)
{
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* PMCTS_ThreadData = (PMCTS_ThreadData_t<Game_Tp,Player_Tp>*) malloc(sizeof(PMCTS_ThreadData_t<Game_Tp,Player_Tp>));
  //printf("PMCTS_ThreadData_t:%p\n",PMCTS_ThreadData);
  PMCTS_ThreadData->TransversedNode = TransversedNode;
  PMCTS_ThreadData->Depth           = ThreadDepth;
  PMCTS_ThreadData->Threads           = Threads;
  PMCTS_ThreadData->Finished        = false;

  pthread_create(&(PMCTS_ThreadData->Thread), NULL, MCTS_Search_thread<Game_Tp,Player_Tp>, PMCTS_ThreadData);
  return PMCTS_ThreadData;
}








template <typename Game_Tp, typename Player_Tp>
void MCTS_DispatchNaively(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int Depth)
{
  double ThreadDepth = (Depth/TransversedNode->Children.size())+1;

  PMCTS_ThreadData_t<Game_Tp,Player_Tp>*            PMCTS_ThreadData;
  std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*> ThreadList;

  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : TransversedNode->Children){
    ThreadList.push_back(
      DispatchMCTS_SearchThread<Game_Tp,Player_Tp>(Node, Threads, ThreadDepth)
    );
  }

  ThreadList = _JoinAllThreads<Game_Tp,Player_Tp>(ThreadList);
}


template <typename Game_Tp, typename Player_Tp>
void MCTS_DispatchEvenly(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int Depth)
{
  MCTS_Search(TransversedNode,2);

  double ThreadDepth = (Depth/TransversedNode->Children.size())+1;
  std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*> ThreadList;
  //////////////////////////////////////////////////////////////////////////////
  //For Each Branch within Game, Dispatch a new thread.
  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : TransversedNode->Children){

    //////////////////////////////////////////////////////////////////////////////
    //For Each Thread to dispatch, wait until there is an available thread to release.
    bool DispatchedForNode = true;
    while(DispatchedForNode)
    {

      //////////////////////////////////////////////////////////////////////////////
      //Dispatch Threads
      if (ThreadList.size() < Threads){
        //PMCTS_ThreadData = _DispatchThread<Game_Tp,Player_Tp>(Node, ThreadDepth);
        ThreadList.push_back(
          DispatchMCTS_SearchThread<Game_Tp,Player_Tp>(Node, Threads, ThreadDepth)
        );
        DispatchedForNode = false;
      }


      //////////////////////////////////////////////////////////////////////////////
      //Join Threads
      if (ThreadList.size() == Threads){
        ThreadList = _JoinFinishedThreads<Game_Tp,Player_Tp>(ThreadList);
      }

    }
  }
  ThreadList = _JoinAllThreads<Game_Tp,Player_Tp>(ThreadList);

}







//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
// PMCTS searches
//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////


/*
  Helper Function for MCTS::Search & EvaluateStep.
  Performs an itteration of the MCTS Algorithm on 'TransversedNode'
*/
/*
template <typename Game_Tp, typename Player_Tp>
PMCTS_Node<Game_Tp,Player_Tp>* PMCTS_Algorithm(PMCTS_Node<Game_Tp,Player_Tp>*TransversedNode)
{
  //////////////////////////////////////////////////////////////////////////////
  //If Node is LeafNode, create Children nodes, and select the first node for
  // rollout.
  //////////////////////////////////////////////////////////////////////////////
  if(TransversedNode->Children.size() == 0){

    //std::cout << "LeafNode Detected  :"   << TransversedNode << "\n";


    /////////////////////////////////////////////////////////////////
    // If Leaf Node has no visits, preform rollout.
    /////////////////////////////////////////////////////////////////
    if(TransversedNode->NodeVisits == 0){
      //std::cout << "About to rool out on:"   << TransversedNode << "\n";

      return TransversedNode->RollOut();
    }


    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
    //UTTT* UTTT_Game = static_cast<UTTT*>(TransversedNode->GivenGame);


    /////////////////////////////////////////////////////////////////
    // Find all possible games from branch.
    /////////////////////////////////////////////////////////////////
    std::list<Game_Tp*> Games = TransversedNode->GivenGame->PossibleGames();
    //std::cout << "Adding Children Size:" << Games.size() << "\n";


    /////////////////////////////////////////////////////////////////
    // verify future games have been found.
    /////////////////////////////////////////////////////////////////
    if (Games.size() == 0)
    {
      return TransversedNode;
    }

    /////////////////////////////////////////////////////////////////
    //Takes the new Games and add them to the tree.
    /////////////////////////////////////////////////////////////////
    //printf("TransversedNode->GivenGame->Players.begin():%p\n",*(TransversedNode->GivenGame->_Players.begin()));
    TransversedNode->AddChildren(Games);

    /////////////////////////////////////////////////////////////////
    //select the first posible node.
    /////////////////////////////////////////////////////////////////
    PMCTS_Node<Game_Tp,Player_Tp>* NextNode = *TransversedNode->Children.begin();

    /////////////////////////////////////////////////////////////////
    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    /////////////////////////////////////////////////////////////////
    return MCTS_Algorithm(NextNode);

  }
  //Otherwise, transverse the tree using the UCB1 formula, looking for an 'optimal' branch to evaluate.
  else{

    //Not Leaf Node, Transverse down the Tree: Find the branch with the MAX UCB1 value.
    PMCTS_Node<Game_Tp,Player_Tp>* MAXNode = TransversedNode->Find_MAX_UCB1_Child();

    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    return MCTS_Algorithm(MAXNode);
  }
}
*/



/*

  //////////////////////////////////////////////////////////////////////////////
  // For each Itteration, preform the following steps.
  //////////////////////////////////////////////////////////////////////////////
  for (int i = 0; i < PMCTS_ThreadData->Depth; i++) {
    PMCTS_Node<Game_Tp,Player_Tp>*TransversedNode = PMCTS_ThreadData->TransversedNode;

    //////////////////////////////////////////////////////////////////////////////
    // Preform Tree transversal, to build tree.
    TransversedNode = PMCTS_Algorithm<Game_Tp,Player_Tp>(TransversedNode);

    //////////////////////////////////////////////////////////////////////////////
    // Preform BackPropagation, to assign weights.
    TransversedNode->BackPropagation(TransversedNode->GivenGame->TestForWinner(),PMCTS_ThreadData->TransversedNode);

  }
  */



//////////////////////////////////////////////////////////////////////////////
// PMCTS_Search has been implemented for parallelism.
//////////////////////////////////////////////////////////////////////////////

template <typename Game_Tp, typename Player_Tp>
double MCTS_FindPriorityByUCB1(PMCTS_Node<Game_Tp,Player_Tp>*Node){
  double UCB1Value = Node->Find_UCB1();
  //printf("UCB1:%lf\n", UCB1Value);
  return sqrt(UCB1Value);
}


template <typename Game_Tp, typename Player_Tp>
double MCTS_FindSUMPriorityByUCB1(PMCTS_Node<Game_Tp,Player_Tp>*TransversedNode){
  double Sum = 0;
  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : TransversedNode->Children){
    Sum += MCTS_FindPriorityByUCB1(Node);
    //Node->DisplayStats();
    //printf("MCTS_FindPriorityByUCB1:%lf\n", MCTS_FindPriorityByUCB1(Node));
    //printf("sum:%lf\n", Sum);
  }
  //printf("sum:%lf\n", Sum);
  return Sum;
}



template <typename Game_Tp, typename Player_Tp>
double MCTS_FindPriorityBySumValue(PMCTS_Node<Game_Tp,Player_Tp>*Node){
  double UCB1Value = Node->ValueSum;

  //printf("UCB1:%lf\n", UCB1Value);
  return std::max(UCB1Value,(double).001);
}


template <typename Game_Tp, typename Player_Tp>
double MCTS_FindSUMPriorityBySumValue(PMCTS_Node<Game_Tp,Player_Tp>*TransversedNode){
  double Sum = 0;
  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : TransversedNode->Children){
    Sum += MCTS_FindPriorityBySumValue(Node);
    //Node->DisplayStats();
    //printf("MCTS_FindPriorityBySumValue:%f\n", MCTS_FindPriorityBySumValue(Node));
    //printf("sum:%f\n", Sum);
  }
  //printf("sum:%lf\n", Sum);
  return Sum;
}





//MCTS_UCB1Threads *UCB1Threads = new MCTS_UCB1Threads();
template <typename Game_Tp, typename Player_Tp>
class MCTS_UCB1Threads
{
public:
  //////////////////////////////////////////////////////////////////////////////
  // Thread Serach Data
  //////////////////////////////////////////////////////////////////////////////
  double MaxThreads;
  double ThreadsDispatched;
  std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*> ThreadList;


  //double MaxThreads;
  double SearchDepth;

  double DepthThreadRatio;

  double MinimumDistribution;
  double UBC1Distribution;
  double ValueSumDistribution;

  double MinimumDepth;
  double UBC1_Depth;
  double ValueSum_Depth;

  int Branches;
  double ValueSum_PrioritySum;
  double UBC1_PrioritySum;
  //////////////////////////////////////////////////////////////////////////////
  // Initialization method.
  MCTS_UCB1Threads(PMCTS_Node<Game_Tp,Player_Tp>*TransversedNode,double GivenThreads, double GivenDepth){

    UBC1Distribution      = .375;
    ValueSumDistribution  = .375;
    MinimumDistribution   = .25;


    ThreadsDispatched = 0;
    MaxThreads        = GivenThreads;
    SearchDepth       = GivenDepth;

    Branches              = TransversedNode->Children.size();
    DepthThreadRatio      = SearchDepth/MaxThreads;
    MinimumDepth          = (MinimumDistribution*SearchDepth)/Branches;

    ValueSum_PrioritySum   = std::max(MCTS_FindSUMPriorityBySumValue(TransversedNode),(double)0);
    if(ValueSum_PrioritySum == 0){
      UBC1Distribution += ValueSumDistribution;
    }
    ValueSum_Depth         = (ValueSumDistribution*SearchDepth);


    UBC1_PrioritySum      = std::max(MCTS_FindSUMPriorityByUCB1(TransversedNode),(double)1);
    UBC1_Depth             = (UBC1Distribution*SearchDepth);




    //Pause;
  }
  ~MCTS_UCB1Threads(){
    _JoinAllThreads<Game_Tp,Player_Tp>(ThreadList);

  }

  void JoinFinishedThreads(std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>ThreadList)
  {

    //printf("/////////////////////////////////////////////////////////////////\n");
    //printf("Starting _JoinFinishedThreads\n");
    //printf("ThreadList.size():%d\n",ThreadList.size());
    //printf("/////////////////////////////////////////////////////////////////\n");
    int ThreadsJoined = 0;

    while(ThreadsJoined <= 0){

        //printf("ThreadList.size():%d\n",ThreadList.size());
        typename std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>::iterator ThreadList_iterator = ThreadList.begin();
        while ( ThreadList_iterator != ThreadList.end())
        {
            if((*ThreadList_iterator)->Finished){
              pthread_join(((*ThreadList_iterator)->Thread), NULL);
              ThreadsJoined++;
              ThreadsDispatched -= ((*ThreadList_iterator)->Threads);
              free((*ThreadList_iterator));
              ThreadList.erase(ThreadList_iterator++);
            }
            else
            {
                // move to next item
                ++ThreadList_iterator;
            }
        }
        std::this_thread::sleep_for(std::chrono::milliseconds(5));
      }
  }

  void Dispatch(PMCTS_Node<Game_Tp,Player_Tp>*Node){

    double UCB1Value            = MCTS_FindPriorityByUCB1(Node);
    double ValueSum             = MCTS_FindPriorityBySumValue(Node);

    double Depth_By_UCB1       = std::max((UCB1Value / UBC1_PrioritySum)     * UBC1_Depth,(double)0);
    double Depth_By_ValueSum   = std::max((ValueSum  / ValueSum_PrioritySum) * ValueSum_Depth,(double)0);
    double BranchDepth         = (Depth_By_ValueSum + Depth_By_UCB1 + ceil(MinimumDepth));

    double BranchThreads       = std::min(ceil(BranchDepth/DepthThreadRatio),MaxThreads);
    //BranchDepth = MinimumDepth;


printf("/////////////////////////////////////////////////////////////////\n");
printf("Branches:%d\n",Branches);
printf("MinimumDepth:%lf\n",MinimumDepth);
printf("UBC1_PrioritySum:%f\n",UBC1_PrioritySum);
printf("ValueSum_PrioritySum:%f\n",ValueSum_PrioritySum);
printf("-----------------------------------------------------------------\n");
printf("ValueSum:                            %f\n",ValueSum);
printf("ValueSum_PrioritySum:                %f\n",ValueSum_PrioritySum);
printf("(ValueSum  / ValueSum_PrioritySum):  %f\n",(ValueSum  / ValueSum_PrioritySum));
printf("ValueSum_Depth:                      %f\n",ValueSum_Depth);
printf("Depth_By_ValueSum:                   %f\n",Depth_By_ValueSum);
printf("-----------------------------------------------------------------\n");
printf("UCB1Value:                           %f\n",UCB1Value);
printf("UBC1_PrioritySum:                    %f\n",UBC1_PrioritySum);
printf("(UCB1Value / UBC1_PrioritySum):      %f\n",(UCB1Value / UBC1_PrioritySum));
printf("UBC1_Depth:                          %f\n",UBC1_Depth);
printf("Depth_By_UCB1:                       %f\n",Depth_By_UCB1);
printf("-----------------------------------------------------------------\n");
printf("MinimumDepth:%f\n",MinimumDepth);
printf("BranchDepth:%f\n",BranchDepth);
printf("-----------------------------------------------------------------\n");
printf("SearchDepth:%f\n",SearchDepth);
printf("MaxThreads:%f\n",MaxThreads);
printf("DepthThreadRatio:%f\n",DepthThreadRatio);
printf("BranchThreads:%lf\n",BranchThreads);
printf("/////////////////////////////////////////////////////////////////\n");
//Pause;



//printf("BranchThreads:%f\n",BranchThreads);
//printf("BranchDepth:%f\n",BranchDepth);

    //double asdf= BranchDepth/DepthToThreadRatio;
    //printf("BranchDepth:%f\n",BranchDepth);
    //printf("DepthToThreadRatio:%f\n",DepthToThreadRatio);
    //printf("asdf:%f\n",asdf);
    //BranchThreads = std::min(asdf,(double)1);
    //BranchThreads = 1;
    //printf("BranchThreads:%f\n",BranchThreads);


    //////////////////////////////////////////////////////////////////////////////
    //For Each Thread to dispatch, wait until there is an available thread to release.
    bool DispatchingForNode = true;
    while(DispatchingForNode)
    {

      //////////////////////////////////////////////////////////////////////////////
      //Dispatch Threads
      if (ThreadsDispatched < (MaxThreads+BranchThreads)){
        //PMCTS_ThreadData = _DispatchThread<Game_Tp,Player_Tp>(Node, ThreadDepth);
        ThreadList.push_back(
          Dispatch_MCTS_UCB1PrioritySearch_Thread<Game_Tp,Player_Tp>(Node, BranchThreads, BranchDepth)
        );
        DispatchingForNode = false;
      }
      else{
        //////////////////////////////////////////////////////////////////////////////
        //Join Threads
        ThreadList = _JoinFinishedThreads<Game_Tp,Player_Tp>(ThreadList);
      }

    }
  }
};





template <typename Game_Tp, typename Player_Tp>
void MCTS_UCB1PriorityAssignment(PMCTS_Node<Game_Tp,Player_Tp>*TransversedNode,double Threads, double Depth){

  MCTS_UCB1Threads<Game_Tp, Player_Tp> *UCB1Threads = new MCTS_UCB1Threads<Game_Tp, Player_Tp>(TransversedNode,Threads,Depth);

  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : TransversedNode->Children){
    //UCB1Threads->Dispatch(Node);
    UCB1Threads->Dispatch(Node);
/*
printf("UCB1_Sum:%f\n",Sum);
printf("Threads:%f\n",Threads);
printf("ThreadList.size() :%d\n",ThreadList.size() );
printf("UCB1_SumToThreadRatio:%f\n",SumToThreadRatio);
printf("ThreadsAssigned:%f\n",ThreadsAssigned);
printf("ThreadDepth:%f\n",ThreadDepth);
printf("/////////////////////////////////////////////////////////////////\n");
    //Pause;
    */


  }

  delete UCB1Threads;

}




template <typename Game_Tp, typename Player_Tp>
void MCTS_UCB1PrioritySearch(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Threads, double Depth)
{
  //TODO: add Segment Dispatch Logic to both:
  // -DispatchByPigeonHole
  // -DispatchByRotation

  double ThreshHold = 50000;
  //printf("TransversedNode->NodeVisits:%f\n",TransversedNode->NodeVisits);
  //Pause;
  /////////////////////////////////////////////////////////////////
  // Determine how to dispatch Threads.
  /////////////////////////////////////////////////////////////////
  if (TransversedNode->NodeVisits > ThreshHold){
  //if (true){
    /////////////////////////////////////////////////////////////////
    // Dispatch by recursive MCTS_UCB1PrioritySearch.
    /////////////////////////////////////////////////////////////////
    MCTS_UCB1PriorityAssignment<Game_Tp,Player_Tp>(TransversedNode,Threads,Depth);

    //PMCTS_DispatchByPigeonHole(TransversedNode,Threads,Depth);

  }
  else{
    /////////////////////////////////////////////////////////////////
    // Dispatch by MCTS_DispatchEvenly,
    /////////////////////////////////////////////////////////////////
    //printf("calling DispatchEvenly\n");
    MCTS_DispatchEvenly(TransversedNode,Threads,Depth);

  }
  TransversedNode->RefreshWeights();
/*
for (PMCTS_Node<Game_Tp,Player_Tp>* Node : TransversedNode->Children){
  TransversedNode->DisplayStats();
  Node->DisplayStats();
}
*/
}


template <typename Game_Tp, typename Player_Tp>
void* MCTS_UCB1PrioritySearch_Thread(void* GivenPMCTS_ThreadData)
{
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* PMCTS_ThreadData = static_cast<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>(GivenPMCTS_ThreadData);

  MCTS_UCB1PrioritySearch<Game_Tp,Player_Tp>(PMCTS_ThreadData->TransversedNode,PMCTS_ThreadData->Threads,PMCTS_ThreadData->Depth);
  //////////////////////////////////////////////////////////////////////////////
  // Thread is finished, Set Flag for Thread Clean up.
  PMCTS_ThreadData->Finished = true;
  return 0;
}


template <typename Game_Tp, typename Player_Tp>
PMCTS_ThreadData_t<Game_Tp,Player_Tp>* Dispatch_MCTS_UCB1PrioritySearch_Thread(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Threads, int ThreadDepth)
{
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* PMCTS_ThreadData = (PMCTS_ThreadData_t<Game_Tp,Player_Tp>*) malloc(sizeof(PMCTS_ThreadData_t<Game_Tp,Player_Tp>));
  //printf("PMCTS_ThreadData_t:%p\n",PMCTS_ThreadData);
  PMCTS_ThreadData->TransversedNode = TransversedNode;
  PMCTS_ThreadData->Depth           = ThreadDepth;
  PMCTS_ThreadData->Threads         = Threads;
  PMCTS_ThreadData->Finished        = false;

  pthread_create(&(PMCTS_ThreadData->Thread), NULL, MCTS_UCB1PrioritySearch_Thread<Game_Tp,Player_Tp>, PMCTS_ThreadData);
  return PMCTS_ThreadData;
}




template <typename Game_Tp, typename Player_Tp>
void MCTS_UCB1Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Threads, double Depth)
{
  //printf("(Depth/10):%f\n",(Depth/10));
  //Pause;
  //double Itterations = ceil(Depth/100000);
  //printf("Itterations:%f\n",Itterations);
  for(int i=0;i<10;i++){
      MCTS_UCB1PrioritySearch(TransversedNode,Threads,(Depth/10));
      //printf("Itterations:%d\n",(i*100000));
  }

}
















/*
MCTS is a tree search that takes a complete view of a game and evaluates the
most optimal moves for both players through a UCB1 algorithm.
This algorithm performs a hybrid of breath and depth search to evenly search a given search space.

Great step by step example found here: https://www.youtube.com/watch?v=UXW2yZndl7U

@Methods:
Search()
Algorithm():: A recursive implementation of the MCTS algorithm. Recursively creates a serach tree based on the MCTS, searching for the most optimal move.

 * @param
    Game*_Game,
    std::list<Player*> _GivenPlayers)

 *
 * @see MCTS_Node::Find_MAX_UCB1_Child()
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
class PMCTS: public TreeSimulation
{
  public:

  //////////////////////////////////////////////////////////////////////////////
  //Thread Information
  double Depth;
  double Threads;
  ParallelControlBlock* ParallelCB;


  //////////////////////////////////////////////////////////////////////////////
  // The current head node.
  //////////////////////////////////////////////////////////////////////////////
  Game_Tp* GivenGame;
  //MCTS_Node* TransversedNode;
  PMCTS_Node<Game_Tp,Player_Tp>* HeadNode;
  Game_Tp* SimulatedGame;


  //////////////////////////////////////////////////////////////////////////////
  // The current head node.
  //////////////////////////////////////////////////////////////////////////////
  std::list<Player_Tp*> Players;
  Player_Tp* GivenPlayer;


  //////////////////////////////////////////////////////////////////////////////
  // Initialization method.
  PMCTS(Game_Tp*_Game,std::list<Player_Tp*> _GivenPlayers){
    Players        = _GivenPlayers;
    GivenPlayer    = *(_GivenPlayers.begin());

    HeadNode  = new PMCTS_Node<Game_Tp,Player_Tp>(_Game,_GivenPlayers);
    GivenGame = _Game;
  }

  //////////////////////////////////////////////////////////////////////////////
  //
  //////////////////////////////////////////////////////////////////////////////
  virtual ~PMCTS(){
    delete HeadNode;
    //delete ParallelCB;
  }

  //////////////////////////////////////////////////////////////////////////////
  // Parallel Functions
  //////////////////////////////////////////////////////////////////////////////
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* DispatchThread(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int ThreadDepth);
  //void DispatchThreads(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int ThreadDepth);
  //void DispatchByPigeonHole(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int ThreadDepth);
  //void DispatchNaively(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int ThreadDepth);
  //void DispatchEvenly(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int ThreadDepth);

  //////////////////////////////////////////////////////////////////////////////
  // 'Single' Threaded Algorithms
  //////////////////////////////////////////////////////////////////////////////
  //PMCTS_Node<Game_Tp,Player_Tp>* PMCTS_Algorithm(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode);
  //TODO Include PMCTS Back Propagation
  //PMCTS_Node<Game_Tp,Player_Tp>* MCTS_Algorithm(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode);
  //TODO Include MCTS Back Propagation


  //////////////////////////////////////////////////////////////////////////////
  // Management Functions
  //////////////////////////////////////////////////////////////////////////////
  void PerformStep(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,Player_Tp* GivenPlayer);
  //double BackPropagation(MCTS_Node* TransversedNode,double GivenPlayer);
  //void PMCTS_Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Depth); //,Player* GivenPlayer
  //void MCTS_Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Depth); //,Player* GivenPlayer
  PMCTS* PruneSearch(PMCTS_Node<Game_Tp,Player_Tp>*SelectedNode);
  void ParallelSearch(int Depth);

  void Search(double Threads, double Depth);
  void Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Threads, double Depth);


  void Node_BackPropagation(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,Player_Tp* GivenPlayer);
  //MCTS* CreateBookMoves();
  //MCTS* SaveBookMoves(char* Path);
  //MCTS* OpenBookMoves(char* Path);
  //MCTS_Node* Find_Highest_UCB1(std::list<MCTS_Node*>MCTS_List);

};


















/**
 * Preforms the Monte Carlo tree search on the game used to initialize the MCTS
 *  Object.
 *
 *
 * @param <int> Depth(Depth of search tree).
 *
 * @return Void, modifies the given MCTS object, adding MCTS_Node elements to
 *   the Head node.
 *
 * @see MCTS
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Threads, double Depth)
{
  MCTS_Search(TransversedNode,37);
  //PMCTS_DispatchThreads(TransversedNode, Threads, Depth);
  //PMCTS_Search(TransversedNode,Threads, Depth);


  //MCTS_DispatchEvenly(TransversedNode,5,10);

  //MCTS_UCB1PrioritySearch(TransversedNode,Threads,Depth);

  MCTS_UCB1Search(TransversedNode,Threads,Depth);
}



template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::Search(double Threads, double Depth)
{
  Search(HeadNode, Threads, Depth);
  HeadNode->DisplayTree(2);
  HeadNode->DisplayStats();
}




#endif //P_MCTS_CU
