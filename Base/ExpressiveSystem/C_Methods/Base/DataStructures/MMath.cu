#include "hip/hip_runtime.h"
#ifndef MMath_CU
#define MMath_CU

#include "MMath.h"

void MMath_V()
{
  printf("Matrix Math \t\tV:2.00\n");
}
void MMath_D()
{
  printf("Matrix Math \t\tV:2.00\n");
}


#define ReLU(X)      ((x > 0) ? x : 0)
#define d_ReLU(X)    ((x > 0) ? 1 : 0)
#define LeakyReLU(X,_hA)   ((x > 0) ? x : (x * A))
#define d_LeakyReLU(X,_hA) ((x > 0) ? 1 : (A))



void int_MinMaxClip(int Clip,int* Value)
{
  if (*Value > Clip)
  {
    *Value = Clip;
  }
  else if(*Value < -Clip)
  {
    *Value = -Clip;
  }
}



void float_MinMaxClip(float Clip,float* Value)
{
  if (*Value > Clip)
  {
    *Value = Clip;
  }
  else if(*Value < -Clip)
  {
    *Value = -Clip;
  }
}

void FMatrix_t_MinMaxClip(float Clip,FMatrix_t* FMatrix)
{
  for (int i = 0; i < FMatrix->X; i++)
  {
    for (int j = 0; j < FMatrix->Y; j++)
    {
      float_MinMaxClip(Clip,(FMatrix->Array + i*FMatrix->Y + j));
    }
  }
}


IMatrix_t* CreateIntegerMatrix(int x,int y)
{
  IMatrix_t* Matrix =(IMatrix_t*) malloc(sizeof(IMatrix_t));

  Matrix->Array = (int *)malloc(x * y * sizeof(int));
  Matrix->X = x;
  Matrix->Y = y;
  qSet_2D_Matrix_Elements(Matrix,0)

  return Matrix;
}

void CopyIntegerMatrix(IMatrix_t* IMatrix0,IMatrix_t* IMatrix1)
{
  for (int i = 0; i < IMatrix0->X; i++)
  {
    for (int j = 0; j < IMatrix0->Y; j++)
    {
      *(IMatrix1->Array + i*IMatrix1->Y + j) = *(IMatrix0->Array + i*IMatrix0->Y + j);
    }
  }
}

IMatrix_t* MakeCopyIntegerMatrix(IMatrix_t* IMatrix0)
{
  IMatrix_t* IMatrix1 = CreateIntegerMatrix(IMatrix0->Y,IMatrix0->Y);
  for (int i = 0; i < IMatrix0->X; i++)
  {
    for (int j = 0; j < IMatrix0->Y; j++)
    {
      *(IMatrix1->Array + i*IMatrix1->Y + j) = *(IMatrix0->Array + i*IMatrix0->Y + j);
    }
  }
  return IMatrix1;
}

int GetSum_IMatrix(IMatrix_t* IMatrix)
{
  register int Sum = 0;
  for (int i = 0; i < IMatrix->X; i++)
  {
    for (int j = 0; j < IMatrix->Y; j++)
    {
      Sum += *(IMatrix->Array + i*IMatrix->Y + j);
    }
  }
  return Sum;
}

IMatrix_t* CreateIntegerIdentityMatrix(int x,int y)
{
  IMatrix_t* Matrix = (IMatrix_t*) malloc(sizeof(IMatrix_t));

  Matrix->Array = (int *)malloc(x * y * sizeof(int));
  Matrix->X = x;
  Matrix->Y = y;
  for (int i = 0; i < Matrix->X; i++)
  {
    for (int j = 0; j < Matrix->Y; j++)
    {
      if(i == j)
      {
        *(Matrix->Array + i*Matrix->Y + j) = 1;
      }
      else
      {
        *(Matrix->Array + i*Matrix->X + j) = 0;
      }

    }
  }
  return Matrix;
}

//&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&

FMatrix_t* Create_FMatrix_t(int x,int y)
{
  FMatrix_t* FMatrix = (FMatrix_t*)malloc(sizeof(FMatrix_t));

  FMatrix->Array = (float *)malloc(x * y * sizeof(float));
  FMatrix->X = x;
  FMatrix->Y = y;
  //float Zero = 0;
  //qSet_2D_Matrix_Elements(FMatrix,0)
  return FMatrix;
}

FMatrix_t* CreateZero_FMatrix_t(int x,int y)
{
  FMatrix_t* FMatrix = (FMatrix_t*)malloc(sizeof(FMatrix_t));

  FMatrix->Array = (float *)malloc(x * y * sizeof(float));
  FMatrix->X = x;
  FMatrix->Y = y;
  //float Zero = 0;
  qSet_2D_Matrix_Elements(FMatrix,0)
  return FMatrix;
}

FMatrix_t* CreateFloatMatrix(int x,int y)
{
  FMatrix_t* FMatrix = (FMatrix_t*)malloc(sizeof(FMatrix_t));

  FMatrix->Array = (float *)malloc(x * y * sizeof(float));
  FMatrix->X = x;
  FMatrix->Y = y;
  //float Zero = 0;
  qSet_2D_Matrix_Elements(FMatrix,0)
  return FMatrix;
}

FMatrix_t* CreatefloatIdentityMatrix(int x,int y)
{
  FMatrix_t* Matrix =(FMatrix_t*) malloc(sizeof(FMatrix_t));

  Matrix->Array = (float *)malloc(x * y * sizeof(float));
  Matrix->X = x;
  Matrix->Y = y;
  float One = 1;
  for (int i = 0; i < Matrix->X; i++)
  {
    for (int j = 0; j < Matrix->Y; j++)
    {
      if(i == j)
      {
        *(Matrix->Array + i*Matrix->Y + j) = One;
      }
      else
      {
        *(Matrix->Array + i*Matrix->Y + j) = 0;
      }

    }
  }
  return Matrix;
}

FMatrix_t* CreateIdentity_FMatrix_t(int x,int y)
{
  FMatrix_t* Matrix =(FMatrix_t*) malloc(sizeof(FMatrix_t));

  Matrix->Array = (float *)malloc(x * y * sizeof(float));
  Matrix->X = x;
  Matrix->Y = y;
  float One = 1;
  for (int i = 0; i < Matrix->X; i++)
  {
    for (int j = 0; j < Matrix->Y; j++)
    {
      if(i == j)
      {
        *(Matrix->Array + i*Matrix->Y + j) = One;
      }
      else
      {
        *(Matrix->Array + i*Matrix->Y + j) = 0;
      }

    }
  }
  return Matrix;
}

void ZeroFloatMatrix(FMatrix_t* FMatrix)
{
  qSet_2D_Matrix_Elements(FMatrix,0)
}

void Zero_FMatrix_t(FMatrix_t* FMatrix)
{
  qSet_2D_Matrix_Elements(FMatrix,0)
}


void CopyFloatMatrix(FMatrix_t* FMatrix0,FMatrix_t* FMatrix1)
{
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      *(FMatrix1->Array + i*FMatrix1->Y + j) = *(FMatrix0->Array + i*FMatrix0->Y + j);
    }
  }
}

FMatrix_t* MakeCopyFMatrix(FMatrix_t* Matrix0)
{
  FMatrix_t* Copy = CreateFloatMatrix(Matrix0->X,Matrix0->Y);
  for(int XAxis = 0; XAxis < Matrix0->X;XAxis++)
  {

    for(int YAxis = 0; YAxis < Matrix0->Y;YAxis++)
    {
      *(Copy->Array+(Copy->X*YAxis+(XAxis))) = *(Matrix0->Array+(Matrix0->X*YAxis+(XAxis)));
    }
  }
  return Copy;
}

float GetSum_FMatrix(FMatrix_t* FMatrix)
{
  register float Sum = 0;
  for (int i = 0; i < FMatrix->X; i++)
  {
    for (int j = 0; j < FMatrix->Y; j++)
    {
      Sum += *(FMatrix->Array + i*FMatrix->Y + j);
    }
  }
  return Sum;
}

void AddFloatMatrix(FMatrix_t* FMatrix0, FMatrix_t* FMatrix1, FMatrix_t* FMatrix2)
{
  for (int i = 0; i < FMatrix2->X; i++)
  {
    for (int j = 0; j < FMatrix2->Y; j++)
    {
      *(FMatrix2->Array + i*FMatrix2->Y + j) = *(FMatrix0->Array + i*FMatrix0->Y + j)+*(FMatrix1->Array + i*FMatrix1->Y + j);
    }
  }
}

void AddTo_FMatrix_t(FMatrix_t* FMatrix0, FMatrix_t* FMatrix1)
{
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      *(FMatrix0->Array + i*FMatrix0->Y + j) += *(FMatrix1->Array + i*FMatrix1->Y + j);
    }
  }
}

void SubTo_FMatrix_t(FMatrix_t* FMatrix0, FMatrix_t* FMatrix1)
{
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      *(FMatrix0->Array + i*FMatrix0->Y + j) -= *(FMatrix1->Array + i*FMatrix1->Y + j);
    }
  }
}

void Add_FMatrix_t_To_FMatrix_t(FMatrix_t* FMatrix0, FMatrix_t* FMatrix1)
{
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      *(FMatrix0->Array + i*FMatrix0->Y + j) += *(FMatrix1->Array + i*FMatrix1->Y + j);
    }
  }
}


//Tested and works
void FAbs_Sigmoid_FMatrix_t(FMatrix_t* PropigatedActivation)
{
  //PrintFloatMatrix(PropigatedActivation);
  //printf("Sigmoid_FMatrix_t recieved proper PropigatedActivation structure\n");

  //x*log(fabs(x)+1)/fabs(x);
  for(int XAxis = 0; XAxis < PropigatedActivation->X;XAxis++)
  {
    for(int YAxis = 0; YAxis < PropigatedActivation->Y;YAxis++)
    {
      (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
        =          (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
          /(1+fabs((*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))));
    }
  }
  //printf("exiting with PropigatedActivation structure\n");
  //PrintFloatMatrix(PropigatedActivation);
  //printf("exiting with Proper PropigatedActivation structure\n");
}




//Two activation methods for back propigation !
//Front end/ Back end DLL storage !
//Stack storage to implement! storage

//NN Layer Nultiplication
// use Assert -> Create To hide/ Produce Test code
void LayerMultiplication(FMatrix_t* WeightMatrix,FMatrix_t** Given_PropigatedActivation)
{
  FMatrix_t* PropigatedActivation = *Given_PropigatedActivation;
  FMatrix_t* NewPropigatedActivation = (FMatrix_t*)CreateZero_FMatrix_t((PropigatedActivation)->X,(PropigatedActivation)->Y);
  float Element0 = 0;
  float Element1 = 0;
  for(int YAxis = 0; YAxis < (PropigatedActivation)->Y; YAxis++)
  {
    for(int XAxis = 0; XAxis < (PropigatedActivation)->X; XAxis++)
    {

      //Loop Through Weight Matrix and Add To NewPropigatedActivation Matrix
      Element0 = *((PropigatedActivation)->Array+((PropigatedActivation)->X*YAxis)+(XAxis));
      //printf("Activation %f\n",Element0);

      for(int XSlide = 0; XSlide < (PropigatedActivation)->X;XSlide++)
      {
        for(int YSlide = 0; YSlide < (PropigatedActivation)->Y;YSlide++)
        {
          Element1 = *(WeightMatrix->Array+(WeightMatrix->X * (PropigatedActivation)->X*YAxis + (PropigatedActivation)->X*(PropigatedActivation)->Y*XAxis +(PropigatedActivation)->Y*XSlide + YSlide));
          float Result = Element0 * Element1;
          (*(NewPropigatedActivation->Array+NewPropigatedActivation->Y*XSlide+YSlide)) += Result;
          /*
          printf("==================================================================\n");
          printf("\tElement0 %f\n",Element0);
          printf("\tElement1 %f\n",Element1);
          printf("\tResult %f\n",(*(NewPropigatedActivation->Array+NewPropigatedActivation->Y*XSlide+YSlide)));
          printf("Result-> %d\n", NewPropigatedActivation->Y*XSlide+YSlide);
          printf("YAxis %d, XAxis %d, XSlide %d, YSlide %d, Position %d \n",YAxis,XAxis,XSlide,YSlide,(WeightMatrix->X*(*PropigatedActivation)->X*YAxis + (*PropigatedActivation)->X*(*PropigatedActivation)->Y*XAxis +(*PropigatedActivation)->Y*XSlide + YSlide));
          //(*(NewPropigatedActivation->Array+(WeightMatrix->X*(*PropigatedActivation)->X*YAxis + (*PropigatedActivation)->X*(*PropigatedActivation)->Y*XAxis +(*PropigatedActivation)->Y*XSlide + YSlide))) += Element0 * Element1;
          printf("==================================================================\n");
          */

        }
      }



    }
  }

  //printf("Propigated Matrix\n");
  //PrintFloatMatrix(NewPropigatedActivation);

  //PrintFloatMatrix(*Given_PropigatedActivation);
  Free_FMatrix_t((PropigatedActivation));
  *Given_PropigatedActivation = NewPropigatedActivation;

  //PrintFloatMatrix(*Given_PropigatedActivation);
  //printf("Exiting Layer Multiplication\n");
}




void NN_MaskedLayerMultiplication(FMatrix_t* WeightMatrix,FMatrix_t** Given_PropigatedActivation,FMatrix_t* DropOutMask)
{
  FMatrix_t* PropigatedActivation = *Given_PropigatedActivation;
  FMatrix_t* NewPropigatedActivation = (FMatrix_t*)CreateZero_FMatrix_t((PropigatedActivation)->X,(PropigatedActivation)->Y);
  float Element0 = 0;
  float Element1 = 0;
  for(int YAxis = 0; YAxis < (PropigatedActivation)->Y; YAxis++)
  {
    for(int XAxis = 0; XAxis < (PropigatedActivation)->X; XAxis++)
    {

      //Loop Through Weight Matrix and Add To NewPropigatedActivation Matrix
      Element0 = *((PropigatedActivation)->Array+((PropigatedActivation)->X*YAxis)+(XAxis));
      //printf("Activation %f\n",Element0);
      if(*((DropOutMask)->Array+((DropOutMask)->X*YAxis)+(XAxis)) == 1)
      {
        for(int XSlide = 0; XSlide < (PropigatedActivation)->X;XSlide++)
        {
          for(int YSlide = 0; YSlide < (PropigatedActivation)->Y;YSlide++)
          {
            Element1 = *(WeightMatrix->Array+(WeightMatrix->X * (PropigatedActivation)->X*YAxis + (PropigatedActivation)->X*(PropigatedActivation)->Y*XAxis +(PropigatedActivation)->Y*XSlide + YSlide));
            float Result = Element0 * Element1;
            (*(NewPropigatedActivation->Array+NewPropigatedActivation->Y*XSlide+YSlide)) += Result;
            /*
            printf("==================================================================\n");
            printf("\tElement0 %f\n",Element0);
            printf("\tElement1 %f\n",Element1);
            printf("\tResult %f\n",(*(NewPropigatedActivation->Array+NewPropigatedActivation->Y*XSlide+YSlide)));
            printf("Result-> %d\n", NewPropigatedActivation->Y*XSlide+YSlide);
            printf("YAxis %d, XAxis %d, XSlide %d, YSlide %d, Position %d \n",YAxis,XAxis,XSlide,YSlide,(WeightMatrix->X*(*PropigatedActivation)->X*YAxis + (*PropigatedActivation)->X*(*PropigatedActivation)->Y*XAxis +(*PropigatedActivation)->Y*XSlide + YSlide));
            //(*(NewPropigatedActivation->Array+(WeightMatrix->X*(*PropigatedActivation)->X*YAxis + (*PropigatedActivation)->X*(*PropigatedActivation)->Y*XAxis +(*PropigatedActivation)->Y*XSlide + YSlide))) += Element0 * Element1;
            printf("==================================================================\n");
            */

          }
        }
      }


    }
  }

  //printf("Propigated Matrix\n");
  //PrintFloatMatrix(NewPropigatedActivation);

  //PrintFloatMatrix(*Given_PropigatedActivation);
  Free_FMatrix_t((PropigatedActivation));
  *Given_PropigatedActivation = NewPropigatedActivation;

  //PrintFloatMatrix(*Given_PropigatedActivation);
  //printf("Exiting Layer Multiplication\n");
}


//Takes the Hypothesised value from a NN for a given input, and takes the
// Desired Output to find the RootMeanSquareError
//RMSE (RootMeanSquareError)
float Cost_RMSE(FMatrix_t* Activation,FMatrix_t* Desired)
{
  //TODO Make Definition for quick defining Methods
  //float Cost = 0;
  float Cost = 0;
  for (int i = 0; i < Activation->X; i++)
  {
    for (int j = 0; j < Activation->Y; j++)
    {
      //TotalChange
      Cost += ((*(Desired->Array + i*Desired->Y + j)) - (*(Activation->Array + i*Activation->Y + j)));
    }
  }
  return Cost/(2* Activation->X * Activation->Y);
}

//TODO Rename this
//Tested and works
void Sigmoid_FMatrix_t(FMatrix_t* PropigatedActivation)
{
  //PrintFloatMatrix(PropigatedActivation);
  //printf("Sigmoid_FMatrix_t recieved proper PropigatedActivation structure\n");

  //x*log(fabs(x)+1)/fabs(x);
  for(int XAxis = 0; XAxis < PropigatedActivation->X;XAxis++)
  {
    for(int YAxis = 0; YAxis < PropigatedActivation->Y;YAxis++)
    {
      (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
        =          (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
          /(1+fabs((*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))));
    }
  }
  //printf("exiting with PropigatedActivation structure\n");
  //PrintFloatMatrix(PropigatedActivation);
  //printf("exiting with Proper PropigatedActivation structure\n");
}

void MaskedSigmoid_FMatrix_t(FMatrix_t* PropigatedActivation,FMatrix_t* DropOutMask)
{
  //PrintFloatMatrix(PropigatedActivation);
  //printf("Sigmoid_FMatrix_t recieved proper PropigatedActivation structure\n");

  //x*log(fabs(x)+1)/fabs(x);
  for(int XAxis = 0; XAxis < PropigatedActivation->X;XAxis++)
  {
    for(int YAxis = 0; YAxis < PropigatedActivation->Y;YAxis++)
    {
      if((*(DropOutMask->Array+(DropOutMask->X*YAxis)+(XAxis))) == 1)
      {
        (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
          =          (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))
            /(1+fabs((*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis)))));
      }
      else
      {
        (*(PropigatedActivation->Array+(PropigatedActivation->X*YAxis)+(XAxis))) = 0 ;
      }

    }
  }
  //printf("exiting with PropigatedActivation structure\n");
  //PrintFloatMatrix(PropigatedActivation);
  //printf("exiting with Proper PropigatedActivation structure\n");
}

void Free_FMatrix_t(FMatrix_t* Matrix)
{
  if (Matrix != NULL)
  {
    if(Matrix->Array != NULL)
    {
      free(Matrix->Array);
    }

    free(Matrix);
  }
}
//void Free_DLL_FMatrix_t(DLL_Handle_t* DLL_Handle);
QDefineFree_DLL_GivenStruct(Free_FMatrix_t,FMatrix_t)
//&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&



//Produces a 'DropOut' Matrix for a described matrix
//Note should always have at least 1 selected element and 1 non selected element
//Rates still apply correctly
//Rate 0-1
FMatrix_t* RandomMask_FMatrixMatrix(int x,int y, float Rate)
{
  FMatrix_t* GeneratedMask = CreateFloatMatrix(x,y);
  //Select 2 Random elements ->
  //increment SumSlide, from Node to Node, and test when Sum Region is reached

  for (int i = 0; i < x; i++)
  {
    for (int j = 0; j < y; j++)
    {
      if ( RandomFloat(0,1) < Rate)
      {
        *(GeneratedMask->Array +y*i +j) = 1;
      }
      else
      {
        *(GeneratedMask->Array +y*i +j) = 0;
      }
    }
  }
  int chance = RandomInteger(0,x*y);
  //printf("%d,%d\n",chance%x,chance/x);
  *(GeneratedMask->Array +y*(chance%x) +(chance/x)) = 1;
  //Get Sum
  return GeneratedMask;
}





FMatrix_t* RandomSelect_FMatrixMatrix(FMatrix_t* FMatrix)
{
  //float Sum = GetSum_FMatrix(FMatrix);
  float SumSlide = 0;
  float RandomNumber = 0;//Generate RandomNumber from [0,Sum]
  FMatrix_t* SelectMatrix = CreateFloatMatrix(FMatrix->X,FMatrix->Y);
  //increment SumSlide, from Node to Node, and test when Sum Region is reached

  for (int i = 0; i < FMatrix->X; i++)
  {
    for (int j = 0; j < FMatrix->Y; j++)
    {
      SumSlide += *(FMatrix->Array + i*FMatrix->Y + j);
      if (SumSlide <= RandomNumber)
      {

        *(SelectMatrix->Array) = *(SelectMatrix->Array + i*FMatrix->Y + j);
      }
    }
  }
  //Get Sum
  return 0;
}


int RandomSelect_FMatrixIndex(FMatrix_t* FMatrix)
{
  float Sum = GetSum_FMatrix(FMatrix);
  int SumSlide = 0;
  int RandomNumber = RandomFloat(0,Sum);//Generate RandomNumber from [0,Sum]
  int SelectedNode = -1;
  //increment SumSlide, from Node to Node, and test when Sum Region is reached
  printf("Matrix(%d,%d)\n", FMatrix->X,FMatrix->Y);

  for (int i = 0; i < FMatrix->X; i++)
  {
    for (int j = 0; j < FMatrix->Y; j++)
    {
      SumSlide += *(FMatrix->Array + i*FMatrix->Y + j);
      printf("%d=>(%d,%d)",RandomNumber, i,j);printf("SS->%d\n",SumSlide);

      if (SumSlide > RandomNumber   &&
        *(FMatrix->Array + i*FMatrix->Y + j) != 0 )
      {
        SelectedNode = (i*FMatrix->Y+j);
        printf("selected %d\n",SelectedNode);
      }
    }
  }
  //Get Sum
  return SelectedNode;
}

int RandomSelect_IMatrixIndex(IMatrix_t* IMatrix)
{
  int Sum = GetSum_IMatrix(IMatrix);
  int SumSlide = 0;
  if (Sum == 0)
  {
    return -1;
  }
  int RandomNumber = RandomInteger(1,Sum);//Generate RandomNumber from [0,Sum]

  int SelectedNode = -1;
  //increment SumSlide, from Node to Node, and test when Sum Region is reached
  //printf("Matrix(%d,%d)\n", IMatrix->X,IMatrix->Y);

  for (int i = 0; i < IMatrix->X; i++)
  {
    for (int j = 0; j < IMatrix->Y; j++)
    {
      SumSlide += *(IMatrix->Array + i*IMatrix->Y + j);
      //printf("%d=>(%d,%d)",RandomNumber, i,j);printf("SS->%d\n",SumSlide);

      if (SumSlide == RandomNumber   &&
        *(IMatrix->Array + i*IMatrix->Y + j) != 0 )
      {
        SelectedNode = (i*IMatrix->Y+j);
        //printf("selected %d\n",SelectedNode);
      }
    }
  }
  //PrintIntegerMatrix(IMatrix);
  //PrintInt(SelectedNode)
  //Get Sum
  return SelectedNode;
}





//Drop out Matrix
IMatrix_t* CreateI_DOM(int x,int y)
{


  IMatrix_t* Matrix = (IMatrix_t*)malloc(sizeof(IMatrix_t));
  Matrix->Array =(int*) malloc(sizeof(int)*x*y);
  Matrix->X = x;
  Matrix->Y = y;
  //initialized N by N matrix
  //now make it an Identity Matrix

  for(int XAxis = 0; XAxis < Matrix->X;XAxis++)
  {

    for(int YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      *(Matrix->Array+(Matrix->X*XAxis)+(YAxis)) = rand()%2;

    }
    //printf("%p",(Matrix->Array+(XAxis*Matrix->Y*sizeof(double))));
  }
  return Matrix;
}


IMatrix_t* CreateI_RW(int y,int x)
{
  srand(time(NULL));   // should only be called once

  IMatrix_t* Matrix = (IMatrix_t*)malloc(sizeof(IMatrix_t));
  Matrix->Array = (int*) malloc(sizeof(int)*x*y);
  Matrix->X = x;
  Matrix->Y = y;
  //initialized N by N matrix
  //now make it an Identity Matrix

  for(int XAxis = 0; XAxis < Matrix->X;XAxis++)
  {

    for(int YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      *(Matrix->Array+(Matrix->X*XAxis)+(YAxis)) = ((int)rand() / (int)RAND_MAX)*(2);
    }
    //printf("%p",(Matrix->Array+(XAxis*Matrix->Y*sizeof(double))));
  }
  return Matrix;
}

FMatrix_t* CreateF_RW(int y,int x)
{
  FMatrix_t* Matrix = (FMatrix_t*) malloc(sizeof(FMatrix_t));
  Matrix->Array = (float*)malloc(sizeof(float)*x*y);
  Matrix->X = x;
  Matrix->Y = y;
  //initialized N by N matrix
  //now make it an Identity Matrix

  for(int XAxis = 0; XAxis < Matrix->X;XAxis++)
  {

    for(int YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      *(Matrix->Array+(Matrix->X*XAxis)+(YAxis)) = ((float)rand() / (float)RAND_MAX)*(2);
    }
    //printf("%p",(Matrix->Array+(XAxis*Matrix->Y*sizeof(double))));
  }
  return Matrix;
}

void MultiplyFMatrices(FMatrix_t* Matrix0,FMatrix_t* Matrix1,FMatrix_t* Result)
{
  double Sum;
  float Element0 = 0;
  float Element1 = 0;
  for(int YAxis = 0; YAxis < Matrix0->Y; YAxis++)
  {
    for(int XAxis = 0; XAxis < Matrix1->X; XAxis++)
    {
    //  printf("%d,%d\n",XAxis,YAxis );
    //  printf("Value:%lf\n",*(Matrix0->Array+(Matrix0->X*XAxis*sizeof(double))+(YAxis*sizeof(double))));
      Sum = 0;
      for(int Slide = 0; Slide < Matrix0->X;Slide++)
      {
        Element0 = *(Matrix0->Array+(Matrix0->X*Slide)+(XAxis));
        Element1 = *(Matrix1->Array+(Matrix1->X*YAxis)+(Slide));
        //printf(" %f ",Element0);
        //printf(" %f ",Element1);
        //printf("=%f\n",Element0);
        Element0 *=  Element1;
        Sum += Element0;

      }

      (*(Result->Array+(Result->X*YAxis)+(XAxis))) = Sum;
    }
  }
}











void PrintIntegerMatrix(IMatrix_t* Matrix)
{
  int XAxis, YAxis;
  printf("Print Matrix:\n");
  printf("-------------\n");
  printf("(%d,%d)\n",Matrix->X,Matrix->Y);
  for(XAxis = 0; XAxis < Matrix->X;XAxis++)
  {
    printf("{");

    for(YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      //printf("( %p,",(Matrix->Array+(Matrix->X*XAxis*sizeof(double))+(YAxis*sizeof(double))));
      //printf(" %p",*(Matrix->Array+(Matrix->X*XAxis*sizeof(double))+(YAxis*sizeof(double))));
      printf(" %d ",*(Matrix->Array+(Matrix->X*YAxis+(XAxis))));
    }
    //printf("%p",(Matrix->Array+(XAxis*Matrix->Y*sizeof(double))));
    printf(" }\n");
  }

}


void CopyMatrices(FMatrix_t* Matrix0,FMatrix_t* Final)
{
  for(int XAxis = 0; XAxis < Matrix0->X;XAxis++)
  {

    for(int YAxis = 0; YAxis < Matrix0->Y;YAxis++)
    {
      *(Final->Array+(Final->X*YAxis+(XAxis))) = *(Matrix0->Array+(Matrix0->X*YAxis+(XAxis)));
    }
  }
}



void Scale_FMatrix_t(float Scale,FMatrix_t* Matrix0)
{
  for(int XAxis = 0; XAxis < Matrix0->X;XAxis++)
  {
    for(int YAxis = 0; YAxis < Matrix0->Y;YAxis++)
    {
      *(Matrix0->Array+(Matrix0->X*YAxis+(XAxis))) *= Scale;
    }
  }
}

void PrintFloatMatrix(FMatrix_t* Matrix)
{
  if(Matrix != NULL)
  {
    int XAxis, YAxis;
    printf("Print Matrix:\n");
    printf("-------------\n");
    printf("(%d,%d)\n",Matrix->X,Matrix->Y);
    for(XAxis = 0; XAxis < Matrix->X;XAxis++)
    {
      printf("{");

      for(YAxis = 0; YAxis < Matrix->Y;YAxis++)
      {
        //printf("( %p,",(Matrix->Array+(Matrix->X*XAxis*sizeof(double))+(YAxis*sizeof(double))));
        //printf(" %p",*(Matrix->Array+(Matrix->X*XAxis*sizeof(double))+(YAxis*sizeof(double))));
        //printf(" %p ",(Matrix->Array+(Matrix->X*XAxis+(YAxis))));
        printf(" %f ",*(Matrix->Array+(Matrix->X*YAxis+(XAxis))));
      }
      //printf("%p",(Matrix->Array+(XAxis*Matrix->Y*sizeof(double))));
      printf(" }\n");
    }
  }
}



void FreeMatrix(IMatrix_t* Matrix)
{
  free(Matrix->Array);
  free(Matrix);
}

void Free_IMatrix(IMatrix_t* Matrix)
{
  free(Matrix->Array);
  free(Matrix);
}





/*
bool Read_IMatrix_Parser(char* FilePath,IMatrix_t* Matrix)
{
  //Check If File Exists
  if(!FileExists(FilePath))
  {
    return false;
  }

  //Create Line To store Data Read From File
  char Line[MAXCHAR];
  //Create LineSlide to Parse through lines, skiping over Blank space,
  //and Configuration Data Structure
  char *LineSlide;
  //Create A File Pointer from the given String
  FILE *FilePointer = fopen(FilePath,"r");
  //fflush(NULL);

  bool Gathering_MV = false;
  while(fgets(Line,MAXCHAR,FilePointer) != NULL)
  {

    //Standard Braket found
    if(SkipToken("<",Line,&LineSlide))
    {
      //Test for ManagedVariable(MV)
      if(SkipToken("MV",LineSlide,&LineSlide))
      {

        //"Name = 'Test'"
        if(SkipToken("Name",LineSlide,&LineSlide))
        {

          SkipGivenCharacters(LineSlide,&Line," \t");
          if(SkipToken("=",LineSlide,&LineSlide))
          {

            SkipGivenCharacters(LineSlide,&LineSlide," \t");
            //look for (")- Start String Varible
            if(SkipToken("\"",LineSlide,&LineSlide))
            {
              //Gather Variable

              SkipGivenCharacters(LineSlide,&LineSlide," \t");
              //look for (")- end String Varible
              if(SkipToken("\"",LineSlide,&LineSlide))
              {
Gathering_MV = true;
              }
            }


          }


        }

      }

    }


  }



  fclose(FilePointer);
}
*/



/*
//"Style = 'FMatrix'"
if(SkipToken("Style",Line,&LineSlide))
{
  //Start Matrix Aquire ProtoColl

}
*/


float MeanAbsError_FMatrix(FMatrix_t* FMatrix0,FMatrix_t* Desired)
{
  float Cost = 0;
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      Cost += abs(*(FMatrix0->Array + i*FMatrix0->Y + j)-*(Desired->Array + i*Desired->Y + j));
    }
  }
  return Cost/(FMatrix0->X * FMatrix0->Y);
}

//Takes the Hypothesised value from a NN for a given input, and takes the
// Desired Output to find the RootMeanSquareError
float RootMeanSquareError(FMatrix_t* FMatrix0,FMatrix_t* Desired)
{
  float Cost = 0;
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      Cost += pow((*(FMatrix0->Array + i*FMatrix0->Y + j)-*(Desired->Array + i*Desired->Y + j)),2);
    }
  }
  return Cost/(2* FMatrix0->X * FMatrix0->Y);
}



//Takes the Hypothesised value from a NN for a given input, and takes the
// Desired Output to find the SquaredError
float SquaredError_FMatrix(FMatrix_t* FMatrix0,FMatrix_t* Desired)
{
  float Cost = 0;
  for (int i = 0; i < FMatrix0->X; i++)
  {
    for (int j = 0; j < FMatrix0->Y; j++)
    {
      Cost += pow((*(FMatrix0->Array + i*FMatrix0->Y + j)-*(Desired->Array + i*Desired->Y + j)),2);
    }
  }
  return Cost/(2* FMatrix0->X * FMatrix0->Y);
}
void Save_SFMatrix(FILE* FilePointer,FMatrix_t* Matrix)
{
  //Opens File at Provided Destination in Congiguration File
  //fflush(NULL);
  //FILE* FilePointer = fopen(FilePath,"a");
  //fflush(File_Pointer);

  //Start Marker for ManagedStructure
  fputs("<MS, Name = \"Test\", Type = \"FMatrix_t\", Function = \"Structure\">\n",FilePointer);
  char General[MAXCHAR];
  char* General_Slide = &General[0];
  char Temp[MAXCHAR];


  IntXY_ToString_WPostOp(Matrix->X,Matrix->Y,&General_Slide);
  fputs("\t{",FilePointer);
  fputs(General,FilePointer);
  fputs("}\n",FilePointer);

  //Save Matrix It self
  for(int XAxis = 0; XAxis < Matrix->X;XAxis++)
  {
    fputs("\t\t",FilePointer);
    for(int YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      snprintf(Temp,MAXCHAR,"%0.5f",*(Matrix->Array+(Matrix->X*YAxis)+(XAxis)));
      fputs(Temp,FilePointer);
      fputs(" ",FilePointer);
      //Save individual matrix value->

    }
    fputs("\n",FilePointer);
  }
  //end Marker for reproof
  fputs("</MS>.",FilePointer);
  fputs(" \n",FilePointer);

  fclose(FilePointer);
}


void Save_VFMatrix(char* FilePath,FMatrix_t* Matrix)
{
  //Opens File at Provided Destination in Congiguration File
  //fflush(NULL);
  FILE* FilePointer = fopen(FilePath,"a");
  //fflush(File_Pointer);

  //Start Marker for ManagedVariable
  fputs("<MV, Name = \"Test\", Type = \"FMatrix_t\", Function = \"Variable\">\n",FilePointer);
  char General[MAXCHAR];
  char* General_Slide = &General[0];
  char Temp[MAXCHAR];


  IntXY_ToString_WPostOp(Matrix->X,Matrix->Y,&General_Slide);
  fputs("\t{",FilePointer);
  fputs(General,FilePointer);
  fputs("}\n",FilePointer);

  //Save Matrix It self
  for(int XAxis = 0; XAxis < Matrix->X;XAxis++)
  {
    fputs("\t\t",FilePointer);
    for(int YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      snprintf(Temp,MAXCHAR,"%0.5f",*(Matrix->Array+(Matrix->X*YAxis)+(XAxis)));
      fputs(Temp,FilePointer);
      fputs(" ",FilePointer);
      //Save individual matrix value->

    }
    fputs("\n",FilePointer);
  }
  //end Marker for reproof
  fputs("</MV>.",FilePointer);
  fputs(" \n",FilePointer);

  fclose(FilePointer);
}
void Save_VIMatrix(char* FilePath,IMatrix_t* Matrix)
{
  //Opens File at Provided Destination in Congiguration File
  //fflush(NULL);
  FILE* FilePointer = fopen(FilePath,"a");
  //fflush(File_Pointer);

  //Start Marker for ManagedVariable
  fputs("<MV, Name = \"Test\", Type = \"IMatrix_t\", Function = \"Variable\">\n",FilePointer);
  char General[MAXCHAR];
  char* General_Slide = &General[0];
  char Temp[MAXCHAR];


  IntXY_ToString_WPostOp(Matrix->X,Matrix->Y,&General_Slide);
  fputs("\t{",FilePointer);
  fputs(General,FilePointer);
  fputs("}\n",FilePointer);

  //Save Matrix It self
  for(int XAxis = 0; XAxis < Matrix->X;XAxis++)
  {
    fputs("\t\t",FilePointer);
    for(int YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      snprintf(Temp,MAXCHAR,"%d",*(Matrix->Array+(Matrix->X*YAxis)+(XAxis)));
      fputs(Temp,FilePointer);
      fputs(" ",FilePointer);
      //Save individual matrix value->

    }
    fputs("\n",FilePointer);
  }
  //end Marker for reproof
  fputs("</MV>.",FilePointer);
  fputs(" \n",FilePointer);

  fclose(FilePointer);
}
void Read_VIMatrix(char* FilePath,IMatrix_t* Matrix)
{
  //Opens File at Provided Destination in Congiguration File
  //fflush(NULL);
  FILE* FilePointer = fopen(FilePath,"a");
  //fflush(File_Pointer);

  //Start Marker for ManagedVariable
  fputs("<MV, Name = \"Test\", Type = \"IMatrix_t\", Function = \"Variable\">\n",FilePointer);
  char General[MAXCHAR];
  char* General_Slide = &General[0];
  char Temp[MAXCHAR];


  IntXY_ToString_WPostOp(Matrix->X,Matrix->Y,&General_Slide);
  fputs("\t{",FilePointer);
  fputs(General,FilePointer);
  fputs("}\n",FilePointer);

  //Save Matrix It self
  for(int XAxis = 0; XAxis < Matrix->X;XAxis++)
  {
    fputs("\t\t",FilePointer);
    for(int YAxis = 0; YAxis < Matrix->Y;YAxis++)
    {
      snprintf(Temp,MAXCHAR,"%d",*(Matrix->Array+(Matrix->X*YAxis)+(XAxis)));
      fputs(Temp,FilePointer);
      fputs(" ",FilePointer);
      //Save individual matrix value->

    }
    fputs("\n",FilePointer);
  }


  //end Marker for reproof
  fputs("</MV>.",FilePointer);
  fputs(" \n",FilePointer);

  fclose(FilePointer);
}

//Testing
void Test_FM_Save()
{
  printf("Testing Float Matrix Save Function \n");
  FMatrix_t* Matrix0 = CreateF_RW(12,12);
  PrintFloatMatrix(Matrix0);

  //Save_VFMatrix("Test.txt",Matrix0);

  Free_FMatrix_t(Matrix0);
}

//Testing
void Test_OperateF_Sigmoid()
{
  printf("Testing Simulated Sigmoid Function \n");
  FMatrix_t* Matrix0 = CreateF_RW(2,2);
  FMatrix_t* Matrix1 = CreateF_RW(2,2);
  PrintFloatMatrix(Matrix0);

  Sigmoid_FMatrix_t(Matrix0);

  PrintFloatMatrix(Matrix1);

  //PrintMatrix(SMatrix);
  //system("gnome-terminal -e \"rm -f Test.txt \"");
  //SaveMatrix("Test.txt",IMatrix);
  Free_FMatrix_t(Matrix0);
  Free_FMatrix_t(Matrix1);
}

bool MakeCopyFMatrix_t_T()
{
  printf("Testing MakeCopyFMatrix_t_T()\n");
  FMatrix_t* Matrix0 = CreateF_RW(2,2);
  FMatrix_t* Matrix1 = MakeCopyFMatrix(Matrix0);

  PrintFloatMatrix(Matrix0);
  Free_FMatrix_t(Matrix0);

  PrintFloatMatrix(Matrix1);
  Free_FMatrix_t(Matrix1);

  return 1;
}


//Testing
bool Test_MultiplyMatrix()
{
  printf("Testing multiplyMatrices\n");
  FMatrix_t* Matrix0 = CreateF_RW(2,2);
  FMatrix_t* Matrix1 = CreateF_RW(2,2);
  FMatrix_t* Matrix2 = CreateF_RW(2,2);
  PrintFloatMatrix(Matrix0);
  PrintFloatMatrix(Matrix1);

  MultiplyFMatrices(Matrix0,Matrix1,Matrix2);

  PrintFloatMatrix(Matrix2);

  //PrintMatrix(SMatrix);
  //system("gnome-terminal -e \"rm -f Test.txt \"");
  //SaveMatrix("Test.txt",IMatrix);
  Free_FMatrix_t(Matrix0);
  Free_FMatrix_t(Matrix1);
  Free_FMatrix_t(Matrix2);
  return 1;
}
//Testing
void Test_I_RW()
{
  printf("Testing Integral Random weight\n");
  IMatrix_t* TMatrix = CreateI_RW(5,5);
  PrintIntegerMatrix(TMatrix);

  //PrintMatrix(SMatrix);
  //system("gnome-terminal -e \"rm -f Test.txt \"");
  //SaveMatrix("Test.txt",IMatrix);
  FreeMatrix(TMatrix);
}

//Testing
void Test_F_RW()
{
  printf("Testing float Random weight\n");
  FMatrix_t* TMatrix = CreateF_RW(5,5);
  PrintFloatMatrix(TMatrix);

  //PrintMatrix(SMatrix);
  //system("gnome-terminal -e \"rm -f Test.txt \"");
  //SaveMatrix("Test.txt",IMatrix);
  Free_FMatrix_t(TMatrix);
}

//Testing
bool Test_I_DOM()
{
  printf("Testing Integral Drop Out Matrix\n");
  IMatrix_t* IMatrix = CreateI_DOM(5,5);
  PrintIntegerMatrix(IMatrix);

  //PrintMatrix(SMatrix);
  //system("gnome-terminal -e \"rm -f Test.txt \"");
  //SaveMatrix("Test.txt",IMatrix);
  FreeMatrix(IMatrix);
  return 1;
}

//Testing
bool Test_F_DOM()
{
  printf("Testing float Drop Out Matrix\n");
  IMatrix_t* IMatrix = CreateI_DOM(5,5);
  PrintIntegerMatrix(IMatrix);

  //PrintMatrix(SMatrix);
  //system("gnome-terminal -e \"rm -f Test.txt \"");
  //SaveMatrix("Test.txt",IMatrix);
  FreeMatrix(IMatrix);
  return 1;
}

void Identity_Matrix_T()
{
  IMatrix_t* IMatrix = CreateIntegerIdentityMatrix(5,5);
  PrintIntegerMatrix(IMatrix);
  //PrintMatrix(SMatrix);
  //system("gnome-terminal -e \"rm -f Test.txt \"");
  //SaveMatrix("Test.txt",IMatrix);
  FreeMatrix(IMatrix);

  srand(time(NULL)); // should only be called once
  Test_I_DOM();
  srand(time(NULL)); // should only be called once
  Test_F_RW();
  Test_MultiplyMatrix();
  Test_OperateF_Sigmoid();
  //Test_IM_Save();
  Test_FM_Save();
  FMatrix_t* FMatrix = CreatefloatIdentityMatrix(5,5);
  PrintFloatMatrix(FMatrix);


  //PrintMatrix(SMatrix);
  //system("gnome-terminal -e \"rm -f Test.txt \"");
  //SaveMatrix("Test.txt",IMatrix);
  Free_FMatrix_t(FMatrix);
}

void RandomSelect_T()
{
  IMatrix_t* IMatrix0 = CreateIntegerIdentityMatrix(2,2);
  PrintIntegerMatrix(IMatrix0);
  for(int x=0; x<100;x++)
  {
    int y = RandomSelect_IMatrixIndex(IMatrix0);
    printf("Selected Random int");
    PrintInt(y)
    PrintLines(2)
  }

  //PrintIntegerMatrix(IMatrix1);

  FreeMatrix(IMatrix0);
  //FreeMatrix(IMatrix1);
}



void MMath_TT()
{
  int SelectedTest = 0;
  printf("0 = MakeCopyFMatrix_t_T()\n");
  printf("1 = \n");
  printf("2 = \n");
  printf("100 =\n");
  GatherTerminalInt("Please Select Test Move:",&SelectedTest);
  if (SelectedTest == 0)
  {
    MakeCopyFMatrix_t_T();
  }
  else if (SelectedTest == 1)
  {

  }
  else if (SelectedTest == 2)
  {

  }
  else if (SelectedTest == 3)
  {

  }
  else if (SelectedTest == 4)
  {

  }
  else if (SelectedTest == 100)
  {

  }
}

void MMath_T(int CallSign)
{
  printf("\n\n");
  printf("Starting MMath Tests:\n");
  printf("----------------------------\n");

  if (CallSign == 1)
  {
    //NNOperations_AT();
  }
  else
  {
    MMath_TT();
  }

  //NNOperations_OpenSave_T();

}

#endif // MMath_CU
