#include "hip/hip_runtime.h"
/*
Anthony M Schroeder


Purpose:
Create ThreadingTools interface to standardize different Threading
Techniques for ease of use.


*/
#ifndef ThreadingTools_CU
#define ThreadingTools_CU
#include "ThreadingTools.h"

#include <string>
#include <iostream>
#include <list>

#include <thread>
#include <mutex>

#include <stdio.h>      /* printf */
#include <time.h>       /* clock_t, clock, CLOCKS_PER_SEC */


typedef void *(*RunTimeFunction)(void *);


class ThreadData_t
{
public:
  ThreadData_t(){
  }
  virtual ~ThreadData_t(){}
};

class ThreadControlBlock
{
  public:
  int ThreadNumber;
  //Integer Representing Program identification

  //Start Time
  double StartTime;
  //Program state
  //  0-New
  //  1-ready
  //  2-Running
  //  3-Blocked
  //  4-Finished
  int ProgramState;
  int AccomplishedTasks;

  ThreadData_t* ThreadData;
  RunTimeFunction ThreadFunction;
  RunTimeFunction Start_Thread;
  RunTimeFunction End_Thread;
  RunTimeFunction Free_Enviornment;
  ThreadControlBlock(ThreadData_t* GivenThreadData){
    StartTime          = clock()*.000001;
  	ThreadNumber       = 0;
    ProgramState       = 0;
  	AccomplishedTasks  = 0;
    ThreadData         = GivenThreadData;

  }
  virtual ~ThreadControlBlock(){}
  virtual void StartThread();
};


class ParallelControlBlock
{

  public:
      //Parallel Components
      int MaximumThreads;
      std::list<ThreadControlBlock*> DispatchedThreads;

      std::mutex RecievingThreads_Mutex;
      std::list<ThreadControlBlock*> RecievingThreads;
      ParallelControlBlock(){

      }
      virtual ~ParallelControlBlock(){}

      virtual void DispatchThread(void *(*start_routine)(void*),void *arg){
        ThreadData_t* ThreadData = new ThreadData_t();
        ThreadControlBlock* Newthread = new ThreadControlBlock(ThreadData);
        DispatchedThreads.push_back(Newthread);
      }

      virtual void RecieveThreads(){
        RecievingThreads_Mutex.lock();
        for (ThreadControlBlock* Thread : RecievingThreads) { // c++11 range-based for loop

            DispatchedThreads.remove(Thread);
          }
        RecievingThreads_Mutex.unlock();
      }

      virtual void RecieveALLThreads(){

      }
};




/*
* @brief Pthread Process For Delaying Time to simulate a Given Process Operation
*
* @details Function Takes an integer, then delays for given time in seconds
*
* @example:
* pthread_t WaitThread;
* int OperationTime = .001;   (Seconds)
* //The Tread will wait for .001 Seconds before starting the next segment of code
* pthread_create(&WaitThread,NULL,WaitFor_x,&OperationTime);
* pthread_join(WaitThread,NULL);
void *WaitFor_x(void *OperationTime)
{
	int *OperationTime_pointer = (int *)OperationTime;

	double CurrentTime = clock()*.000001;

  //float OperationTimeAsDouble = *OperationTime;

  float OperationFinishTime = CurrentTime + (*OperationTime_pointer)* .000001;
  while(CurrentTime <= OperationFinishTime)
  {
    CurrentTime = clock()*.000001;
  }

	return NULL;
}
*/







#endif //ThreadingTools_CU
