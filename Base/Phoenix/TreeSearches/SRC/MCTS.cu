#include "hip/hip_runtime.h"
/*
====================================================================================================
Description MCTS(Conte-Carlo-Tree-Search):
- Contains MCTS(Conte-Carlo-Tree-Search), and MCTS_Node to search a search space
based on the given rules within Game.cu
====================================================================================================
Date:           13 September 2021
Script Version: 1.0
Name:           Anthony M Schroeder
Email:          as3379@nau.edu


==========================================================
Date:           15 September 2021
Script Version: 1.1
Description: Started modifying MCTS as a template<typename Game_Tp>.
==========================================================
Date:           16 September 2021
Script Version: 1.2
Description: Started modifying MCTS as a template<typename Game_Tp, typename Player_Tp>.
==========================================================
*/

#ifndef MCTS_CU
#define MCTS_CU


#include <iostream>
#include <string>
#include <list>
#include <cmath>
#include <bits/stdc++.h>

#include "TreeSearch.cu"
#include "../../Games/SRC/Game.cpp"



#define Pause int ASDF; std::cin >> ASDF;


/*
MCTS_Node

Great step by step example found here: https://www.youtube.com/watch?v=UXW2yZndl7U

@Methods:

 * @param
    Game* Instance,

 *
 * @see MCTS::Find_MAX_UCB1_Child()
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
class MCTS_Node
{
private:

public:
  //////////////////////////////////////////////////////////////////////////////
  // Values to evaluate UCB1 preformance.
  //////////////////////////////////////////////////////////////////////////////
  double NodeVisits;
  double ValueSum;
  Game_Tp* GivenGame = NULL;

  //////////////////////////////////////////////////////////////////////////////
  // List of Players to maintain turn order.
  //////////////////////////////////////////////////////////////////////////////
  std::list<Player_Tp*> Players;

  //////////////////////////////////////////////////////////////////////////////
  // pointers to maintain tree structure.
  //////////////////////////////////////////////////////////////////////////////
  MCTS_Node*           Parent       = NULL;
  MCTS_Node*           RollOutChild = NULL;
  std::list<MCTS_Node*> Children;


    //////////////////////////////////////////////////////////////////////////////
    // Initialization method.
    MCTS_Node(Game_Tp* Instance,std::list<Player_Tp*> _GivenPlayers){
      for (Player_Tp* _Player : _GivenPlayers){
            //printf("adding Player:%p\n",(_Player));
            Players.push_back(_Player);
      }
      GivenGame  = Instance;
      Children   = {};
      NodeVisits = 0;
      ValueSum   = 0;
      //printf("Creating MCTS Node w Player:%p\n",*(Players.begin()));
      //std::cin.get();
    }


    ~MCTS_Node(){
      for (MCTS_Node<Game_Tp,Player_Tp>* Node : Children){
        delete Node;
      }
      if (RollOutChild != NULL)
      {
          delete RollOutChild;
      }
      delete GivenGame;
    }

    //////////////////////////////////////////////////////////////////////////////
    // Method Declarations.
    //////////////////////////////////////////////////////////////////////////////
    double     Find_UCB1();
    void       RotatePlayers();

    MCTS_Node* Find_MAX_UCB1_Child();
    MCTS_Node* ReturnBestMove();
    MCTS_Node* RollOut();
    int        AddChildren(std::list<Game_Tp*> PossibleMoves);
    void       BackPropagation(Player_Tp* GivenPlayer);
    double     GetAverageValue();
    void       DisplayTree();
    void       DisplayTree(int Depth);
    void       DisplayStats();
    std::size_t GetHash();
};

/*
MCTS_Node* get(std::list<MCTS_Node*> _list, int _i){
    std::list<MCTS_Node*>::iterator it = _list.begin();
    for(int i=0; i<_i; i++){
        ++it;
    }
    return *it;
}*/

template <typename Game_Tp, typename Player_Tp>
void MCTS_Node<Game_Tp,Player_Tp>::RotatePlayers(){
  Players.splice(Players.end(),        // destination position
                 Players,              // source list
                 Players.begin());     // source position

};



//Preform MonteCarlo's UCB1 evaluation algorithm on a given node.
template <typename Game_Tp, typename Player_Tp>
double MCTS_Node<Game_Tp,Player_Tp>::Find_UCB1(){
  double ExploreBy = 1.4142;
  if(NodeVisits == 0)
	{
		return DBL_MAX;
	}
  float _NodeVisits;
  if (Parent != NULL){
    _NodeVisits = Parent->NodeVisits;
  }
  else{
    _NodeVisits = 0;
  }
  //Preform UCB1 Formula
  double Value = (ValueSum/NodeVisits) + ExploreBy*sqrt(log(_NodeVisits/NodeVisits));
/*
printf("Parent_NodeVisits:%f\n",_NodeVisits);
printf("log(_NodeVisits/NodeVisits):%f\n",log(_NodeVisits/NodeVisits));
printf("sqrt(log(_NodeVisits/NodeVisits):%f\n",sqrt(log(_NodeVisits/NodeVisits)));
printf("ExploreBy*sqrt(log(_NodeVisits/NodeVisits)):%f\n",ExploreBy*sqrt(log(_NodeVisits/NodeVisits)));
printf("Value:%f\n", Value);
printf("\tNodeVisits:%f\n", NodeVisits);
printf("\tValueSum:%f\n", ValueSum);*/

  return Value;
}

template <typename Game_Tp, typename Player_Tp>
MCTS_Node<Game_Tp,Player_Tp>* MCTS_Node<Game_Tp,Player_Tp>::Find_MAX_UCB1_Child(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  MCTS_Node* HighestNode  = (*Children.begin());
/*
printf("Children.size(): %lu\n",Children.size());
printf("HighestValue: %f\n",HighestValue);
printf("HighestValue-1: %f\n",HighestValue-1);*/

  for (MCTS_Node<Game_Tp,Player_Tp>* Node : Children){
      NodesValue = Node->Find_UCB1();
/*
printf("potential Node: %p\n",Node);
printf("HighestValue: %f\n",HighestValue);
printf("NodesValue: %f\n",NodesValue);
*/


      if (HighestValue <= NodesValue)
      {
      //printf("Swaping Max Node\n");
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //printf("HighestNode: %p\n",HighestNode);
  //Note: Doesnt account for NULL Node
  return HighestNode;
}

//Preform MonteCarlo's UCB1 evaluation algorithm on a given node, and return
//the node with the highest UCB1 Value.
template <typename Game_Tp, typename Player_Tp>
MCTS_Node<Game_Tp,Player_Tp>* MCTS_Node<Game_Tp,Player_Tp>::ReturnBestMove(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  MCTS_Node* HighestNode  = NULL;

  for (MCTS_Node<Game_Tp,Player_Tp>* Node : Children){
      NodesValue = Node->GetAverageValue();

      if (HighestValue < NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}




template <typename Game_Tp, typename Player_Tp>
int MCTS_Node<Game_Tp,Player_Tp>::AddChildren(std::list<Game_Tp*> PossibleInstances){
  int ChildrenAdded = 0;
  MCTS_Node* NewNode;

  //////////////////////////////////////////////////////////////////////////////
  // For each element within a list of PossibleInstances(Different Game States)
  // Add as different Childeren/Leaf Nodes
  for (Game_Tp* Instance : PossibleInstances){

      if(Instance != NULL)
      {

        //////////////////////////////////////////////////////////////////////////////
        // For Each Possible Game, Create New MCTS_Node<Game_Tp>, and add it to
        // children list.
        NewNode = new MCTS_Node<Game_Tp,Player_Tp>(Instance,(Instance->Players));
        NewNode->Parent = this;
        NewNode->RotatePlayers();

        Children.push_back(NewNode);
        ChildrenAdded++;
      }
  }
  return ChildrenAdded;
}

/*
Takes the Node itself, copies itself.
(This also copies the corresponding game state And performs Rollout on the new copy.)
Please note: also sets the copy node's parent as the given Node. (This is
for the BackPropagation step for attributing the Final game state's value back up the tree)
Afterward, it returns the new copy.

@param Nothing
@return pointer to Copied Rollout Node.

*/
template <typename Game_Tp, typename Player_Tp>
MCTS_Node<Game_Tp,Player_Tp>* MCTS_Node<Game_Tp,Player_Tp>::RollOut(){

  Game_Tp* RollOutGame = GivenGame->CopyGame();
  RollOutGame->RollOut();

  //printf("RO_WinningPlayer:%p\n",RollOutGame->WinningPlayer);
  //TODO Check if game is finished
  RollOutChild = new MCTS_Node(RollOutGame,Players);
  RollOutChild->Parent = this;
  return RollOutChild;
}


/*
BackPropagation is the final step of the MCTS. It backtracks from a rollout leaf node,
 back up the tree. This attributes Values to each parent node based on the out
 come of the current branch, for each node it tests if the current Player is the winner of the transversal.
 A winning state for that player recieves +1, Losing -1, tie +0

@param (Player* GivenPlayer)The final winner from the rollout evaluation.
@return Nothing(void)

*/
template <typename Game_Tp, typename Player_Tp>
void MCTS_Node<Game_Tp,Player_Tp>::BackPropagation(Player_Tp* GivenPlayer)
{

  NodeVisits++;


  //If no matching condition is found an apposing player won the RollOut game.
  double EvaluatedValue = -1;
  if(*(Players.begin()) == GivenPlayer)
  {
    EvaluatedValue = 1;
  }
  else if(GivenPlayer == NULL)
  {
    EvaluatedValue = 0;
  }
  else{

    EvaluatedValue = -1;
  }
  //std::cout << GivenGame->Generate_StringRepresentation();
  //printf("MCTS Node Player:%p\n",*(Players.begin()));
  //printf("     GivenPlayer:%p\n",GivenPlayer);
  //printf("  EvaluatedValue:%f\n",EvaluatedValue);
  //printf("           Value:%f\n",ValueSum);
  //printf("          Visits:%f\n",NodeVisits);
  ValueSum += EvaluatedValue;
  //printf(" Parent:%p\n",Parent);
  //If not the head Node, Keep transversing up the Search Tree.
  if (Parent != NULL)
  {
    Parent->BackPropagation(GivenPlayer);
  }
}


/*gets the average Value of a node.
 this is desired over the
O(1) vs O(1)

@param Nothing
@return pointer to Copied Rollout Node.

*/
template <typename Game_Tp, typename Player_Tp>
double MCTS_Node<Game_Tp,Player_Tp>::GetAverageValue()
{
  return ValueSum/NodeVisits;
}

template <typename Game_Tp, typename Player_Tp>
std::size_t MCTS_Node<Game_Tp,Player_Tp>::GetHash()
{
  //std::hash<Game_Tp>* Hash = new std::hash<Game_Tp>;// = std::hash<TTT>(* _Game);
  //std::size_t HashValue = Hash(GivenGame);
  //delete Hash;
  return GivenGame->Hash();
}
/*
DisplayStats


*/
template <typename Game_Tp, typename Player_Tp>
void MCTS_Node<Game_Tp,Player_Tp>::DisplayStats(){
  std::cout << "----------------------------------------\n";
  printf("\tLocation: %p\n",this);
  printf("\tPlayer: %c\n",(*Players.begin())->GameRepresentation);
  printf("\tNodeVisits:%f\n", NodeVisits);
  printf("\tValueSum:%f\n", ValueSum);
  printf("\tNode Ratio:%f\n", (ValueSum/NodeVisits));
  printf("\tUCB1:%f\n", Find_UCB1());
  printf("\tHash: %zu\n",GivenGame->Hash());
  printf("\tChilderen: %zu\n",Children.size());
  std::cout << GivenGame->Generate_StringRepresentation();
  for (MCTS_Node<Game_Tp,Player_Tp>* Node : Children){
    printf("\t\tChilderen: %p \t%f \t%f\n",Node,Node->ValueSum,Node->NodeVisits);
  }

}

/*
DisplayTree(int Depth)
  DisplayTree is a recursive function that displays the tree's structure, allowing for further
  analysis of the tree search.

@param (int Depth)
@return Void

*/
template <typename Game_Tp, typename Player_Tp>
void MCTS_Node<Game_Tp,Player_Tp>::DisplayTree(int Depth){

  //std::cout << "Displaying Depth:" << Depth << "\n";
  //std::cout << "Children length:" << Children.size() << "\n";
  if (Children.size() > 0){
    for (MCTS_Node* Child : Children) { // c++11 range-based for loop
         Child->DisplayStats();
      }
    if((Depth-1)>0){
      for (MCTS_Node* Child : Children) { // c++11 range-based for loop
           Child->DisplayTree(Depth-1);
        }
      }
  }
}

/*
DisplayTree(int Depth)
  DisplayTree is a recursive function that displays the tree's structure, allowing for further
  analysis of the tree search.
  *Shows entire TreeSearch.

@param ()
@return Void

*/
template <typename Game_Tp, typename Player_Tp>
void MCTS_Node<Game_Tp,Player_Tp>::DisplayTree(){
  // For each branch, display the game's statistics.
  //////////////////////////////////////////////////////////////////////////////
  for (MCTS_Node* Child : Children) {
      Child->DisplayStats();
    }

  std::cout << "----------------------------------------\n";
  std::cout << GivenGame->Generate_StringRepresentation();
  for (MCTS_Node* Child : Children) {
       Child->DisplayTree();
    }

}








/*
MCTS is a tree search that takes a complete view of a game and evaluates the
most optimal moves for both players through a UCB1 algorithm.
This algorithm performs a hybrid of breath and depth search to evenly search a given search space.

Great step by step example found here: https://www.youtube.com/watch?v=UXW2yZndl7U

@Methods:
Search()
Algorithm():: A recursive implementation of the MCTS algorithm. Recursively creates a serach tree based on the MCTS, searching for the most optimal move.

 * @param
    Game*_Game,
    std::list<Player*> _GivenPlayers)

 *
 * @see MCTS_Node::Find_MAX_UCB1_Child()
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
class MCTS: public TreeSimulation
{
public:

  //////////////////////////////////////////////////////////////////////////////
  // The current head node.
  //////////////////////////////////////////////////////////////////////////////
  Game_Tp* GivenGame;
  //MCTS_Node* TransversedNode;
  MCTS_Node<Game_Tp,Player_Tp>* HeadNode;
  Game_Tp* SimulatedGame;


  //////////////////////////////////////////////////////////////////////////////
  // The current head node.
  //////////////////////////////////////////////////////////////////////////////
  std::list<Player_Tp*> Players;
  Player_Tp* GivenPlayer;


  //////////////////////////////////////////////////////////////////////////////
  // Initialization method.
  MCTS(Game_Tp*_Game,std::list<Player_Tp*> _GivenPlayers){

    Players = _GivenPlayers;
    GivenPlayer = *(_GivenPlayers.begin());
    for (Player_Tp* _Player : _GivenPlayers){
          //printf("MCTS Playerlist:%p\n",(_Player));
    }
    //HeadNode  = NULL;
    //printf("new MCTS_Node's Player:%p\n",Player);
    //std::cin.get();

    HeadNode  = new MCTS_Node<Game_Tp,Player_Tp>(_Game,_GivenPlayers);
    GivenGame = _Game;
  }

  //////////////////////////////////////////////////////////////////////////////
  //
  //////////////////////////////////////////////////////////////////////////////
  virtual ~MCTS(){
    delete HeadNode;
  }

  //////////////////////////////////////////////////////////////////////////////
  // Method Declarations.
  //////////////////////////////////////////////////////////////////////////////
    MCTS_Node<Game_Tp,Player_Tp>* Algorithm(MCTS_Node<Game_Tp,Player_Tp>* TransversedNode);

    void CreateChildren();
    void TreeTraversal();
    void CreateNode();
    void RollOut();

    void EvaluateStep(MCTS_Node<Game_Tp,Player_Tp>* TransversedNode,Player_Tp* GivenPlayer);
    //double BackPropagation(MCTS_Node* TransversedNode,double GivenPlayer);
    void Search(int Depth); //,Player* GivenPlayer
    MCTS* PruneSearch(MCTS_Node<Game_Tp,Player_Tp>*SelectedNode);
    void ParallelSearch(int Depth);


    //MCTS* CreateBookMoves();
    //MCTS* SaveBookMoves(char* Path);
    //MCTS* OpenBookMoves(char* Path);
    //MCTS_Node* Find_Highest_UCB1(std::list<MCTS_Node*>MCTS_List);



    void Save(std::string FilePath);
    //MCTS* Read_MCTS_UTTT_JSON(std::string FilePath);
};








/**
   A recursive impementation of the MCTS algorithm. Recursively creates a serach
    tree based on the MCTS, searching for the most optimal move.

  This modifies the given MCTS search tree, adding MCTS_Node's.

 * @param
 *   <MCTS_Node*> TransversedNode(Is the next node to be evaluated on, either recursively or initialy).
 *
 * @return MCTS_Node,
 *
 * @see MCTS_Node::Find_MAX_UCB1_Child()
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
MCTS_Node<Game_Tp,Player_Tp>* MCTS<Game_Tp,Player_Tp>::Algorithm(MCTS_Node<Game_Tp,Player_Tp>* TransversedNode)
{
  /*
    Helper Function for MCTS::Search & EvaluateStep.
    Performs an itteration of the MCTS Algorithm on 'TransversedNode'
  */


//TransversedNode->Children.size()
//int Leaf =TransversedNode->Children.size();

  //TransversedNode->DisplayStats();
  //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
  /*
  std::cout << "TransversedNode:" <<TransversedNode << "\n";
  std::cout << "NodeVisits:" <<TransversedNode->NodeVisits << "\n";
  std::cout << "Children:"   <<TransversedNode->Children.size() << "\n";
  */

  //Pause;

  //std::cout << "TransversedNode->Children.size()  :"   << TransversedNode->Children.size() << "\n";

  if(TransversedNode->Children.size() == 0){
    //////////////////////////////////////////////////////////////////////////////
    //If Node is LeafNode, create Children nodes, and select the first node for
    // rollout.
    //////////////////////////////////////////////////////////////////////////////
    //std::cout << "TransversedNode->Children.size()  :"   << TransversedNode->Children.size() << "\n";

    if(TransversedNode->GivenGame->SimulationFinished){
      //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
      //Pause;
      return TransversedNode;
    }

    //std::cout << "NodeVisits:" <<TransversedNode->NodeVisits << "\n";
    /////////////////////////////////////////////////////////////////
    // If Leaf Node has no visits, preform rollout.
    /////////////////////////////////////////////////////////////////
    if(TransversedNode->NodeVisits == 0){
      //std::cout << "About to rool out on:"   << TransversedNode << "\n";

      return TransversedNode->RollOut();
    }


    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
    //UTTT* UTTT_Game = static_cast<UTTT*>(TransversedNode->GivenGame);


    /////////////////////////////////////////////////////////////////
    // Find all possible games from branch.
    /////////////////////////////////////////////////////////////////
    std::list<Game_Tp*> Games = TransversedNode->GivenGame->PossibleGames();
    //std::cout << "Adding Children Size:" << Games.size() << "\n";


    /////////////////////////////////////////////////////////////////
    // verify future games have been found.
    /////////////////////////////////////////////////////////////////
    if (Games.size() == 0)
    {
      return TransversedNode;
    }

    /////////////////////////////////////////////////////////////////
    //Takes the new Games and add them to the tree.
    /////////////////////////////////////////////////////////////////
    //printf("TransversedNode->GivenGame->Players.begin():%p\n",*(TransversedNode->GivenGame->Players.begin()));
    TransversedNode->AddChildren(Games);

    /////////////////////////////////////////////////////////////////
    //select the first posible node.
    /////////////////////////////////////////////////////////////////
    MCTS_Node<Game_Tp,Player_Tp>* NextNode = *TransversedNode->Children.begin();

    /////////////////////////////////////////////////////////////////
    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    /////////////////////////////////////////////////////////////////
    return Algorithm(NextNode);

  }
  //Otherwise, transverse the tree using the UCB1 formula, looking for an 'optimal' branch to evaluate.
  else{

    //Not Leaf Node, Transverse down the Tree: Find the branch with the MAX UCB1 value.
    MCTS_Node<Game_Tp,Player_Tp>* MAXNode = TransversedNode->Find_MAX_UCB1_Child();

    //printf("MAXNode: %p\n",MAXNode);
    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    return Algorithm(MAXNode);
  }
}




/**
   Helper Function for MCTS::Search. Performs an iteration of the MCTS on the parameter 'TransversedNode.' Then takes the result of Search/RollOut and performs BackPropagation to adjust the weights of each MCTS_Node within the search tree.
 *
 * @param
 *   <MCTS_Node*> TransversedNode().
 *   <Player*> GivenPlayer
          (A pointer of the current Player's turn. This is used during the
          backpropagation step to evaluate winning and losing game positions.).
 *
 * @return Void, modifies the given MCTS object, adding MCTS_Node elements to
 *   the Head node.
 *
 * @see MCTS
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
void MCTS<Game_Tp,Player_Tp>::EvaluateStep(MCTS_Node<Game_Tp,Player_Tp>* TransversedNode,Player_Tp* GivenPlayer)
{

    TransversedNode = Algorithm(TransversedNode);
    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();

    TransversedNode->BackPropagation(TransversedNode->GivenGame->TestForWinner());
}



/**
 * Preforms the Monte Carlo tree search on the game used to initialize the MCTS
 *  Object.
 *
 *
 * @param <int> Depth(Depth of search tree).
 *
 * @return Void, modifies the given MCTS object, adding MCTS_Node elements to
 *   the Head node.
 *
 * @see MCTS
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
void MCTS<Game_Tp,Player_Tp>::Search(int Depth)
{

    std::cout << "Searching Depth:" << Depth << "\n";
    // Increment counter, and perform another step within the search.
    for (int i = 0; i < Depth; i++) {
      //printf("\tDepth: %d\n",i);

      // Use helper Method EvaluateStep to increment the search.
      EvaluateStep(HeadNode,GivenPlayer);
    }
//Pause
    HeadNode->DisplayTree(1);
    HeadNode->DisplayStats();


}

template <typename Game_Tp, typename Player_Tp>
MCTS<Game_Tp,Player_Tp>* MCTS<Game_Tp,Player_Tp>::PruneSearch(MCTS_Node<Game_Tp,Player_Tp>*SelectedNode)
{

    return NULL;
}


template <typename Game_Tp, typename Player_Tp>
void MCTS<Game_Tp,Player_Tp>::ParallelSearch(int Depth)
{
    std::cout << "Searching Depth:" << Depth << "\n";

}


void MCTS<Game_Tp,Player_Tp>::Save(std::string FilePath){

}





#endif //MCTS_CU
