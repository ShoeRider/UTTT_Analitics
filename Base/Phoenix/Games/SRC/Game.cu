/*
Anthony M Schroeder


Purpose:
Create a Game interface in order to implement and integrate different games
easily.

Using a method called: Pure Virtual Functions.

*/


#include <hip/hip_runtime.h>
#include <string>
#include <iostream>

struct GameMove
{
  public:
      GameMove(){}
      virtual ~GameMove()= default;
};

class Game
{
private:

  public:
      Game(){}
      ~Game(){}

        //The following Methods use the 'Pure Virtual Function' method,
        //  where "= 0" part makes this method pure virtual,
        //  and also makes this class abstract.

        //** Not standardized ....
      //virtual bool ValidMove(int Row,int Col) = 0;
      virtual bool ValidMove(GameMove* Move)      = 0;
      virtual bool Move(GameMove* Move)           = 0;
      //virtual void AvaliableMoves(int Depth) = 0;

      virtual std::string GenerateStringRepresentation() = 0;

      //virtual void DisplayInTerminal(int Depth) = 0;
      virtual void PlayAsHuman()   = 0;
      virtual bool TestForWinner() = 0;
      //virtual void StepSimulation() = 0;
      //virtual void CopySimulation() = 0;
      //virtual void SaveSimulation() = 0;
      //virtual void ReadSimulation() = 0;
      //virtual void RollOut()        = 0;
};
