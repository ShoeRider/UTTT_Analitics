#include "hip/hip_runtime.h"
/*
Anthony M Schroeder


Purpose:
Create ThreadingTools interface to standardize different Threading
Techniques for ease of use.


*/
#ifndef ThreadingTools_CU
#define ThreadingTools_CU
#include "ThreadingTools.h"

#include <string>
#include <iostream>
#include <list>

#include <thread>
#include <mutex>

#include <stdio.h>      /* printf */
#include <time.h>       /* clock_t, clock, CLOCKS_PER_SEC */


typedef void *(*RunTimeFunction)(void *);


class ThreadData_t
{
public:
  ThreadData_t(){
  }
  virtual ~ThreadData_t(){}
};


class ThreadControl
{
  public:
  int ThreadNumber;
  //Integer Representing Program identification

  //Start Time
  double StartTime;
  //Program state
  //  0-New
  //  1-ready
  //  2-Running
  //  3-Blocked
  //  4-Finished
  int ProgramState;
  int AccomplishedTasks;

  void* ThreadData;

  pthread_t Thread;

  RunTimeFunction ThreadFunction;
  RunTimeFunction Start_Thread;
  RunTimeFunction End_Thread;
  RunTimeFunction Free_Enviornment;


  ThreadControl(void* GivenThreadData){
    StartTime          = clock()*.000001;
  	ThreadNumber       = 0;
    ProgramState       = 0;
  	AccomplishedTasks  = 0;
    ThreadData         = GivenThreadData;

  }
  void Start(){
    //pthread_create(Thread, NULL, CrazyThread, (Manager->Thread0));
  }
  virtual ~ThreadControl(){}
  virtual void StartThread();
};


template <typename Game_Tp, typename Player_Tp>
class ParallelControlBlock
{

  public:
      //Parallel Components
      int MaximumThreads;

      //////////////////////////////////////////////////////////////////////////////
      // pointers to maintain tree structure.
      //////////////////////////////////////////////////////////////////////////////
      std::list<ThreadControl*> DispatchedThreads;

      std::mutex RecievingThreads_Mutex;
      std::list<ThreadControl*> RecievingThreads;
      ParallelControlBlock(){

      }
      virtual ~ParallelControlBlock(){}


      //////////////////////////////////////////////////////////////////////////////
      // Method Declarations.
      //////////////////////////////////////////////////////////////////////////////
      void DispatchThread(void *(*start_routine)(void*),void *arg){
        ThreadData_t* ThreadData = new ThreadData_t();
        ThreadControl* Newthread = new ThreadControl(NULL);
        DispatchedThreads.push_back(Newthread);
      }

      void RecieveThreads(){
        RecievingThreads_Mutex.lock();
        for (ThreadControl* Thread : RecievingThreads) { // c++11 range-based for loop

            DispatchedThreads.remove(Thread);
          }
        RecievingThreads_Mutex.unlock();
      }

      void RecieveALLThreads(){

      }
};




/*
* @brief Pthread Process For Delaying Time to simulate a Given Process Operation
*
* @details Function Takes an integer, then delays for given time in seconds
*
* @example:
* pthread_t WaitThread;
* int OperationTime = .001;   (Seconds)
* //The Tread will wait for .001 Seconds before starting the next segment of code
* pthread_create(&WaitThread,NULL,WaitFor_x,&OperationTime);
* pthread_join(WaitThread,NULL);
void *WaitFor_x(void *OperationTime)
{
	int *OperationTime_pointer = (int *)OperationTime;

	double CurrentTime = clock()*.000001;

  //float OperationTimeAsDouble = *OperationTime;

  float OperationFinishTime = CurrentTime + (*OperationTime_pointer)* .000001;
  while(CurrentTime <= OperationFinishTime)
  {
    CurrentTime = clock()*.000001;
  }

	return NULL;
}
*/







#endif //ThreadingTools_CU
