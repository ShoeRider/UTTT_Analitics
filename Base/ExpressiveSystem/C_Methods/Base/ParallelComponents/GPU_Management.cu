#include "hip/hip_runtime.h"
#ifndef GPU_Management_CU
#define GPU_Management_CU

#include "GPU_Management.h"

void GPU_Management_V()
{
	printf("GPU_Management \t\tV:1.00\n");
}


bool SetGPU(std::string DesiredGPU)
{
		int devicesCount;
		hipGetDeviceCount(&devicesCount);
		for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
		{
				hipDeviceProp_t deviceProperties;
				hipGetDeviceProperties(&deviceProperties, deviceIndex);
				if (deviceProperties.name == DesiredGPU)
				{
						hipSetDevice(deviceIndex);
						return true;
				}
		}

		return false;
}


void List_GPUS()
{
	int devicesCount;
	hipGetDeviceCount(&devicesCount);
	for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
	{
	    hipDeviceProp_t deviceProperties;
	    hipGetDeviceProperties(&deviceProperties, deviceIndex);
			printf("%s\n",deviceProperties.name);
	}
}

__global__ void warmup(unsigned int * tmp)
{
if (threadIdx.x==0)
{
	*tmp=555;
}
return;
}

void warmUpGPU()
{
	printf("\nWarming up GPU for time trialing...\n");
	unsigned int * dev_tmp;
	unsigned int * tmp;
	tmp=(unsigned int*)malloc(sizeof(unsigned int));
	*tmp=0;
  hipError_t errCode = hipSuccess;
  errCode=hipMalloc((unsigned int**)&dev_tmp, sizeof(unsigned int));
	if(errCode != hipSuccess)
	{
		printf("\nError: dev_tmp error with code:%d\n",errCode);
	}
	warmup<<<1,256>>>(dev_tmp);

//copy data from device to host
	errCode=hipMemcpy( tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);
	if(errCode != hipSuccess)
	{
		printf("\nError: getting tmp result form GPU error with code :%d\n",errCode);
	}
hipDeviceSynchronize();
printf("\ntmp (changed to 555 on GPU): %d\n",*tmp);
hipFree(dev_tmp);

return;
}

/*
* @brief
*
* @details
*
* @example:
*/
void GPU_Management_T(int CallSign)
{
	int devicesCount;
	hipGetDeviceCount(&devicesCount);
	for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
	{
	    hipDeviceProp_t deviceProperties;
	    hipGetDeviceProperties(&deviceProperties, deviceIndex);
			printf("%s\n",deviceProperties.name);
	}
	printf("GPU Set? : %d\n",SetGPU("GeForce GTX 1070"));
	warmUpGPU();
}

#endif // GPU_Management_CU
