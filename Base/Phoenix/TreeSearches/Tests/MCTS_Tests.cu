#include "hip/hip_runtime.h"
#ifndef MCTS_Tests_CU
#define MCTS_Tests_CU

#include "../../Games/SRC/Game.cpp"
#include "../../Games/SRC/TTT/TTT.cpp"
#include "../../Games/SRC/UTTT/UTTT.cpp"
#include "../SRC/MCTS.cu"



#include <iostream>
#include <chrono>




/*
//Test TTT with MCTS
TTT_Player Player0 = TTT_Player(0,'X');
TTT_Player Player1 = TTT_Player(1,'O');
Game *_Game = new TTT({&Player0,&Player1});

TreeSimulation *Sim = new MCTS(_Game);
Sim->Search(15000,&Player0);

//delete &Player0;
//delete &Player1;
//delete _Game;
delete Sim;
*/
bool UTTT_Player_Init(){
  //std::cout << "Hello World!";
  //UTTT_Player Player0 = UTTT_Player(0,'X');
  //delete Player0;
  return 0;
}

/*
TTT_Player Player0 = TTT_Player(0,'X');
TTT_Player Player1 = TTT_Player(1,'O');

//Player* Player0 = static_cast<Player*>(&TTTPlayer0);
//Player* Player1 = static_cast<Player*>(&TTTPlayer1);

TTT *_Game = new TTT({&Player0,&Player1});

MCTS<TTT,TTT_Player> *Sim = new MCTS<TTT,TTT_Player>(_Game,{&Player0,&Player1});
Sim->Search(100000);


//delete &Player0;
//delete &Player1;
//delete _Game;
*/


/*
TODO: Fix Rotating Winner priority.
As of right now, Both players are attempting to give Player0 the win.
+I belive its fixed, need further testing, MCTS_Node values are (negative).
*/
int main() {
  std::clock_t    start;
  start = std::clock();

  UTTT_Player Player0 = UTTT_Player(0,'X');
  UTTT_Player Player1 = UTTT_Player(1,'O');

  //Player* Player0 = static_cast<Player*>(&TTTPlayer0);
  //Player* Player1 = static_cast<Player*>(&TTTPlayer1);

  UTTT *_Game = new UTTT({&Player0,&Player1});

  MCTS<UTTT,UTTT_Player> *Sim = new MCTS<UTTT,UTTT_Player>(_Game,{&Player0,&Player1});
  Sim->Search(1000000);

  //delete &Player0;
  //delete &Player1;
  //delete _Game;
  std::cout << "Time: " << (std::clock() - start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;



  delete Sim;
 return 0;
}

#endif //MCTS_Tests_CU
