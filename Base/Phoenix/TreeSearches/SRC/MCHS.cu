#include "hip/hip_runtime.h"
/*
====================================================================================================
Description MCHS(Conte-Carlo-Hash-Search):
- Contains MCHS(Conte-Carlo-Hash-Search), and MCHS_Node to search a search space
based on the given rules within Game.cu


====================================================================================================
Date:           16 October 2021
Script Version: 2.0
Description: MCHS is a  modified version of MCTS to utilize a hash table in
conjunction with the standard MCTS search tree. This uses the game hash to
quickly find the duplicate game’s within different branches and prevents
identical branches from searching the same space.
==========================================================

*/

#ifndef MCHS_CU
#define MCHS_CU


#include <iostream>
#include <string>
#include <list>
#include <cmath>
#include <bits/stdc++.h>

#include "HashTable.cu"
#include "TreeSearch.cu"
#include "../../Games/SRC/Game.cu"



#define Pause int ASDF; std::cin >> ASDF;


/*
MCHS_Node

Great step by step example found here: https://www.youtube.com/watch?v=UXW2yZndl7U

@Methods:

 * @param
    Game* Instance,

 *
 * @see MCHS::Find_MAX_UCB1_Child()
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
class MCHS_Node
{
private:

public:
  //////////////////////////////////////////////////////////////////////////////
  // Values to evaluate UCB1 preformance.
  //////////////////////////////////////////////////////////////////////////////
  double NodeVisits;
  double ValueSum;
  Game_Tp* GivenGame = NULL;

  //////////////////////////////////////////////////////////////////////////////
  // List of _Players to maintain turn order.
  //////////////////////////////////////////////////////////////////////////////
  std::list<Player_Tp*> Players;

  //////////////////////////////////////////////////////////////////////////////
  // pointers to maintain tree structure.
  //////////////////////////////////////////////////////////////////////////////
  //MCHS_Node*           Parent       = NULL;
  std::list<MCHS_Node<Game_Tp,Player_Tp>*> Parents;
  MCHS_Node<Game_Tp,Player_Tp>*           RollOutChild = NULL;
  std::list<MCHS_Node<Game_Tp,Player_Tp>*> Children;

  //////////////////////////////////////////////////////////////////////////////
  //HashTable
  HashTable_t<MCHS_Node<Game_Tp,Player_Tp>>*HashTable;

    //////////////////////////////////////////////////////////////////////////////
    // Initialization method.
    //<Game_Tp,Player_Tp>
    MCHS_Node(Game_Tp* Instance,std::list<Player_Tp*> _GivenPlayers,HashTable_t<MCHS_Node>*GivenHashTable){
      for (Player_Tp* _Player : _GivenPlayers){
            //printf("adding Player:%p\n",(_Player));
            Players.push_back(_Player);
      }

      Parents = {};
      HashTable  = GivenHashTable;
      GivenGame  = Instance;
      Children   = {};
      NodeVisits = 0;
      ValueSum   = 0;

      //printf("Creating MCHS Node w Player:%p\n",*(_Players.begin()));
      //std::cin.get();
    }


    ~MCHS_Node(){
      //for (MCHS_Node<Game_Tp,Player_Tp>* Node : Children){
        //delete Node;
      //}

      if (RollOutChild != NULL)
      {
          delete RollOutChild;
      }

      //delete Parents;

      delete GivenGame;
    }

    bool equal(MCHS_Node * Node0)
    {
      return (GivenGame->equal(Node0->GivenGame));
    }
    //////////////////////////////////////////////////////////////////////////////
    // Method Declarations.
    //////////////////////////////////////////////////////////////////////////////
    double     Find_UCB1();
    void       RotatePlayers();

    MCHS_Node* Find_MAX_UCB1_Child();
    MCHS_Node* ReturnBestMove();
    MCHS_Node* RollOut();
    int        AddChildren(std::list<Game_Tp*> PossibleMoves);
    void       BackPropagation(Player_Tp* GivenPlayer);
    double     GetAverageValue();
    void       DisplayTree();
    void       DisplayTree(int Depth);
    void       DisplayStats();
    std::size_t GetHash();
    void RefreshWeights();
};

/*
MCHS_Node* get(std::list<MCHS_Node*> _list, int _i){
    std::list<MCHS_Node*>::iterator it = _list.begin();
    for(int i=0; i<_i; i++){
        ++it;
    }
    return *it;
}*/

template <typename Game_Tp, typename Player_Tp>
void MCHS_Node<Game_Tp,Player_Tp>::RotatePlayers(){
  Players.splice(Players.end(),        // destination position
                 Players,              // source list
                 Players.begin());     // source position

};




//Preform MonteCarlo's UCB1 evaluation algorithm on a given node.
template <typename Game_Tp, typename Player_Tp>
double MCHS_Node<Game_Tp,Player_Tp>::Find_UCB1(){

  double MaxValue = INT_MAX;
  double Value = 0;



  for (MCHS_Node<Game_Tp,Player_Tp>* Parent : Parents){
    double ExploreBy = 1.4142;
    if(NodeVisits == 0)
    {
      return DBL_MAX;
    }
    float _NodeVisits = 1;
    _NodeVisits = Parent->NodeVisits;

    //Preform UCB1 Formula
    Value = (ValueSum/NodeVisits) + ExploreBy*sqrt(log(_NodeVisits/NodeVisits));
    if(MaxValue<Value){
      MaxValue = Value;
    }
  }


/*
printf("Value:%f\n", Value);
printf("\tNodeVisits:%i\n", NodeVisits);
printf("\tValueSum:%f\n", ValueSum);
*/
  return Value;
}

template <typename Game_Tp, typename Player_Tp>
MCHS_Node<Game_Tp,Player_Tp>* MCHS_Node<Game_Tp,Player_Tp>::Find_MAX_UCB1_Child(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  MCHS_Node* HighestNode  = (*Children.begin());

  for (MCHS_Node<Game_Tp,Player_Tp>* Node : Children){
      NodesValue = Node->Find_UCB1();

      if (HighestValue <= NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}

//Preform MonteCarlo's UCB1 evaluation algorithm on a given node, and return
//the node with the highest UCB1 Value.
template <typename Game_Tp, typename Player_Tp>
MCHS_Node<Game_Tp,Player_Tp>* MCHS_Node<Game_Tp,Player_Tp>::ReturnBestMove(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  MCHS_Node* HighestNode  = NULL;

  for (MCHS_Node<Game_Tp,Player_Tp>* Node : Children){
      NodesValue = Node->GetAverageValue();

      if (HighestValue < NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}


template <typename Game_Tp, typename Player_Tp>
void MCHS_Node<Game_Tp,Player_Tp>::DisplayStats(){
  std::cout << "----------------------------------------\n";
  printf("\tLocation: %p\n",this);
  printf("\tPlayer: %c\n",(*Players.begin())->GameRepresentation);
  printf("\tNodeVisits:%f\n", NodeVisits);
  printf("\tValueSum:%f\n", ValueSum);
  printf("\tNode Ratio:%f\n", (ValueSum/NodeVisits));
  printf("\tUCB1:%f\n", Find_UCB1());
  printf("\tHash: %zu\n",GivenGame->Hash());
  printf("\tChilderen: %zu\n",Children.size());
  std::cout << GivenGame->Generate_StringRepresentation();
  for (MCHS_Node<Game_Tp,Player_Tp>* Node : Children){
    printf("\t\tChilderen: %p \t%f \t%f\n",Node,Node->ValueSum,Node->NodeVisits);
  }
}

template <typename Game_Tp, typename Player_Tp>
int MCHS_Node<Game_Tp,Player_Tp>::AddChildren(std::list<Game_Tp*> PossibleInstances){
  int ChildrenAdded = 0;
  MCHS_Node* NewNode;

  //////////////////////////////////////////////////////////////////////////////
  // For each element within a list of PossibleInstances(Different Game States)
  // Add as different Childeren/Leaf Nodes
  for (Game_Tp* Instance : PossibleInstances){

      if(Instance != NULL)
      {


        //////////////////////////////////////////////////////////////////////////////
        // For Each Possible Game, Create New MCHS_Node<Game_Tp>, and add it to
        // children list.
        NewNode = new MCHS_Node<Game_Tp,Player_Tp>(Instance,(Instance->Players),HashTable);
        bool InsertedNode = false;
        NewNode->RotatePlayers();
        std::tie(NewNode, InsertedNode) = HashTable->AddGetReference(NewNode);

        Children.push_back(NewNode);
        NewNode->Parents.push_back(this);
        //std::cout << "\t"<<NewNode->Parents.size() << "\n";
/*
if(InsertedNode){
  this->RefreshWeights();
}
this->RefreshWeights();
*/
  if(!InsertedNode){

    //std::cout << "updating weights\n";
    //std::cout << "old*\t:"<<NewNode->NodeVisits<< "\n";
    //std::cout << "\t"<<this->NodeVisits<< "\n";
    RefreshWeights();
    //std::cout << "\t"<<this->NodeVisits<< "\n";
    //this->DisplayStats();
    //NewNode->DisplayStats();
  }


        ChildrenAdded++;
      }
  }
  return ChildrenAdded;
}

/*
Takes the Node itself, copies itself.        if(InsertedNode){
          this->RefreshWeights();
        }
        this->RefreshWeights();
(This also copies the corresponding game state And performs Rollout on the new copy.)
Please note: also sets the copy node's parent as the given Node. (This is
for the BackPropagation step for attributing the Final game state's value back up the tree)
Afterward, it returns the new copy.

@param Nothing
@return pointer to Copied Rollout Node.

*/
template <typename Game_Tp, typename Player_Tp>
MCHS_Node<Game_Tp,Player_Tp>* MCHS_Node<Game_Tp,Player_Tp>::RollOut(){

  Game_Tp* RollOutGame = GivenGame->CopyGame();
  RollOutGame->RollOut();

  //printf("RO_WinningPlayer:%p\n",RollOutGame->WinningPlayer);
  //TODO Check if game is finished
  //No need to free externally, MCHS_Node deconstructer will free.
  RollOutChild = new MCHS_Node<Game_Tp,Player_Tp>(RollOutGame,Players,HashTable);

  RollOutChild->Parents.push_back(this);
  //RollOutChild->Parent = this;
  return RollOutChild;
}

template <typename Game_Tp, typename Player_Tp>
void MCHS_Node<Game_Tp,Player_Tp>::RefreshWeights()
{
  if(Parents.size()){
    NodeVisits = 1;
    ValueSum   = 0;

    for (MCHS_Node<Game_Tp,Player_Tp>* Node : Children){
      NodeVisits += Node->NodeVisits;
      ValueSum   += -(Node->ValueSum);


    }
  }
  else{
    NodeVisits = 1;
    ValueSum   = 0;

    for (MCHS_Node<Game_Tp,Player_Tp>* Node : Children){
      NodeVisits += Node->NodeVisits;
      ValueSum   += (Node->ValueSum);


    }
  }

  //this->DisplayStats();
  if(Parents.size()){
    for (MCHS_Node<Game_Tp,Player_Tp>* Parent : Parents){
      //std::cout << "\tParent: " << Parent << "\n";

      Parent->RefreshWeights();

      //Parent->
    }
  }
}

/*
BackPropagation is the final step of the MCHS. It backtracks from a rollout leaf node,
 back up the tree. This attributes Values to each parent node based on the out
 come of the current branch, for each node it tests if the current Player is the winner of the transversal.
 A winning state for that player recieves +1, Losing -1, tie +0

@param (Player* GivenPlayer)The final winner from the rollout evaluation.
@return Nothing(void)

*/
template <typename Game_Tp, typename Player_Tp>
void MCHS_Node<Game_Tp,Player_Tp>::BackPropagation(Player_Tp* GivenPlayer)
{

  NodeVisits++;


  //If no matching condition is found an apposing player won the RollOut game.
  //double EvaluatedValue = -1;
  if(*(Players.begin()) == GivenPlayer)
  {
    ValueSum += 1;
  }
  else if(GivenPlayer == NULL)
  {
    ValueSum += 0;
  }
  else{

    ValueSum -= 1;
  }
  //std::cout << GivenGame->Generate_StringRepresentation();
  //printf("MCHS Node Player:%p\n",*(_Players.begin()));
  //printf("     GivenPlayer:%p\n",GivenPlayer);
  //printf("  EvaluatedValue:%f\n",EvaluatedValue);
  //printf("           Value:%f\n",ValueSum);
  //printf("          Visits:%f\n",NodeVisits);
  //ValueSum += EvaluatedValue;
  //printf(" Parent:%p\n",Parent);
  //If not the head Node, Keep transversing up the Search Tree.

    //std::cout << "Parents:"<<Parents.size()<<"\n";
  if(Parents.size()){
    for (MCHS_Node<Game_Tp,Player_Tp>* Parent : Parents){
      //std::cout << "\tParent: " << Parent << "\n";
      Parent->BackPropagation(GivenPlayer);
      //Parent->RefreshWeights();
    }
  }

  //RefreshWeights();


}


/*gets the average Value of a node.
 this is desired over the
O(1) vs O(1)

@param Nothing
@return pointer to Copied Rollout Node.

*/
template <typename Game_Tp, typename Player_Tp>
double MCHS_Node<Game_Tp,Player_Tp>::GetAverageValue()
{
  return ValueSum/NodeVisits;
}

template <typename Game_Tp, typename Player_Tp>
std::size_t MCHS_Node<Game_Tp,Player_Tp>::GetHash()
{
  //std::hash<Game_Tp>* Hash = new std::hash<Game_Tp>;// = std::hash<TTT>(* _Game);
  //std::size_t HashValue = Hash(GivenGame);
  //delete Hash;
  return GivenGame->Hash();
}
/*
DisplayStats

template <typename Game_Tp, typename Player_Tp>
void MCHS_Node<Game_Tp,Player_Tp>::DisplayStats(){
  if(NodeVisits>0)
  {
    std::cout << "----------------------------------------\n";
    printf("\tNodeVisits:%f\n", NodeVisits);
    printf("\tValueSum:%f\n", ValueSum);
    printf("\tNode Ratio:%f\n", (ValueSum/NodeVisits));
    printf("\tUCB1:%f\n", Find_UCB1());
    std::cout << GivenGame->Generate_StringRepresentation();
  }

}
*/


/*
DisplayTree(int Depth)
  DisplayTree is a recursive function that displays the tree's structure, allowing for further
  analysis of the tree search.

@param (int Depth)
@return Void

*/
template <typename Game_Tp, typename Player_Tp>
void MCHS_Node<Game_Tp,Player_Tp>::DisplayTree(int Depth){

  //std::cout << "Displaying Depth:" << Depth << "\n";
  //std::cout << "Children length:" << Children.size() << "\n";
  if (Children.size() > 0){
    for (MCHS_Node* Child : Children) { // c++11 range-based for loop
         Child->DisplayStats();
      }
    if((Depth-1)>0){
      for (MCHS_Node* Child : Children) { // c++11 range-based for loop
           Child->DisplayTree(Depth-1);
        }
      }
  }
}

/*
DisplayTree(int Depth)
  DisplayTree is a recursive function that displays the tree's structure, allowing for further
  analysis of the tree search.
  *Shows entire TreeSearch.

@param ()
@return Void

*/
template <typename Game_Tp, typename Player_Tp>
void MCHS_Node<Game_Tp,Player_Tp>::DisplayTree(){
  // For each branch, display the game's statistics.
  //////////////////////////////////////////////////////////////////////////////
  for (MCHS_Node* Child : Children) {
      Child->DisplayStats();
    }

  std::cout << "----------------------------------------\n";
  std::cout << GivenGame->Generate_StringRepresentation();
  for (MCHS_Node* Child : Children) {
       Child->DisplayTree();
    }

}







#include "HashTable.cu"
/*
MCHS is a tree search that takes a complete view of a game and evaluates the
most optimal moves for both players through a UCB1 algorithm.
This algorithm performs a hybrid of breath and depth search to evenly search a given search space.

Great step by step example found here: https://www.youtube.com/watch?v=UXW2yZndl7U

@Methods:
Search()
Algorithm():: A recursive implementation of the MCHS algorithm. Recursively creates a serach tree based on the MCHS, searching for the most optimal move.

 * @param
    Game*_Game,
    std::list<Player*> _GivenPlayers)

 *
 * @see MCHS_Node::Find_MAX_UCB1_Child()
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
class MCHS: public TreeSimulation
{
public:

  //////////////////////////////////////////////////////////////////////////////
  // The current head node.
  //////////////////////////////////////////////////////////////////////////////
  Game_Tp* GivenGame;
  MCHS_Node<Game_Tp,Player_Tp>* HeadNode;
  //Pointer to current game state.
  Game_Tp* SimulatedGame;

  //HashTable
  HashTable_t<MCHS_Node<Game_Tp,Player_Tp>>*HashTable;

  //////////////////////////////////////////////////////////////////////////////
  // The current head node.
  //////////////////////////////////////////////////////////////////////////////
  std::list<Player_Tp*> Players;
  Player_Tp* GivenPlayer;


  //////////////////////////////////////////////////////////////////////////////
  // Initialization method.
  MCHS(Game_Tp*_Game,std::list<Player_Tp*> _GivenPlayers){

    Players = _GivenPlayers;
    GivenPlayer = *(_GivenPlayers.begin());
    for (Player_Tp* _Player : _GivenPlayers){
          //printf("MCHS Playerlist:%p\n",(_Player));
    }
    //HeadNode  = NULL;
    //printf("new MCHS_Node's Player:%p\n",Player);
    //std::cin.get();
    HashTable = new HashTable_t<MCHS_Node<Game_Tp,Player_Tp>>(250000);
    HeadNode  = new MCHS_Node<Game_Tp,Player_Tp>(_Game,_GivenPlayers,HashTable);
    GivenGame = _Game;

    /*
    auto Compare = [](MCHS_Node<Game_Tp,Player_Tp> Node0,MCHS_Node<Game_Tp,Player_Tp> Node1)->bool {
      Node0;
    };
    */
    bool NewHash = false;

    std::tie(HeadNode, NewHash) = HashTable->AddGetReference(HeadNode);

  }

  //////////////////////////////////////////////////////////////////////////////
  //
  //////////////////////////////////////////////////////////////////////////////
  virtual ~MCHS(){


    delete HashTable;
    //delete HeadNode;
  }

  //////////////////////////////////////////////////////////////////////////////
  // Method Declarations.
  //////////////////////////////////////////////////////////////////////////////
    MCHS_Node<Game_Tp,Player_Tp>* Algorithm(MCHS_Node<Game_Tp,Player_Tp>* TransversedNode);
    void EvaluateStep(MCHS_Node<Game_Tp,Player_Tp>* TransversedNode,Player_Tp* GivenPlayer);
    //double BackPropagation(MCHS_Node* TransversedNode,double GivenPlayer);
    void Search(int Depth); //,Player* GivenPlayer
    MCHS* PruneSearch(MCHS_Node<Game_Tp,Player_Tp>*SelectedNode);
    void ParallelSearch(int Depth);


    //MCHS* CreateBookMoves();
    //MCHS* SaveBookMoves(char* Path);
    //MCHS* OpenBookMoves(char* Path);
    //MCHS_Node* Find_Highest_UCB1(std::list<MCHS_Node*>MCHS_List);

    void CreateChildren();
    void TreeTraversal();
    void CreateNode();
    void RollOut();
};








/**
   A recursive impementation of the MCHS algorithm. Recursively creates a serach
    tree based on the MCHS, searching for the most optimal move.

  This modifies the given MCHS search tree, adding MCHS_Node's.

 * @param
 *   <MCHS_Node*> TransversedNode(Is the next node to be evaluated on, either recursively or initialy).
 *
 * @return MCHS_Node,
 *
 * @see MCHS_Node::Find_MAX_UCB1_Child()
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
MCHS_Node<Game_Tp,Player_Tp>* MCHS<Game_Tp,Player_Tp>::Algorithm(MCHS_Node<Game_Tp,Player_Tp>* TransversedNode)
{
  /*
    Helper Function for MCHS::Search & EvaluateStep.
    Performs an itteration of the MCHS Algorithm on 'TransversedNode'
  */


//TransversedNode->Children.size()
//int Leaf =TransversedNode->Children.size();

  //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
  /*
  std::cout << "TransversedNode:" <<TransversedNode << "\n";
  std::cout << "NodeVisits:" <<TransversedNode->NodeVisits << "\n";
  std::cout << "Children:"   <<TransversedNode->Children.size() << "\n";
  */


  //Pause;


  if(TransversedNode->Children.size() == 0){
    //////////////////////////////////////////////////////////////////////////////
    //If Node is LeafNode, create Children nodes, and select the first node for
    // rollout.
    //////////////////////////////////////////////////////////////////////////////
    //std::cout << "LeafNode Detected  :"   << TransversedNode << "\n";


    /////////////////////////////////////////////////////////////////
    // If Leaf Node has no visits, preform rollout.
    /////////////////////////////////////////////////////////////////
    if(TransversedNode->NodeVisits == 0){
      //std::cout << "About to rool out on:"   << TransversedNode << "\n";

      return TransversedNode->RollOut();
    }


    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
    //UTTT* UTTT_Game = static_cast<UTTT*>(TransversedNode->GivenGame);


    /////////////////////////////////////////////////////////////////
    // Find all possible games from branch.
    /////////////////////////////////////////////////////////////////

    std::list<Game_Tp*> Games = TransversedNode->GivenGame->PossibleGames();
    //std::cout << "Adding Children Size:" << Games.size() << "\n";


    /////////////////////////////////////////////////////////////////
    // verify future games have been found.
    /////////////////////////////////////////////////////////////////
    if (Games.size() == 0)
    {
      return TransversedNode;
    }

    /////////////////////////////////////////////////////////////////
    //Takes the new Games and add them to the tree.
    /////////////////////////////////////////////////////////////////
    //printf("TransversedNode->GivenGame->Players.begin():%p\n",*(TransversedNode->GivenGame->_Players.begin()));
    TransversedNode->AddChildren(Games);

    /////////////////////////////////////////////////////////////////
    //select the first posible node.
    /////////////////////////////////////////////////////////////////
    MCHS_Node<Game_Tp,Player_Tp>* NextNode = *TransversedNode->Children.begin();

    /////////////////////////////////////////////////////////////////
    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    /////////////////////////////////////////////////////////////////
    return Algorithm(NextNode);

  }
  //Otherwise, transverse the tree using the UCB1 formula, looking for an 'optimal' branch to evaluate.
  else{

    //Not Leaf Node, Transverse down the Tree: Find the branch with the MAX UCB1 value.
    MCHS_Node<Game_Tp,Player_Tp>* MAXNode = TransversedNode->Find_MAX_UCB1_Child();

    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    return Algorithm(MAXNode);
  }
}




/**
   Helper Function for MCHS::Search. Performs an iteration of the MCHS on the parameter 'TransversedNode.' Then takes the result of Search/RollOut and performs BackPropagation to adjust the weights of each MCHS_Node within the search tree.
 *
 * @param
 *   <MCHS_Node*> TransversedNode().
 *   <Player*> GivenPlayer
          (A pointer of the current Player's turn. This is used during the
          backpropagation step to evaluate winning and losing game positions.).
 *
 * @return Void, modifies the given MCHS object, adding MCHS_Node elements to
 *   the Head node.
 *
 * @see MCHS
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
void MCHS<Game_Tp,Player_Tp>::EvaluateStep(MCHS_Node<Game_Tp,Player_Tp>* TransversedNode,Player_Tp* GivenPlayer)
{

    TransversedNode = Algorithm(TransversedNode);
    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();

    TransversedNode->BackPropagation(TransversedNode->GivenGame->TestForWinner());
}



/**
 * Preforms the Monte Carlo tree search on the game used to initialize the MCHS
 *  Object.
 *
 *
 * @param <int> Depth(Depth of search tree).
 *
 * @return Void, modifies the given MCHS object, adding MCHS_Node elements to
 *   the Head node.
 *
 * @see MCHS
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
void MCHS<Game_Tp,Player_Tp>::Search(int Depth)
{

    std::cout << "Searching Depth:" << Depth << "\n";
    // Increment counter, and perform another step within the search.
    while(HeadNode->NodeVisits < Depth){
        // Use helper Method EvaluateStep to increment the search.
        EvaluateStep(HeadNode,GivenPlayer);
    }
/*
for (int i = 0; i < Depth; i++) {
  //printf("\tDepth: %d\n",i);

  // Use helper Method EvaluateStep to increment the search.
  EvaluateStep(HeadNode,GivenPlayer);
}
*/
//Pause
    HeadNode->RefreshWeights();
    HeadNode->DisplayTree(1);
    HeadNode->DisplayStats();


}

template <typename Game_Tp, typename Player_Tp>
MCHS<Game_Tp,Player_Tp>* MCHS<Game_Tp,Player_Tp>::PruneSearch(MCHS_Node<Game_Tp,Player_Tp>*SelectedNode)
{

    return NULL;
}


template <typename Game_Tp, typename Player_Tp>
void MCHS<Game_Tp,Player_Tp>::ParallelSearch(int Depth)
{
    std::cout << "Searching Depth:" << Depth << "\n";

}


#endif //MCHS_CU
