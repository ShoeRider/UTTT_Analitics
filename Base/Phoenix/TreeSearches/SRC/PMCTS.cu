#include "hip/hip_runtime.h"
/*
====================================================================================================
Description PMCTS(Parallel Monte Carlo Tree Search):
This takes the Monte-Carlo Tree search and adds some multithreading to Create Faster Searches.

Still uses the Game interface, and template structure from MCTS.

Requires:
  "TreeSearch.cu"
  "MCTS.cu"

Possibly requires:
  "ThreadingTools.cu"

TODO:
  Implement Break Out Search from notes.
====================================================================================================
Date:           NA
Script Version: 1.0
Name:           Anthony M Schroeder
Email:          as3379@nau.edu
Implemented a 'dispatch evenly' algorithm.
==========================================================
Date:           26 September 2021
Script Version: 1.1
Name:           Anthony M Schroeder
Email:          as3379@nau.edu
Implementing different dispatch thread algorithms.
- _PMCTS: for a more directed search algorithm.
==========================================================
*/

#ifndef P_MCTS_CU
#define P_MCTS_CU


#include <iostream>
#include <string>
#include <list>
#include <cmath>
#include <bits/stdc++.h>

#include <thread>
#include <mutex>
#include "PMCTS.h"
#include "TreeSearch.cu"
#include "../SRC/MCTS.cu"
#include "../../ThreadingTools/SRC/ThreadingTools.cu"


template <typename Game_Tp, typename Player_Tp>
class PMCTS_Node;


template <typename Game_Tp, typename Player_Tp>
struct PMCTS_ThreadData_t {
    pthread_t Thread;
    //Player_Tp*
    PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode;
    double Threads;
    double Depth;
    bool Finished;
};










/*
MCTS_Node

Great step by step example found here: https://www.youtube.com/watch?v=UXW2yZndl7U

@Methods:

 * @param
    Game* Instance,

 *
 * @see MCTS::Find_MAX_UCB1_Child()
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
class PMCTS_Node
{
private:

public:
  //////////////////////////////////////////////////////////////////////////////
  // Values to evaluate UCB1 preformance.
  //////////////////////////////////////////////////////////////////////////////
  double NodeVisits;
  double ValueSum;
  Game_Tp* GivenGame = NULL;

  //////////////////////////////////////////////////////////////////////////////
  // List of _Players to maintain turn order.
  //////////////////////////////////////////////////////////////////////////////
  std::list<Player_Tp*> _Players;

  //////////////////////////////////////////////////////////////////////////////
  // pointers to maintain tree structure.
  //////////////////////////////////////////////////////////////////////////////
  PMCTS_Node*           Parent       = NULL;
  PMCTS_Node*           RollOutChild = NULL;
  std::list<PMCTS_Node*> Children;


    //////////////////////////////////////////////////////////////////////////////
    // Initialization method.
    PMCTS_Node(Game_Tp* Instance,std::list<Player_Tp*> _GivenPlayers){
      for (Player_Tp* _Player : _GivenPlayers){
            //printf("adding Player:%p\n",(_Player));
            _Players.push_back(_Player);
      }
      GivenGame  = Instance;
      Children   = {};
      NodeVisits = 0;
      ValueSum   = 0;
      //printf("Creating MCTS Node w Player:%p\n",*(_Players.begin()));
      //std::cin.get();
    }


    ~PMCTS_Node(){
      for (PMCTS_Node<Game_Tp,Player_Tp>* Node : Children){
        delete Node;
      }
      if (RollOutChild != NULL)
      {
          delete RollOutChild;
      }
      delete GivenGame;
    }

    //////////////////////////////////////////////////////////////////////////////
    // Method Declarations.
    //////////////////////////////////////////////////////////////////////////////

double     Find_UCB1();

PMCTS_Node<Game_Tp,Player_Tp>* Find_MAX_UCB1_Child();
PMCTS_Node<Game_Tp,Player_Tp>* ReturnBestMove();
PMCTS_Node<Game_Tp,Player_Tp>* RollOut();
int        AddChildren(std::list<Game_Tp*> PossibleMoves);
void       BackPropagation(Player_Tp* GivenPlayer,PMCTS_Node<Game_Tp,Player_Tp>* HeadNode);
double     GetAverageValue();
void       DisplayTree();
void       DisplayTree(int Depth);
void       DisplayStats();

};


//Preform MonteCarlo's UCB1 evaluation algorithm on a given node.
template <typename Game_Tp, typename Player_Tp>
double PMCTS_Node<Game_Tp,Player_Tp>::Find_UCB1(){
  double ExploreBy = 1.4142;
  if(NodeVisits == 0)
	{
		return INT_MAX;
	}
  float _NodeVisits;
  if (Parent != NULL){
    _NodeVisits = Parent->NodeVisits;
  }
  else{
    _NodeVisits = 0;
  }
  //Preform UCB1 Formula
  double Value = (ValueSum/NodeVisits) + ExploreBy*sqrt(log(_NodeVisits/NodeVisits));
/*
printf("Value:%f\n", Value);
printf("\tNodeVisits:%i\n", NodeVisits);
printf("\tValueSum:%f\n", ValueSum);
*/
  return Value;
}


template <typename Game_Tp, typename Player_Tp>
PMCTS_Node<Game_Tp,Player_Tp>* PMCTS_Node<Game_Tp,Player_Tp>::Find_MAX_UCB1_Child(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  PMCTS_Node<Game_Tp,Player_Tp>* HighestNode  = NULL;

  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : Children){
      NodesValue = Node->Find_UCB1();

      if (HighestValue < NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}


//Preform MonteCarlo's UCB1 evaluation algorithm on a given node, and return
//the node with the highest UCB1 Value.
template <typename Game_Tp, typename Player_Tp>
PMCTS_Node<Game_Tp,Player_Tp>* PMCTS_Node<Game_Tp,Player_Tp>::ReturnBestMove(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  PMCTS_Node<Game_Tp,Player_Tp>* HighestNode  = NULL;

  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : Children){
      NodesValue = Node->GetAverageValue();

      if (HighestValue < NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}


template <typename Game_Tp, typename Player_Tp>
int PMCTS_Node<Game_Tp,Player_Tp>::AddChildren(std::list<Game_Tp*> PossibleInstances){
  int ChildrenAdded = 0;
  PMCTS_Node<Game_Tp,Player_Tp>* NewNode;

  //////////////////////////////////////////////////////////////////////////////
  // For each element within a list of PossibleInstances(Different Game States)
  // Add as different Childeren/Leaf Nodes
  for (Game_Tp* Instance : PossibleInstances){

      if(Instance != NULL)
      {

        //////////////////////////////////////////////////////////////////////////////
        // For Each Possible Game, Create New MCTS_Node<Game_Tp>, and add it to
        // children list.
        NewNode = new PMCTS_Node<Game_Tp,Player_Tp>(Instance,(Instance->Players));
        NewNode->Parent = this;
        Children.push_back(NewNode);
        ChildrenAdded++;
      }
  }
  return ChildrenAdded;
}



/*
Takes the Node itself, copies itself.
(This also copies the corresponding game state And performs Rollout on the new copy.)
Please note: also sets the copy node's parent as the given Node. (This is
for the BackPropagation step for attributing the Final game state's value back up the tree)
Afterward, it returns the new copy.

@param Nothing
@return pointer to Copied Rollout Node.

*/
template <typename Game_Tp, typename Player_Tp>
PMCTS_Node<Game_Tp,Player_Tp>* PMCTS_Node<Game_Tp,Player_Tp>::RollOut(){

  Game_Tp* RollOutGame = GivenGame->CopyGame();
  RollOutGame->RollOut();

  //printf("RO_WinningPlayer:%p\n",RollOutGame->WinningPlayer);
  //TODO Check if game is finished
  RollOutChild = new PMCTS_Node<Game_Tp,Player_Tp>(RollOutGame,_Players);
  RollOutChild->Parent = this;
  return RollOutChild;
}



/*
BackPropagation is the final step of the MCTS. It backtracks from a rollout leaf node,
 back up the tree. This attributes Values to each parent node based on the out
 come of the current branch, for each node it tests if the current Player is the winner of the transversal.
 A winning state for that player recieves +1, Losing -1, tie +0

@param (Player* GivenPlayer)The final winner from the rollout evaluation.
@return Nothing(void)

*/
template <typename Game_Tp, typename Player_Tp>
void PMCTS_Node<Game_Tp,Player_Tp>::BackPropagation(Player_Tp* WinningPlayer,PMCTS_Node<Game_Tp,Player_Tp>* HeadNode)
{
  NodeVisits++;


  //If no matching condition is found an apposing player won the RollOut game.
  double EvaluatedValue = -1;
  if(*(_Players.begin()) == WinningPlayer)
  {
    EvaluatedValue = 1;
  }
  else if(WinningPlayer == NULL)
  {
    EvaluatedValue = 0;
  }
  //std::cout << GivenGame->Generate_StringRepresentation();
  //printf("MCTS Node Player:%p\n",*(_Players.begin()));
  //printf("     GivenPlayer:%p\n",GivenPlayer);
  //printf("  EvaluatedValue:%f\n",EvaluatedValue);
  //printf("           Value:%f\n",ValueSum);
  //printf("          Visits:%f\n",NodeVisits);
  ValueSum += EvaluatedValue;
  //printf(" Parent:  %p\n",Parent);
  //printf(" HeadNode:%p\n",HeadNode);
  //If not the head Node, Keep transversing up the Search Tree.
  //<Game_Tp,Player_Tp>


  if (
    Parent != NULL  &&
    this   != HeadNode
  )
  {
    Parent->BackPropagation(WinningPlayer,HeadNode);
  }
}

/*gets the average Value of a node.
 this is desired over the
O(1) vs O(1)

@param Nothing
@return pointer to Copied Rollout Node.

*/
template <typename Game_Tp, typename Player_Tp>
double PMCTS_Node<Game_Tp,Player_Tp>::GetAverageValue()
{
  return ValueSum/NodeVisits;
}




template <typename Game_Tp, typename Player_Tp>
void PMCTS_Node<Game_Tp,Player_Tp>::DisplayStats(){
  if(NodeVisits>0)
  {
    std::cout << "----------------------------------------\n";
    printf("\tNodeVisits:%f\n", NodeVisits);
    printf("\tValueSum:%f\n", ValueSum);
    printf("\tNode Ratio:%f\n", (ValueSum/NodeVisits));
    printf("\tUCB1:%f\n", Find_UCB1());
    std::cout << GivenGame->Generate_StringRepresentation();
  }

}



/*
DisplayTree(int Depth)
  DisplayTree is a recursive function that displays the tree's structure, allowing for further
  analysis of the tree search.

@param (int Depth)
@return Void

*/
template <typename Game_Tp, typename Player_Tp>
void PMCTS_Node<Game_Tp,Player_Tp>::DisplayTree(int Depth){


  if (Children.size() > 0){
    for (PMCTS_Node* Child : Children) { // c++11 range-based for loop
         Child->DisplayStats();
      }
    if((Depth-1)>0){
      for (PMCTS_Node* Child : Children) { // c++11 range-based for loop
           Child->DisplayTree(Depth-1);
        }
      }
  }
}

/*
DisplayTree(int Depth)
  DisplayTree is a recursive function that displays the tree's structure, allowing for further
  analysis of the tree search.
  *Shows entire TreeSearch.

@param ()
@return Void

*/
template <typename Game_Tp, typename Player_Tp>
void PMCTS_Node<Game_Tp,Player_Tp>::DisplayTree(){
  // For each branch, display the game's statistics.
  //////////////////////////////////////////////////////////////////////////////
  for (PMCTS_Node* Child : Children) {
      Child->DisplayStats();
    }

  std::cout << "----------------------------------------\n";
  std::cout << GivenGame->Generate_StringRepresentation();
  for (PMCTS_Node* Child : Children) {
       Child->DisplayTree();
    }

}



/*
MCTS is a tree search that takes a complete view of a game and evaluates the
most optimal moves for both players through a UCB1 algorithm.
This algorithm performs a hybrid of breath and depth search to evenly search a given search space.

Great step by step example found here: https://www.youtube.com/watch?v=UXW2yZndl7U

@Methods:
Search()
Algorithm():: A recursive implementation of the MCTS algorithm. Recursively creates a serach tree based on the MCTS, searching for the most optimal move.

 * @param
    Game*_Game,
    std::list<Player*> _GivenPlayers)

 *
 * @see MCTS_Node::Find_MAX_UCB1_Child()
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
class PMCTS: public TreeSimulation
{
public:

  //////////////////////////////////////////////////////////////////////////////
  //Thread Information
  double Depth;
  double Threads;
  ParallelControlBlock* ParallelCB;


  //////////////////////////////////////////////////////////////////////////////
  // The current head node.
  //////////////////////////////////////////////////////////////////////////////
  Game_Tp* GivenGame;
  //MCTS_Node* TransversedNode;
  PMCTS_Node<Game_Tp,Player_Tp>* HeadNode;
  Game_Tp* SimulatedGame;


  //////////////////////////////////////////////////////////////////////////////
  // The current head node.
  //////////////////////////////////////////////////////////////////////////////
  std::list<Player_Tp*> Players;
  Player_Tp* GivenPlayer;


  //////////////////////////////////////////////////////////////////////////////
  // Initialization method.
  PMCTS(Game_Tp*_Game,std::list<Player_Tp*> _GivenPlayers){
    Players        = _GivenPlayers;
    GivenPlayer    = *(_GivenPlayers.begin());

    HeadNode  = new PMCTS_Node<Game_Tp,Player_Tp>(_Game,_GivenPlayers);
    GivenGame = _Game;

    ParallelCB = new ParallelControlBlock();
  }

  //////////////////////////////////////////////////////////////////////////////
  //
  //////////////////////////////////////////////////////////////////////////////
  virtual ~PMCTS(){
    delete HeadNode;
    delete ParallelCB;
  }

  //////////////////////////////////////////////////////////////////////////////
  // Parallel Functions
  //////////////////////////////////////////////////////////////////////////////
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* DispatchThread(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int ThreadDepth);
  void DispatchThreads(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int ThreadDepth);
  void DispatchByPigeonHole(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int ThreadDepth);
  void DispatchByRotation(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int ThreadDepth);
  void DispatchEvenly(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int ThreadDepth);

  //////////////////////////////////////////////////////////////////////////////
  // 'Single' Threaded Algorithms
  //////////////////////////////////////////////////////////////////////////////
  PMCTS_Node<Game_Tp,Player_Tp>* PMCTS_Algorithm(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode);
  //TODO Include PMCTS Back Propagation
  PMCTS_Node<Game_Tp,Player_Tp>* MCTS_Algorithm(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode);
  //TODO Include MCTS Back Propagation


  //////////////////////////////////////////////////////////////////////////////
  // Management Functions
  //////////////////////////////////////////////////////////////////////////////
  void PerformStep(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,Player_Tp* GivenPlayer);
  //double BackPropagation(MCTS_Node* TransversedNode,double GivenPlayer);
  void PMCTS_Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Depth); //,Player* GivenPlayer
  void MCTS_Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Depth); //,Player* GivenPlayer
  PMCTS* PruneSearch(PMCTS_Node<Game_Tp,Player_Tp>*SelectedNode);
  void ParallelSearch(int Depth);

  void Search(double Threads, double Depth);
  void Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Threads, double Depth);


  void Node_BackPropagation(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,Player_Tp* GivenPlayer);
  //MCTS* CreateBookMoves();
  //MCTS* SaveBookMoves(char* Path);
  //MCTS* OpenBookMoves(char* Path);
  //MCTS_Node* Find_Highest_UCB1(std::list<MCTS_Node*>MCTS_List);


/*
void CreateChildren();
void TreeTraversal();
void CreateNode();
void RollOut();
*/

};

/**
   A recursive impementation of the MCTS algorithm. Recursively creates a serach
    tree based on the MCTS, searching for the most optimal move.

  This modifies the given MCTS search tree, adding MCTS_Node's.

 * @param
 *   <MCTS_Node*> TransversedNode(Is the next node to be evaluated on, either recursively or initialy).
 *
 * @return MCTS_Node,
 *
 * @see MCTS_Node::Find_MAX_UCB1_Child()
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
PMCTS_Node<Game_Tp,Player_Tp>* PMCTS<Game_Tp,Player_Tp>::MCTS_Algorithm(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode)
{
  /*
    Helper Function for MCTS::Search & EvaluateStep.
    Performs an itteration of the MCTS Algorithm on 'TransversedNode'
  */


//TransversedNode->Children.size()
//int Leaf =TransversedNode->Children.size();

  //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
  /*
  std::cout << "TransversedNode:" <<TransversedNode << "\n";
  std::cout << "NodeVisits:" <<TransversedNode->NodeVisits << "\n";
  std::cout << "Children:"   <<TransversedNode->Children.size() << "\n";
  */


  //Pause;

  //////////////////////////////////////////////////////////////////////////////
  //If Node is LeafNode, create Children nodes, and select the first node for
  // rollout.
  //////////////////////////////////////////////////////////////////////////////
  if(TransversedNode->Children.size() == 0){

    //std::cout << "LeafNode Detected  :"   << TransversedNode << "\n";


    /////////////////////////////////////////////////////////////////
    // If Leaf Node has no visits, preform rollout.
    /////////////////////////////////////////////////////////////////
    if(TransversedNode->NodeVisits == 0){
      //std::cout << "About to rool out on:"   << TransversedNode << "\n";

      return TransversedNode->RollOut();
    }


    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
    //UTTT* UTTT_Game = static_cast<UTTT*>(TransversedNode->GivenGame);


    /////////////////////////////////////////////////////////////////
    // Find all possible games from branch.
    /////////////////////////////////////////////////////////////////
    std::list<Game_Tp*> Games = TransversedNode->GivenGame->PossibleGames();
    //std::cout << "Adding Children Size:" << Games.size() << "\n";


    /////////////////////////////////////////////////////////////////
    // verify future games have been found.
    /////////////////////////////////////////////////////////////////
    if (Games.size() == 0)
    {
      return TransversedNode;
    }

    /////////////////////////////////////////////////////////////////
    //Takes the new Games and add them to the tree.
    /////////////////////////////////////////////////////////////////
    //printf("TransversedNode->GivenGame->Players.begin():%p\n",*(TransversedNode->GivenGame->_Players.begin()));
    TransversedNode->AddChildren(Games);

    /////////////////////////////////////////////////////////////////
    //select the first posible node.
    /////////////////////////////////////////////////////////////////
    PMCTS_Node<Game_Tp,Player_Tp>* NextNode = *TransversedNode->Children.begin();

    /////////////////////////////////////////////////////////////////
    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    /////////////////////////////////////////////////////////////////
    return MCTS_Algorithm(NextNode);

  }
  //Otherwise, transverse the tree using the UCB1 formula, looking for an 'optimal' branch to evaluate.
  else{

    //Not Leaf Node, Transverse down the Tree: Find the branch with the MAX UCB1 value.
    PMCTS_Node<Game_Tp,Player_Tp>* MAXNode = TransversedNode->Find_MAX_UCB1_Child();

    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    return MCTS_Algorithm(MAXNode);
  }
}




/*
*/


/**
   Helper Function for MCTS::Search. Performs an iteration of the MCTS on the parameter 'TransversedNode.' Then takes the result of Search/RollOut and performs BackPropagation to adjust the weights of each MCTS_Node within the search tree.
 *
 * @param
 *   <MCTS_Node*> TransversedNode().
 *   <Player*> GivenPlayer
          (A pointer of the current Player's turn. This is used during the
          backpropagation step to evaluate winning and losing game positions.).
 *
 * @return Void, modifies the given MCTS object, adding MCTS_Node elements to
 *   the Head node.
 *
 * @see MCTS
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::PerformStep(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,Player_Tp* GivenPlayer)
{


    TransversedNode = MCTS_Algorithm(TransversedNode);
    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();


    TransversedNode->BackPropagation(TransversedNode->GivenGame->TestForWinner(),HeadNode);
    //Node_BackPropagation(TransversedNode,TransversedNode->GivenGame->TestForWinner());
}


/**
 * Preforms the Monte Carlo tree search on the game used to initialize the MCTS
 *  Object.
 *
 *
 * @param <int> Depth(Depth of search tree).
 *
 * @return Void, modifies the given MCTS object, adding MCTS_Node elements to
 *   the Head node.
 *
 * @see MCTS
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::MCTS_Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Depth)
{
    // Increment counter, and perform another step within the search.

    for (int i = 0; i < Depth; i++) {

      // Use helper Method EvaluateStep to increment the search.
      PerformStep(TransversedNode,GivenPlayer);
    }
}





/**
 * Preforms the Monte Carlo tree search on the game used to initialize the MCTS
 *  Object.
 *
 *
 * @param <int> Depth(Depth of search tree).
 *
 * @return Void, modifies the given MCTS object, adding MCTS_Node elements to
 *   the Head node.
 *
 * @see MCTS
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::PMCTS_Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Depth)
{
  /////////////////////////////////////////////////////////////////
  // Using multiple Threads.Prep the Tree until Child nodes have been created.
  /////////////////////////////////////////////////////////////////

  /////////////////////////////////////////////////////////////////
  //Check if Leaf Node
  if(TransversedNode->Children.size() == 0){
    //////////////////////////////////////////////////////////////////////////////
    //If Node is LeafNode, create Children nodes, for threads.
    //////////////////////////////////////////////////////////////////////////////

    /////////////////////////////////////////////////////////////////
    // Find all possible games from branch.
    /////////////////////////////////////////////////////////////////
    std::list<Game_Tp*> Games = TransversedNode->GivenGame->PossibleGames();

    /////////////////////////////////////////////////////////////////
    // verify future games have been found.
    // If size zero Win state found ...
    /////////////////////////////////////////////////////////////////
    if (Games.size() == 0)
    {
      //Instead of returning node, should i just apply the BackPropagation values to the tree directly ?
      //return TransversedNode;
    }
    /////////////////////////////////////////////////////////////////
    //Takes the new Games and add them to the tree.
    /////////////////////////////////////////////////////////////////
    //printf("TransversedNode->GivenGame->Players.begin():%p\n",*(TransversedNode->GivenGame->_Players.begin()));
    TransversedNode->AddChildren(Games);
  }

  /////////////////////////////////////////////////////////////////
  //Use the threads given.
  DispatchThreads(TransversedNode,Threads,Depth);


  /////////////////////////////////////////////////////////////////
  //Preform BackPropagation to balance tree.
}




template <typename Game_Tp, typename Player_Tp>
PMCTS_Node<Game_Tp,Player_Tp>* PMCTS<Game_Tp,Player_Tp>::PMCTS_Algorithm(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode)
{

}











template <typename Game_Tp, typename Player_Tp>
PMCTS_Node<Game_Tp,Player_Tp>* _PMCTS_Algorithm(PMCTS_Node<Game_Tp,Player_Tp>*TransversedNode)
{
  /*
    Helper Function for MCTS::Search & EvaluateStep.
    Performs an itteration of the MCTS Algorithm on 'TransversedNode'
  */

  //////////////////////////////////////////////////////////////////////////////
  //If Node is LeafNode, create Children nodes, and select the first node for
  // rollout.
  //////////////////////////////////////////////////////////////////////////////
  if(TransversedNode->Children.size() == 0){

    //std::cout << "LeafNode Detected  :"   << TransversedNode << "\n";


    /////////////////////////////////////////////////////////////////
    // If Leaf Node has no visits, preform rollout.
    /////////////////////////////////////////////////////////////////
    if(TransversedNode->NodeVisits == 0){
      //std::cout << "About to rool out on:"   << TransversedNode << "\n";

      return TransversedNode->RollOut();
    }


    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
    //UTTT* UTTT_Game = static_cast<UTTT*>(TransversedNode->GivenGame);


    /////////////////////////////////////////////////////////////////
    // Find all possible games from branch.
    /////////////////////////////////////////////////////////////////
    std::list<Game_Tp*> Games = TransversedNode->GivenGame->PossibleGames();
    //std::cout << "Adding Children Size:" << Games.size() << "\n";


    /////////////////////////////////////////////////////////////////
    // verify future games have been found.
    /////////////////////////////////////////////////////////////////
    if (Games.size() == 0)
    {
      return TransversedNode;
    }

    /////////////////////////////////////////////////////////////////
    //Takes the new Games and add them to the tree.
    /////////////////////////////////////////////////////////////////
    //printf("TransversedNode->GivenGame->Players.begin():%p\n",*(TransversedNode->GivenGame->_Players.begin()));
    TransversedNode->AddChildren(Games);

    /////////////////////////////////////////////////////////////////
    //select the first posible node.
    /////////////////////////////////////////////////////////////////
    PMCTS_Node<Game_Tp,Player_Tp>* NextNode = *TransversedNode->Children.begin();

    /////////////////////////////////////////////////////////////////
    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    /////////////////////////////////////////////////////////////////
    return _PMCTS_Algorithm(NextNode);

  }
  //Otherwise, transverse the tree using the UCB1 formula, looking for an 'optimal' branch to evaluate.
  else{

    //Not Leaf Node, Transverse down the Tree: Find the branch with the MAX UCB1 value.
    PMCTS_Node<Game_Tp,Player_Tp>* MAXNode = TransversedNode->Find_MAX_UCB1_Child();

    //Recursivly search down the tree looking for an 'optimal' branch to evaluate.
    return _PMCTS_Algorithm(MAXNode);
  }
}


template <typename Game_Tp, typename Player_Tp>
void * _PMCTS_Search(void*GivenPMCTS_ThreadData)
{
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* PMCTS_ThreadData = static_cast<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>(GivenPMCTS_ThreadData);

  for (int i = 0; i < PMCTS_ThreadData->Depth; i++) {
  PMCTS_Node<Game_Tp,Player_Tp>*TransversedNode = PMCTS_ThreadData->TransversedNode;
    TransversedNode = _PMCTS_Algorithm<Game_Tp,Player_Tp>(TransversedNode);
    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();


    TransversedNode->BackPropagation(TransversedNode->GivenGame->TestForWinner(),PMCTS_ThreadData->TransversedNode);
    //Node_BackPropagation(TransversedNode,TransversedNode->GivenGame->TestForWinner());


  }
  PMCTS_ThreadData->Finished = true;
  //TransversedNode->DisplayTree(1);
  return 0;
  //returning PMCTS_Node<Game_Tp,Player_Tp>*
}





template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::DispatchByPigeonHole(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int ThreadDepth)
{

}

/*
template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::DispatchEvenly(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int Depth)
{
  double ThreadDepth = (Depth/TransversedNode->Children.size())+1;
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* PMCTS_ThreadData;
  //printf("Depth                           :%d\n",Depth);
  //printf("TransversedNode->Children.size():%d\n",TransversedNode->Children.size());
  //printf("ThreadDepth                     :%f\n",ThreadDepth);
  std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*> ThreadList;

  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : TransversedNode->Children){
    //MCTS_Search(Node,ThreadDepth);

    PMCTS_ThreadData = DispatchThread(TransversedNode, ThreadDepth);
    ThreadList.push_back(PMCTS_ThreadData);

  }

  for (PMCTS_ThreadData_t<Game_Tp,Player_Tp>* Node : ThreadList){

    pthread_join((Node->Thread), NULL);
    free(Node);
  }
}
*/


template <typename Game_Tp, typename Player_Tp>
PMCTS_ThreadData_t<Game_Tp,Player_Tp>* _DispatchThread(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode, int ThreadDepth)
{
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* PMCTS_ThreadData = (PMCTS_ThreadData_t<Game_Tp,Player_Tp>*) malloc(sizeof(PMCTS_ThreadData_t<Game_Tp,Player_Tp>));
  //printf("PMCTS_ThreadData_t:%p\n",PMCTS_ThreadData);
  PMCTS_ThreadData->TransversedNode = TransversedNode;
  PMCTS_ThreadData->Depth           = ThreadDepth;
  PMCTS_ThreadData->Finished        = false;
  //_PMCTS_Search<Game_Tp,Player_Tp>(PMCTS_ThreadData);
  pthread_create(&(PMCTS_ThreadData->Thread), NULL, _PMCTS_Search<Game_Tp,Player_Tp>, PMCTS_ThreadData);
  return PMCTS_ThreadData;
}

#include <chrono>
#include <thread>

/*
JoinThreads.
 * @param
 *    Takes a std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>, and takes the
 *   Finished threads and joins them.
 * Also has a internal wait 100 miliseconds to prevent overutilization of resources.

 */
template <typename Game_Tp, typename Player_Tp>
std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*> _JoinThreads(std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>ThreadList)
{
  int ThreadsJoined = 0;

  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* RemovingThread;
  while(ThreadsJoined <= 0){
      typename std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*>::iterator ThreadList_iterator = ThreadList.begin();
      while ( ThreadList_iterator != ThreadList.end())
      {
          if((*ThreadList_iterator)->Finished){
            pthread_join(((*ThreadList_iterator)->Thread), NULL);
            ThreadsJoined++;
            free((*ThreadList_iterator));
            ThreadList.erase(ThreadList_iterator++);
          }
          else
          {
              // move to next item
              ++ThreadList_iterator;
          }
      }
      std::this_thread::sleep_for(std::chrono::milliseconds(100));
    }

  return ThreadList;
}



template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::DispatchEvenly(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int Depth)
{
  double ThreadDepth = (Depth/TransversedNode->Children.size())+1;
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* PMCTS_ThreadData;


  //printf("Depth                           :%d\n",Depth);
  //printf("TransversedNode->Children.size():%d\n",TransversedNode->Children.size());
  //printf("ThreadDepth                     :%f\n",ThreadDepth);
  std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*> ThreadList;

  int ThreadsJoined    = 0;
  int TotalDispatches  = 0;
  int ThreadsDispatched = 0;
  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : TransversedNode->Children){
    //MCTS_Search(Node,ThreadDepth);

    //////////////////////////////////////////////////////////////////////////////
    //For Each Thread to dispatch, preform the following function untill complete.
    bool DispatchedForNode = true;
    while(DispatchedForNode)
    {
      //printf("DispatchedForNode:%d\n",DispatchedForNode);
      //printf("ThreadsDispatched:%d\n",ThreadsDispatched);
      //printf("Threads:%d\n",Threads);
      //printf("//////////////////////////////////////////////////////////////////////////////\n");
      //////////////////////////////////////////////////////////////////////////////
      //Dispatch Threads
      if (ThreadList.size() < Threads){
        //PMCTS_ThreadData = _DispatchThread<Game_Tp,Player_Tp>(Node, ThreadDepth);
        ThreadList.push_back(
          _DispatchThread<Game_Tp,Player_Tp>(Node, ThreadDepth)
        );
        DispatchedForNode = false;
      }


      //////////////////////////////////////////////////////////////////////////////
      //Join Threads
      if (ThreadList.size() == Threads){
        ThreadList = _JoinThreads<Game_Tp,Player_Tp>(ThreadList);
      }
    }
  }

}





template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::DispatchByRotation(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int Depth)
{
  double ThreadDepth = (Depth/TransversedNode->Children.size())+1;
  PMCTS_ThreadData_t<Game_Tp,Player_Tp>* PMCTS_ThreadData;
  //printf("Depth                           :%d\n",Depth);
  //printf("TransversedNode->Children.size():%d\n",TransversedNode->Children.size());
  //printf("ThreadDepth                     :%f\n",ThreadDepth);
  std::list<PMCTS_ThreadData_t<Game_Tp,Player_Tp>*> ThreadList;

  for (PMCTS_Node<Game_Tp,Player_Tp>* Node : TransversedNode->Children){
    //MCTS_Search(Node,ThreadDepth);
    PMCTS_ThreadData = (PMCTS_ThreadData_t<Game_Tp,Player_Tp>*) malloc(sizeof(PMCTS_ThreadData_t<Game_Tp,Player_Tp>));
    printf("PMCTS_ThreadData_t:%p\n",PMCTS_ThreadData);
    PMCTS_ThreadData->TransversedNode = Node;
    PMCTS_ThreadData->Depth = ThreadDepth;
    PMCTS_ThreadData->Finished = false;

    //_PMCTS_Search<Game_Tp,Player_Tp>(PMCTS_ThreadData);
    pthread_create(&(PMCTS_ThreadData->Thread), NULL, _PMCTS_Search<Game_Tp,Player_Tp>, PMCTS_ThreadData);
    ThreadList.push_back(PMCTS_ThreadData);
  }

/*
//////////////////////////////////////////////////////////////////////////////
// Original Free Threads Code.
for (PMCTS_ThreadData_t<Game_Tp,Player_Tp>* Node : ThreadList){
  pthread_join((Node->Thread), NULL);
  free(Node);
}

*/

  while(ThreadList.size() > 0){
    ThreadList = _JoinThreads<Game_Tp,Player_Tp>(ThreadList);
  }
  printf("Size Remaining: %lu\n",ThreadList.size());



  //delete ThreadList;
}


template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::DispatchThreads(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,int Threads, int Depth)
{
  //TODO: add Segment Dispatch Logic to both:
  // -DispatchByPigeonHole
  // -DispatchByRotation


  /////////////////////////////////////////////////////////////////
  // Determine how to dispatch Threads.
  /////////////////////////////////////////////////////////////////
  if (Threads > TransversedNode->Children.size()){
    /////////////////////////////////////////////////////////////////
    // Dispatch by PigeonHole. Giving the highest UCB1 nodes more Threads.
    /////////////////////////////////////////////////////////////////
    DispatchByPigeonHole(TransversedNode,Threads,Depth);

  }
  else{
    /////////////////////////////////////////////////////////////////
    // Dispatch by Rotation, Each Branch will eventually get a Thread gets an even Search Depth.
    /////////////////////////////////////////////////////////////////
    //DispatchByRotation(TransversedNode,Threads,Depth);
    printf("calling DispatchEvenly\n");
    DispatchEvenly(TransversedNode,Threads,Depth);
  }

}





/**
 * Preforms the Monte Carlo tree search on the game used to initialize the MCTS
 *  Object.
 *
 *
 * @param <int> Depth(Depth of search tree).
 *
 * @return Void, modifies the given MCTS object, adding MCTS_Node elements to
 *   the Head node.
 *
 * @see MCTS
 * @see Game interface(Found within Game.cu)
 */
template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::Search(PMCTS_Node<Game_Tp,Player_Tp>* TransversedNode,double Threads, double Depth)
{
  MCTS_Search(TransversedNode,5);
  DispatchThreads(TransversedNode, Threads, Depth);

/*
/////////////////////////////////////////////////////////////////
// Determine if Multiple Threads are being used.
/////////////////////////////////////////////////////////////////
if (Threads > 1){
  DispatchThreads(TransversedNode, Threads, Depth);
}
else{
  /////////////////////////////////////////////////////////////////
  // Only using one thread, preform MCTS normaly for given Depth.
  /////////////////////////////////////////////////////////////////
  MCTS_Search(TransversedNode,Depth);
}
*/

}


template <typename Game_Tp, typename Player_Tp>
void PMCTS<Game_Tp,Player_Tp>::Search(double Threads, double Depth)
{
  Search(HeadNode, Threads, Depth);
  HeadNode->DisplayTree(1);
}




#endif //P_MCTS_CU
