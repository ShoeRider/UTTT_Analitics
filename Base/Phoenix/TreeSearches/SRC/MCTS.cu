#include "hip/hip_runtime.h"
#ifndef MCTS_CU
#define MCTS_CU


#include <iostream>
#include <string>
#include <list>
#include <cmath>
#include <bits/stdc++.h>

#include "TreeSearch.cu"
#include "../../Games/SRC/Game.cu"
#include "../../Games/SRC/TTT.cu"



#define Pause int ASDF; std::cin >> ASDF;



class MCTS_Node
{
private:

public:
    double NodeVisits;
    double ValueSum;
    Game* GivenGame = NULL;
    //Player* _Player;
    std::list<Player*> _Players;

    MCTS_Node*           Parent       = NULL;
    MCTS_Node*           RollOutChild = NULL;
    std::list<MCTS_Node*> Children;

    MCTS_Node(Game* Instance,std::list<Player*> _GivenPlayers){
      for (Player* _Player : _GivenPlayers){
            printf("adding Player:%p\n",(_Player));
            _Players.push_back(_Player);
      }
      GivenGame  = Instance;
      Children   = {};
      NodeVisits = 0;
      ValueSum   = 0;
      printf("Creating MCTS Node w Player:%p\n",*(_Players.begin()));
      //std::cin.get();
    }


    ~MCTS_Node(){
      for (MCTS_Node* Node : Children){
        delete Node;
      }
      if (RollOutChild != NULL)
      {
          delete RollOutChild;
      }
      delete GivenGame;
    }
    double     Find_UCB1();

    MCTS_Node* Find_MAX_UCB1_Child();
    MCTS_Node* ReturnBestMove();
    MCTS_Node* RollOut();
    int        AddChildren(std::list<Game*> PossibleMoves);
    void       BackPropagation(Player* GivenPlayer);
    double     GetAverageValue();
    void       DisplayTree();
    void       DisplayTree(int Depth);
    void       DisplayStats();
};

/*
MCTS_Node* get(std::list<MCTS_Node*> _list, int _i){
    std::list<MCTS_Node*>::iterator it = _list.begin();
    for(int i=0; i<_i; i++){
        ++it;
    }
    return *it;
}*/

template <typename T>
T* get(std::list<T*> _list, int _i){
    typename std::list<T*>::iterator it = _list.begin();
    for(int i=0; i<_i; i++){
        ++it;
    }
    return *it;
}



//Preform MonteCarlo's UCB1 evaluation algorithm on a given node.

double MCTS_Node::Find_UCB1(){
  double ExploreBy = 1.4142;
  if(NodeVisits == 0)
	{
		return INT_MAX;
	}
  float _NodeVisits;
  if (Parent != NULL){
    _NodeVisits = Parent->NodeVisits;
  }
  else{
    _NodeVisits = 0;
  }
  //Preform UCB1 Formula
  double Value = (ValueSum/NodeVisits) + ExploreBy*sqrt(log(_NodeVisits/NodeVisits));
/*
printf("Value:%f\n", Value);
printf("\tNodeVisits:%i\n", NodeVisits);
printf("\tValueSum:%f\n", ValueSum);
*/
  return Value;
}


MCTS_Node* MCTS_Node::Find_MAX_UCB1_Child(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  MCTS_Node* HighestNode  = NULL;

  for (MCTS_Node* Node : Children){
      NodesValue = Node->Find_UCB1();

      if (HighestValue < NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}

//Preform MonteCarlo's UCB1 evaluation algorithm on a given node, and return
//the node with the highest UCB1 Value.

MCTS_Node* MCTS_Node::ReturnBestMove(){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  MCTS_Node* HighestNode  = NULL;

  for (MCTS_Node* Node : Children){
      NodesValue = Node->GetAverageValue();

      if (HighestValue < NodesValue)
      {
        HighestNode  = Node;
        HighestValue = NodesValue;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}


//For each element within a list of PossibleInstances(Different Game States)
//Add as different Childeren


int MCTS_Node::AddChildren(std::list<Game*> PossibleInstances){
  int ChildrenAdded = 0;
  MCTS_Node* NewNode;
  for (Game* Instance : PossibleInstances){
      if(Instance != NULL)
      {
        std::list<Player*> ProgressedOrder = *(new std::list<Player*>(Instance->_Players));
      /*
      ProgressedOrder.splice(ProgressedOrder.end(),        // destination position
                     ProgressedOrder,              // source list
                     ProgressedOrder.begin());     // source position
                     */
        //std::next(ProgressedOrder, 1);
        //GivenGame->_Players.begin()
        TTT* _Instance = static_cast<TTT*>(Instance);
        printf("%p\n",&(_Instance));
        printf("Create Instance->Players:%p\n",(_Instance->_Players));
        printf("Create Instance->Players:%p\n",&(_Instance->_Players));
        for (Player* _Pl : _Instance->_Players){
              printf("\t-:%p\n",(_Pl));
        }
        NewNode = new MCTS_Node(Instance,(_Instance->_Players));
        NewNode->Parent = this;
        Children.push_back(NewNode);
        ChildrenAdded++;
      }
  }
  return ChildrenAdded;
}

/*
Takes the Node itself, copies itself.
(This also copies the corresponding game state And performs Rollout on the new copy.)
Please note: also sets the copy node's parent as the given Node. (This is
for the BackPropagation step for attributing the Final game state's value back up the tree)
Afterward, it returns the new copy.

@param Nothing
@return pointer to Copied Rollout Node.

*/

MCTS_Node* MCTS_Node::RollOut(){

  Game* RollOutGame = GivenGame->CopyGame();
  RollOutGame->RollOut();

  //printf("RO_WinningPlayer:%p\n",RollOutGame->WinningPlayer);
  //TODO Check if game is finished
  RollOutChild = new MCTS_Node(RollOutGame,_Players);
  RollOutChild->Parent = this;
  return RollOutChild;
}


/*
BackPropagation is the final step of the MCTS. It backtracks from rollout leaf node,
 back up the tree, attributing the final game Value to each parent node, for each
 node it tests if the current Player is the winner of the transversal.
 A winning state for that player recieves +1, Losing -1, tie +0

@param (Player* GivenPlayer)The final winner from the rollout evaluation.
@return Nothing(void)

*/

void MCTS_Node::BackPropagation(Player* GivenPlayer)
{
  NodeVisits++;


  //If no matching condition is found an apposing player won the RollOut game.
  double EvaluatedValue = -1;
  if(*(_Players.begin()) == GivenPlayer)
  {
    EvaluatedValue = 1;
  }
  else if(GivenPlayer == NULL)
  {
    EvaluatedValue = 0;
  }
  std::cout << GivenGame->Generate_StringRepresentation();
  printf("MCTS Node Player:%p\n",*(_Players.begin()));
  printf("     GivenPlayer:%p\n",GivenPlayer);
  printf("  EvaluatedValue:%f\n",EvaluatedValue);
  printf("           Value:%f\n",ValueSum);
  printf("          Visits:%f\n",NodeVisits);
  ValueSum += EvaluatedValue;
  printf(" Parent:%p\n",Parent);
  //If not the head Node, Keep transversing up the Search Tree.
  if (Parent != NULL)
  {
    Parent->BackPropagation(GivenPlayer);
  }
}

/*gets the average Value of a node.
 this is desired over the
O(1) vs O(1)

@param Nothing
@return pointer to Copied Rollout Node.

*/

double MCTS_Node::GetAverageValue()
{
  return ValueSum/NodeVisits;
}


void MCTS_Node::DisplayStats(){
  if(NodeVisits>0)
  {
    std::cout << "----------------------------------------\n";
    printf("ValueSum:%f\n", ValueSum);
    printf("\tNodeVisits:%f\n", NodeVisits);
    printf("\tValueSum:%f\n", ValueSum);
    std::cout << GivenGame->Generate_StringRepresentation();
  }

}

/*
DisplayTree(int Depth)
  DisplayTree is a recursive function that displays the tree's structure, allowing for further
  analysis of the tree search.

@param (int Depth)
@return Void

*/

void MCTS_Node::DisplayTree(int Depth){

  std::cout << "Displaying Depth:" << Depth << "\n";
  std::cout << "Children length:" << Children.size() << "\n";
  if (Children.size() > 0){
    for (MCTS_Node* Child : Children) { // c++11 range-based for loop
         Child->DisplayStats();
      }
    if((Depth-1)>0){
      for (MCTS_Node* Child : Children) { // c++11 range-based for loop
           Child->DisplayTree(Depth-1);
        }
      }
  }
}

/*
DisplayTree(int Depth)
  DisplayTree is a recursive function that displays the tree's structure, allowing for further
  analysis of the tree search.
  *Shows entire TreeSearch.

@param ()
@return Void

*/

void MCTS_Node::DisplayTree(){
  for (MCTS_Node* Child : Children) { // c++11 range-based for loop
      Child->DisplayStats();
    }
  std::cout << "----------------------------------------\n";
  std::cout << GivenGame->Generate_StringRepresentation();
  for (MCTS_Node* Child : Children) { // c++11 range-based for loop
       Child->DisplayTree();
    }

}



//template <class C, template <class C> class M>


class MCTS: public TreeSimulation
{
public:
  double Value;
  double Visits;
  Game* GivenGame;

  std::list<Player*> _Players;
  Player* GivenPlayer;

  //MCTS_Node* TransversedNode;
  MCTS_Node* HeadNode;
  std::list<MCTS_Node*>MCTS_List;
  Game* SimulatedGame;

    MCTS(Game*_Game,std::list<Player*> _GivenPlayers){
      Value  = 0;
      Visits = 0;
      _Players = _GivenPlayers;
      GivenPlayer = *(_GivenPlayers.begin());
      for (Player* _Player : _GivenPlayers){
            printf("MCTS Playerlist:%p\n",(_Player));
      }
      //HeadNode  = NULL;
      //printf("new MCTS_Node's Player:%p\n",Player);
      //std::cin.get();

      HeadNode  = new MCTS_Node(_Game,_GivenPlayers);
      GivenGame = _Game;
    }

    ~MCTS(){
      delete HeadNode;
    }

    MCTS_Node* Algorithm(MCTS_Node* TransversedNode);
    void EvaluateTransversal(MCTS_Node* TransversedNode,Player* GivenPlayer);
    //double BackPropagation(MCTS_Node* TransversedNode,double GivenPlayer);
    void Search(int Depth); //,Player* GivenPlayer
    MCTS* PruneSearch(MCTS_Node*SelectedNode);
    void ParallelSearch(int Depth);


    MCTS* CreateBookMoves();
    MCTS* SaveBookMoves(char* Path);
    MCTS* OpenBookMoves(char* Path);
    //MCTS_Node* Find_Highest_UCB1(std::list<MCTS_Node*>MCTS_List);
    void GetPossibleMoves();

    void CreateChildren();
    void TreeTraversal();
    void CreateNode();
    void RollOut();
};







void MCTS::GetPossibleMoves()
{
    std::list<GameMove*> Moves = SimulatedGame->PossibleMoves();
    std::list<Game*> Games = SimulatedGame->PossibleGames();
}




MCTS_Node* MCTS::Algorithm(MCTS_Node* TransversedNode)
{
  /*
    Helper Function for MCTS::Search & EvaluateTransversal.
    Performs an itteration of the MCTS Algorithm on 'TransversedNode'
  */


//TransversedNode->Children.size()
//int Leaf =TransversedNode->Children.size();

  //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();
  /*
  std::cout << "TransversedNode:" <<TransversedNode << "\n";
  std::cout << "NodeVisits:" <<TransversedNode->NodeVisits << "\n";
  std::cout << "Children:"   <<TransversedNode->Children.size() << "\n";
  */


  //Pause;


  if(TransversedNode->Children.size() == 0){
    //If Node is LeafNode
    //std::cout << "LeafNode Detected  :"   << TransversedNode << "\n";

    if(TransversedNode->NodeVisits == 0){
      //std::cout << "About to rool out on:"   << TransversedNode << "\n";

      return TransversedNode->RollOut();
    }


    //std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();

    //UTTT* UTTT_Game = static_cast<UTTT*>(TransversedNode->GivenGame);
    std::list<Game*> Games = TransversedNode->GivenGame->PossibleGames();
    //std::cout << "Adding Children Size:" << Games.size() << "\n";
    if (Games.size() == 0)
    {
      return TransversedNode;
    }

    printf("TransversedNode->GivenGame->Players.begin():%p\n",*(TransversedNode->GivenGame->_Players.begin()));
    TransversedNode->AddChildren(Games);


    MCTS_Node* NextNode = *TransversedNode->Children.begin();
    //std::cout << "Selecting Next Node:" <<NextNode << "\n";
    return Algorithm(NextNode);
    //return NULL;

  }
  else{

    //Not Leaf Node, Transverse TreeSearch: Find Max Child UCB1 value.
    MCTS_Node* MAXNode = TransversedNode->Find_MAX_UCB1_Child();
    //TODO: Implement Node Visits within BackProp...
    //TransversedNode->NodeVisits++;
    return Algorithm(MAXNode);
    //return NULL;
  }
}


//



void MCTS::EvaluateTransversal(MCTS_Node* TransversedNode,Player* GivenPlayer)
{
/*
  Helper Function for MCTS::Search.
  Performs an itteration of the MCTS Algorithm on the parameter 'TransversedNode'.
  Depending on the winner of the Transversal/RollOut aggregates the appropriate
  Value through MCTS's Back Propagation.
*/
    TransversedNode = Algorithm(TransversedNode);
    std::cout << TransversedNode->GivenGame->Generate_StringRepresentation();

    TransversedNode->BackPropagation(TransversedNode->GivenGame->TestForWinner());
}




void MCTS::Search(int Depth)
{
  /*
    Interface to initate MCTS Tree Searches.
    Given Depth,
  */
    std::cout << "Searching Depth:" << Depth << "\n";
    for (int i = 0; i < Depth; i++) {
      //printf("\tDepth: %d\n",i);
      EvaluateTransversal(HeadNode,GivenPlayer);
    }
//Pause
    HeadNode->DisplayTree(2);


}


MCTS* MCTS::PruneSearch(MCTS_Node*SelectedNode)
{

    return NULL;
}


MCTS* MCTS::CreateBookMoves()
{

    return NULL;
}


MCTS* MCTS::SaveBookMoves(char* Path)
{

    return NULL;
}


MCTS* MCTS::OpenBookMoves(char* Path)
{

    return NULL;
}



void MCTS::ParallelSearch(int Depth)
{
    std::cout << "Searching Depth:" << Depth << "\n";

}


#endif //MCTS_CU
