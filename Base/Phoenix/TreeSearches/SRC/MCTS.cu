#include "hip/hip_runtime.h"
#ifndef MCTS_CU
#define MCTS_CU


#include <iostream>
#include <string>
#include <list>
#include <cmath>
#include <bits/stdc++.h>

#include "TreeSearch.cu"

#include "../../Games/SRC/Game.cu"
#include "../../Games/SRC/TTT.cu"
#include "../../Games/SRC/UTTT.cu"
class MCTS_Node
{
private:
    int    NodeVisits;
    int    Depth;
    double ValueSum;
    double SimulationRep;
    bool LeafNode;

    MCTS_Node*           Parent;
    std::list<MCTS_Node> Children;
public:
    MCTS_Node(){}
    MCTS_Node(void* Instance){}

    ~MCTS_Node(){}
    double     Find_UCB1();
    int        AddChildren(std::list<void*> PossibleMoves);

};

//Preform MonteCarlo's UCB1 evaluation algorithm on a given node.
double MCTS_Node::Find_UCB1(){
  double ExploreBy = 1.4142;
  if(NodeVisits == 0)
	{
		return INT_MAX;
	}
  //Preform UCB1 Formula
  return (ValueSum/NodeVisits + ExploreBy*sqrt(log((float)Parent->NodeVisits)/NodeVisits));
}



//For each element within a list of PossibleInstances(Different Game States)
//Add as different Childeren
int MCTS_Node::AddChildren(std::list<void*> PossibleInstances){
  int ChildrenAdded = 0;
  for (void* Instance : PossibleInstances){
      Children.push_back(MCTS_Node(Instance));
      ChildrenAdded++;
  }
  return ChildrenAdded;
}






class MCTS : public TreeSimulation
{
public:
  int Nodes;
  MCTS_Node* TransversedNode;
  MCTS_Node* HeadNode;
  std::list<MCTS_Node*>MCTS_List;

  Game* SimulatedGame;
    MCTS(){}
    ~MCTS(){}
    void Search(int Depth);
    MCTS_Node* Find_Highest_UCB1(std::list<MCTS_Node*>MCTS_List);
    void GetPossibleMoves();

    void CreateChildren();
    void TreeTraversal();
    void CreateNode();
    void RollOut();
};


MCTS_Node* MCTS::Find_Highest_UCB1(std::list<MCTS_Node*>MCTS_List){
  double     HighestValue = -DBL_MAX;
  double     NodesValue;
  MCTS_Node* HighestNode  = NULL;

  for (MCTS_Node Node : MCTS_List){
      NodesValue = Node.Find_UCB1();
      if (HighestValue < NodesValue)
      {
        HighestNode = &Node;
      }
  }
  //Note: Doesnt account for NULL Node
  return HighestNode;
}

// Provide implementation for the first method
void MCTS::GetPossibleMoves()
{
    std::list<GameMove*> Moves = SimulatedGame->PossibleMoves();
    std::list<Game*> Games = SimulatedGame->PossibleGames();
}

// Provide implementation for the first method
void MCTS::Search(int Depth)
{
    std::cout << "Searching Depth:" << Depth << "\n";

}



#endif //MCTS_CU
